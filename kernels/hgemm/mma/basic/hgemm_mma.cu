#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only
// support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define LDMATRIX_X1(R, addr)                                                   \
  asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n"        \
               : "=r"(R)                                                       \
               : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr)                                              \
  asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n"    \
               : "=r"(R0), "=r"(R1)                                            \
               : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                      \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"     \
      : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                                 \
      : "r"(addr))
#define LDMATRIX_X1_T(R, addr)                                                 \
  asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n"  \
               : "=r"(R)                                                       \
               : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr)                                            \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n"       \
      : "=r"(R0), "=r"(R1)                                                     \
      : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr)                                    \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, "      \
      "[%4];\n"                                                                \
      : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                                 \
      : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1)            \
  asm volatile(                                                                \
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, "  \
      "%4, %5}, {%6, %7}, {%8, %9};\n"                                         \
      : "=r"(RD0), "=r"(RD1)                                                   \
      : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0),  \
        "r"(RC1))

HOST_DEVICE_INLINE
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// only 1 warp per block(32 threads), m16n8k16. A, B, C: all row_major.
template <const int MMA_M = 16, const int MMA_N = 8, const int MMA_K = 16>
__global__ void hgemm_mma_m16n8k16_naive_kernel(half *A, half *B, half *C,
                                                int M, int N, int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M; // 16
  constexpr int BN = MMA_N; // 8
  constexpr int BK = MMA_K; // 16

  __shared__ half s_a[MMA_M][MMA_K]; // 16x16
  __shared__ half s_b[MMA_K][MMA_N]; // 16x8
  __shared__ half s_c[MMA_M][MMA_N]; // 16x8

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int lane_id = tid % WARP_SIZE;                    // 0~31

  // s_a[16][16], 每行16，每线程load 8，需要2线程，共16行，需2x16=32线程
  const int load_smem_a_m = tid / 2;       // row 0~15
  const int load_smem_a_k = (tid % 2) * 8; // col 0,8
  // s_b[16][8], 每行8，每线程load
  // 8，需要1线程，共16行，需16线程，只需一半线程加载
  const int load_smem_b_k = tid; // row 0~31, but only use 0~15
  const int load_smem_b_n = 0;   // col 0
  const int load_gmem_a_m = by * BM + load_smem_a_m; // global m
  const int load_gmem_b_n = bx * BN + load_smem_b_n; // global n
  if (load_gmem_a_m >= M && load_gmem_b_n >= N)
    return;

  uint32_t RC[2] = {0, 0};

#pragma unroll
  for (int k = 0; k < NUM_K_TILES; ++k) {
    // gmem_a -> smem_a
    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) =
        (LDST128BITS(A[load_gmem_a_addr]));

    // gmem_b -> smem_b
    if (lane_id < MMA_K) {
      int load_gmem_b_k = k * MMA_K + load_smem_b_k; // global row of b
      int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
      LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) =
          (LDST128BITS(B[load_gmem_b_addr]));
    }
    __syncthreads();

    uint32_t RA[4];
    uint32_t RB[2];

    // ldmatrix for s_a, ldmatrix.trans for s_b.
    // s_a: (0,1)*8 -> 0,8 -> [(0~15),(0,8)]
    uint32_t load_smem_a_ptr =
        __cvta_generic_to_shared(&s_a[lane_id % 16][(lane_id / 16) * 8]);
    LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], load_smem_a_ptr);
    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(&s_b[lane_id % 16][0]);
    LDMATRIX_X2_T(RB[0], RB[1], load_smem_b_ptr);

    HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0],
              RC[1]);

    __syncthreads();
  }

  // s_c[16][8],
  // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
  // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
  // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
  LDST32BITS(s_c[lane_id / 4][(lane_id % 4) * 2]) = LDST32BITS(RC[0]);
  LDST32BITS(s_c[lane_id / 4 + 8][(lane_id % 4) * 2]) = LDST32BITS(RC[1]);

  __syncthreads();

  // store s_c[16][8]
  if (lane_id < MMA_M) {
    // store 128 bits per memory issue.
    int store_gmem_c_m = by * BM + lane_id;
    int store_gmem_c_n = bx * BN;
    int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
    LDST128BITS(C[store_gmem_c_addr]) = (LDST128BITS(s_c[lane_id][0]));
  }
}

// 128x128, mma2x4, warp4x4(64,32,16)
template <const int MMA_M = 16, const int MMA_N = 8, const int MMA_K = 16,
          const int MMA_TILE_M = 2, const int MMA_TILE_N = 4,
          const int WARP_TILE_M = 4, const int WARP_TILE_N = 4,
          const int A_PAD = 0, const int B_PAD = 0>
__global__ void __launch_bounds__(256)
    hgemm_mma_m16n8k16_mma2x4_warp4x4_kernel(half *A, half *B, half *C, int M,
                                             int N, int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M * MMA_TILE_M * WARP_TILE_M; // 16*2*4=128
  constexpr int BN = MMA_N * MMA_TILE_N * WARP_TILE_N; // 8*4*4=128
  constexpr int BK = MMA_K;                            // 16

  __shared__ half s_a[BM][BK + A_PAD]; // 128*16*2=4KB
  __shared__ half s_b[BK][BN + B_PAD]; // 16*128*2=4KB, 16*(128+16)*2=4.5KB

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE; // 0~31
  const int warp_m = warp_id % 2;      // 0,1
  const int warp_n = warp_id / 2;      // 0,1,2,3

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=16 按行读取
  // A行主序
  // 对于s_a每行16个数据，每个线程读取8个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2;                // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 8; // col 0,8
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=16 BN=128 按行读取
  // B行主序
  // 对于s_b每行128个数据，每个线程读8个数据，需要16个线程；总共16行，需要16x16=256个线程
  int load_smem_b_k = tid / 16;       // row 0~15
  int load_smem_b_n = (tid % 16) * 8; // col 0,8,...,120
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数
  // 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  if (load_gmem_a_m >= M || load_gmem_b_n >= N)
    return;

  uint32_t RC[WARP_TILE_M][WARP_TILE_N][2];
#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      RC[i][j][0] = 0;
      RC[i][j][1] = 0;
    }
  }

#pragma unroll
  for (int k = 0; k < NUM_K_TILES; ++k) {
    // gmem -> smem
    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
    LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) =
        (LDST128BITS(B[load_gmem_b_addr]));
    LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) =
        (LDST128BITS(A[load_gmem_a_addr]));
    __syncthreads();

    // ldmatrix for s_a, ldmatrix.trans for s_b.
    uint32_t RA[WARP_TILE_M][4];
    uint32_t RB[WARP_TILE_N][2];

// smem -> reg
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      int warp_smem_a_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int lane_smem_a_m = warp_smem_a_m + lane_id % 16; // 0~15
      int lane_smem_a_k = (lane_id / 16) * 8;           // 0,8
      uint32_t lane_smem_a_ptr =
          __cvta_generic_to_shared(&s_a[lane_smem_a_m][lane_smem_a_k]);
      LDMATRIX_X4(RA[i][0], RA[i][1], RA[i][2], RA[i][3], lane_smem_a_ptr);
    }

#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int warp_smem_b_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      int lane_smem_b_k = lane_id % 16;  // 0~15
      int lane_smem_b_n = warp_smem_b_n; // 0, MMA_N=8
      uint32_t lane_smem_b_ptr =
          __cvta_generic_to_shared(&s_b[lane_smem_b_k][lane_smem_b_n]);
      LDMATRIX_X2_T(RB[j][0], RB[j][1], lane_smem_b_ptr);
    }

// MMA compute
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        HMMA16816(RC[i][j][0], RC[i][j][1], RA[i][0], RA[i][1], RA[i][2],
                  RA[i][3], RB[j][0], RB[j][1], RC[i][j][0], RC[i][j][1]);
      }
    }
    __syncthreads();
  }

// reg -> gmem, MMA_MxMMA_N=16x8
#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      int store_warp_smem_c_m = warp_m * (MMA_M * WARP_TILE_M) + i * MMA_M;
      int store_warp_smem_c_n = warp_n * (MMA_N * WARP_TILE_N) + j * MMA_N;
      // mapping lane smem index -> global index.
      // [16][8],
      // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
      // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
      // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
      int store_lane_gmem_c_m = by * BM + store_warp_smem_c_m + lane_id / 4;
      int store_lane_gmem_c_n =
          bx * BN + store_warp_smem_c_n + (lane_id % 4) * 2;
      int store_gmem_c_addr_0 = store_lane_gmem_c_m * N + store_lane_gmem_c_n;
      int store_gmem_c_addr_1 =
          (store_lane_gmem_c_m + 8) * N + store_lane_gmem_c_n;
      // TODO: how to use LDST128BITS here ? reverse the loop order ?
      LDST32BITS(C[store_gmem_c_addr_0]) = LDST32BITS(RC[i][j][0]);
      LDST32BITS(C[store_gmem_c_addr_1]) = LDST32BITS(RC[i][j][1]);
    }
  }
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

// only 1 warp per block(32 threads), m16n8k16. A, B, C: all row_major.
void hgemm_mma_m16n8k16_naive(torch::Tensor a, torch::Tensor b,
                              torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;

  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

  hgemm_mma_m16n8k16_naive_kernel<MMA_M, MMA_N, MMA_K>
      <<<grid, block>>>(reinterpret_cast<half *>(a.data_ptr()),
                        reinterpret_cast<half *>(b.data_ptr()),
                        reinterpret_cast<half *>(c.data_ptr()), M, N, K);
}

// 128x128, mma2x4, warp4x4(64,32,16)
void hgemm_mma_m16n8k16_mma2x4_warp4x4(torch::Tensor a, torch::Tensor b,
                                       torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kHalf)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kHalf)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  constexpr int MMA_TILE_M = 2;
  constexpr int MMA_TILE_N = 4;
  constexpr int WARP_TILE_M = 4;
  constexpr int WARP_TILE_N = 4;
  // bank conflicts free via pad = 8, reject fantasy, trust the profile.
  // ncu --metrics l1tex__data_bank_conflicts_pipe_lsu_mem_shared_op_ld
  // ./hgemm_mma_stage.89.debug.bin ncu --metrics
  // sm__sass_l1tex_data_bank_conflicts_pipe_lsu_mem_shared_op_ldsm
  // ./hgemm_mma_stage.89.debug.bin
  constexpr int A_PAD = 8;
  constexpr int B_PAD = 8;
  constexpr int NUM_THREADS =
      (MMA_TILE_M * MMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256

  dim3 block(NUM_THREADS);
  dim3 grid(div_ceil(N, MMA_N * MMA_TILE_N * WARP_TILE_N),
            div_ceil(M, MMA_M * MMA_TILE_M * WARP_TILE_M));

  hgemm_mma_m16n8k16_mma2x4_warp4x4_kernel<MMA_M, MMA_N, MMA_K, MMA_TILE_M,
                                           MMA_TILE_N, WARP_TILE_M, WARP_TILE_N,
                                           A_PAD, B_PAD>
      <<<grid, block>>>(reinterpret_cast<half *>(a.data_ptr()),
                        reinterpret_cast<half *>(b.data_ptr()),
                        reinterpret_cast<half *>(c.data_ptr()), M, N, K);
}
