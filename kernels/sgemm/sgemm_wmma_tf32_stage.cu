#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <mma.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only
// support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
// Support A and B matrix with row-major inorder to compare with the kernels
// using CUDA Cores in sgemm.cu and sgemm_async.cu. also need flag when
// compiling.

HOST_DEVICE_INLINE
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

__global__ void f32x4_tf32x4_kernel(float *x, float *y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = wmma::__float_to_tf32(reg_x.x);
    reg_y.y = wmma::__float_to_tf32(reg_x.y);
    reg_y.z = wmma::__float_to_tf32(reg_x.z);
    reg_y.w = wmma::__float_to_tf32(reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}

// stage2/3/4 (stage2=double buffers+copy async)
// 1. When using shared memory exceeds 48 KB, dynamic shared memory needs to be
// used, i.e., declare a block of dynamic shared memory with extern shared half
// smem[];. When calling the kernel, the size of the dynamic shared memory needs
// to be specified, and smem addressing should be used in a one-dimensional
// array manner.
// 2. Improve L2 Cache locality (Thread Block Swizzle):
// https://zhuanlan.zhihu.com/p/555339335
// 3. __launch_bounds__: avoid error 'too many resources required for launch'
// reference: https://blog.csdn.net/feng__shuai/article/details/124395023
template <const int WMMA_M = 16, const int WMMA_N = 16, const int WMMA_K = 8,
          const int WMMA_TILE_M = 4, const int WMMA_TILE_N = 2,
          const int WARP_TILE_M = 2, const int WARP_TILE_N = 4,
          const int A_PAD = 0, const int B_PAD = 0, const int K_STAGE = 2,
          const bool BLOCK_SWIZZLE = false>
__global__ void
sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_kernel(float *A, float *B, float *C,
                                                 int M, int N, int K) {
  // 256 threads(8 warps) per block.
  // const int bx = blockIdx.x;
  // BLOCK_SWIZZLE 0/1 控制是否使用 block swizzle
  const int bx = ((int)BLOCK_SWIZZLE) * blockIdx.z * gridDim.x + blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K;                             // 8
  __shared__ float s_a[K_STAGE][BM][BK + A_PAD], s_b[K_STAGE][BK][BN + B_PAD];

  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int warp_m = warp_id / 2;      // 0,1,2,3
  const int warp_n = warp_id % 2;      // 0,1

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2;                // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 4; // col 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32;       // row 0~7
  int load_smem_b_n = (tid % 32) * 4; // col 0,4,...,124,...
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数
  // 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float>
      C_frag[WARP_TILE_M][WARP_TILE_N];

#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

#pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) {         // 0, 1
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    uint32_t load_smem_a_ptr =
        __cvta_generic_to_shared(&s_a[k][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr =
        __cvta_generic_to_shared(&s_b[k][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE - 2); // s2->0, s3->1, s4->2
  __syncthreads();

#pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) {
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
        &s_a[smem_sel_next][load_smem_a_m][load_smem_a_k]);
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(
        &s_b[smem_sel_next][load_smem_b_k][load_smem_b_n]);
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        B_frag[WARP_TILE_N];

// compute stage 0
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      wmma::load_matrix_sync(A_frag[i], &s_a[smem_sel][warp_smem_a_m][0],
                             BK + A_PAD);
    }

#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::load_matrix_sync(B_frag[j], &s_b[smem_sel][0][warp_smem_b_n],
                             BN + B_PAD);
    }

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
    __syncthreads();
  }

  // make sure all memory issues ready.
  if ((K_STAGE - 2) > 0) {
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads();
  }
  // processing last (K_STAGE-1) k iters.
  {
#pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          A_frag[WARP_TILE_M];
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          B_frag[WARP_TILE_N];

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        const int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        wmma::load_matrix_sync(A_frag[i], &s_a[stage_sel][warp_smem_a_m][0],
                               BK + A_PAD);
      }

#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        const int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
        wmma::load_matrix_sync(B_frag[j], &s_b[stage_sel][0][warp_smem_b_n],
                               BN + B_PAD);
      }

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

// finally, store back to C matrix.
#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m =
          by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n =
          bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n,
                              C_frag[i][j], N, wmma::mem_row_major);
    }
  }
}

// stage2/3/4 (stage2=double buffers+copy async)
// 1. When using shared memory exceeds 48 KB, dynamic shared memory needs to be
// used, i.e., declare a block of dynamic shared memory with extern shared half
// smem[];. When calling the kernel, the size of the dynamic shared memory needs
// to be specified, and smem addressing should be used in a one-dimensional
// array manner.
// 2. Improve L2 Cache locality (Thread Block Swizzle):
// https://zhuanlan.zhihu.com/p/555339335
// 3. __launch_bounds__: avoid error 'too many resources required for launch'
// reference: https://blog.csdn.net/feng__shuai/article/details/124395023
template <const int WMMA_M = 16, const int WMMA_N = 16, const int WMMA_K = 8,
          const int WMMA_TILE_M = 4, const int WMMA_TILE_N = 2,
          const int WARP_TILE_M = 2, const int WARP_TILE_N = 4,
          const int A_PAD = 0, const int B_PAD = 0, const int K_STAGE = 2,
          const bool BLOCK_SWIZZLE = false>
__global__ void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem_kernel(
    float *A, float *B, float *C, int M, int N, int K) {
  // 256 threads(8 warps) per block.
  // const int bx = blockIdx.x;
  // BLOCK_SWIZZLE 0/1 控制是否使用 block swizzle
  const int bx = ((int)BLOCK_SWIZZLE) * blockIdx.z * gridDim.x + blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, WMMA_K);
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4*2=128
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2*4=128
  constexpr int BK = WMMA_K;                             // 8
  // s2: 2*128*(8+4)*4=12KB, 2*8*(128+4)*4=8.25KB,   ~21KB
  // s3: 3*128*(8+4)*4=18KB, 3*8*(128+4)*4=12.375KB, ~31KB
  // s4: 4*128*(8+4)*4=24KB, 4*8*(128+4)*4=16.5KB,   ~41KB
  extern __shared__ float smem[];
  float *s_a = smem;
  float *s_b = smem + K_STAGE * BM * (BK + A_PAD);
  constexpr int s_a_stage_offset = BM * (BK + A_PAD);
  constexpr int s_b_stage_offset = BK * (BN + B_PAD);

  // 要保证相同的warp下thread执行相同的指令
  const int tid = threadIdx.y * blockDim.x + threadIdx.x;
  const int warp_id = tid / WARP_SIZE; // 0~7 warp_id within block
  const int warp_m = warp_id / 2;      // 0,1,2,3
  const int warp_n = warp_id % 2;      // 0,1

  // 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2;                // row 0~127
  int load_smem_a_k = (tid % 2 == 0) ? 0 : 4; // col 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32;       // row 0~7
  int load_smem_b_n = (tid % 32) * 4; // col 0,4,...,124,...
  // 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数
  // 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float>
      C_frag[WARP_TILE_M][WARP_TILE_N];

#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      wmma::fill_fragment(C_frag[i][j], 0.0);
    }
  }

  // only cvta smem base ptr once for cp.async.
  uint32_t smem_a_base_ptr = __cvta_generic_to_shared(s_a);
  uint32_t smem_b_base_ptr = __cvta_generic_to_shared(s_b);

#pragma unroll
  for (int k = 0; k < (K_STAGE - 1); ++k) {         // 0, 1
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    uint32_t load_smem_a_ptr =
        (smem_a_base_ptr +
         (k * s_a_stage_offset + load_smem_a_m * (BK + A_PAD) + load_smem_a_k) *
             sizeof(float));
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr =
        (smem_b_base_ptr +
         (k * s_b_stage_offset + load_smem_b_k * (BN + B_PAD) + load_smem_b_n) *
             sizeof(float));
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);

    CP_ASYNC_COMMIT_GROUP();
  }

  CP_ASYNC_WAIT_GROUP(K_STAGE - 2); // s2->0, s3->1, s4->2
  __syncthreads();

#pragma unroll
  for (int k = (K_STAGE - 1); k < NUM_K_TILES; k++) {
    // s2/4 can use bitwise ops but s3 can not, so, we use mod
    // ops for all stages kernel. s2: (k + 1)&1, s4: (k + 1)&3
    // s3: (k + 1) % 3
    int smem_sel = (k + 1) % K_STAGE; // s3 k 2->0, k 3->1, k 4->2...
    int smem_sel_next = k % K_STAGE;  // s3 k 2->2, k 3->0, k 4->1...

    // k * WMMA_K, WMMA_K=16 -> (k << 4)
    int load_gmem_a_k = k * WMMA_K + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    int load_gmem_b_k = k * WMMA_K + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;

    // load stage 2, k start from 2
    uint32_t load_smem_a_ptr =
        (smem_a_base_ptr + (smem_sel_next * s_a_stage_offset +
                            load_smem_a_m * (BK + A_PAD) + load_smem_a_k) *
                               sizeof(float));
    CP_ASYNC_CG(load_smem_a_ptr, &A[load_gmem_a_addr], 16);

    uint32_t load_smem_b_ptr =
        (smem_b_base_ptr + (smem_sel_next * s_b_stage_offset +
                            load_smem_b_k * (BN + B_PAD) + load_smem_b_n) *
                               sizeof(float));
    CP_ASYNC_CG(load_smem_b_ptr, &B[load_gmem_b_addr], 16);
    CP_ASYNC_COMMIT_GROUP();

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        A_frag[WARP_TILE_M];
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                   wmma::precision::tf32, wmma::row_major>
        B_frag[WARP_TILE_N];

// compute stage 0
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
      // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
      int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      float *load_smem_a_frag_ptr =
          (s_a + smem_sel * s_a_stage_offset + warp_smem_a_m * (BK + A_PAD) +
           0); // BK=WMMA_K=8
      wmma::load_matrix_sync(A_frag[i], load_smem_a_frag_ptr, BK + A_PAD);
    }

#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
      int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      float *load_smem_b_frag_ptr =
          (s_b + smem_sel * s_b_stage_offset + 0 * (BN + B_PAD) +
           warp_smem_b_n); // BK=WMMA_K=8
      wmma::load_matrix_sync(B_frag[j], load_smem_b_frag_ptr, BN + B_PAD);
    }

#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
      }
    }

    CP_ASYNC_WAIT_GROUP(K_STAGE - 2);
    __syncthreads();
  }

  // make sure all memory issues ready.
  if ((K_STAGE - 2) > 0) {
    CP_ASYNC_WAIT_GROUP(0);
    __syncthreads();
  }
  // processing last (K_STAGE-1) k iters.
  {
#pragma unroll
    for (int k = 0; k < (K_STAGE - 1); k++) {
      const int stage_sel = ((NUM_K_TILES - (K_STAGE - 1) + k) % K_STAGE);
      wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          A_frag[WARP_TILE_M];
      wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K,
                     wmma::precision::tf32, wmma::row_major>
          B_frag[WARP_TILE_N];

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
        // load 2 tiles -> reg, smem a -> frags a, warp_m 0~3
        int warp_smem_a_m = warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
        float *load_smem_a_frag_ptr =
            (s_a + stage_sel * s_a_stage_offset + warp_smem_a_m * (BK + A_PAD) +
             0); // BK=WMMA_K=8
        wmma::load_matrix_sync(A_frag[i], load_smem_a_frag_ptr, BK + A_PAD);
      }

#pragma unroll
      for (int j = 0; j < WARP_TILE_N; ++j) {
        // load 4 tiles -> reg, smem b -> frags b, warp_n 0~2
        int warp_smem_b_n = warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
        float *load_smem_b_frag_ptr =
            (s_b + stage_sel * s_b_stage_offset + 0 * (BN + B_PAD) +
             warp_smem_b_n); // BK=WMMA_K=8
        wmma::load_matrix_sync(B_frag[j], load_smem_b_frag_ptr, BN + B_PAD);
      }

#pragma unroll
      for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
          wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
        }
      }
    }
  }

// finally, store back to C matrix.
#pragma unroll
  for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
    for (int j = 0; j < WARP_TILE_N; ++j) {
      const int store_gmem_a_m =
          by * BM + warp_m * (WMMA_M * WARP_TILE_M) + i * WMMA_M;
      const int store_gmem_a_n =
          bx * BN + warp_n * (WMMA_N * WARP_TILE_N) + j * WMMA_N;
      wmma::store_matrix_sync(C + store_gmem_a_m * N + store_gmem_a_n,
                              C_frag[i][j], N, wmma::mem_row_major);
    }
  }
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

// 128x128 w/o dynamic smem
#define LAUNCH_16168_STAGE_SWIZZLE_KERNEL(stages, stride)                      \
  {                                                                            \
    const int N_SWIZZLE = (N + (stride) - 1) / (stride);                       \
    dim3 block(NUM_THREADS);                                                   \
    dim3 grid((div_ceil(N, BN) + N_SWIZZLE - 1) / N_SWIZZLE, div_ceil(M, BM),  \
              N_SWIZZLE);                                                      \
    sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_kernel<                          \
        WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,         \
        WARP_TILE_N, A_PAD, B_PAD, (stages), true>                             \
        <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),             \
                          reinterpret_cast<float *>(b.data_ptr()),             \
                          reinterpret_cast<float *>(c.data_ptr()), M, N, K);   \
  }

#define LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(stages)                           \
  {                                                                            \
    dim3 block(NUM_THREADS);                                                   \
    dim3 grid(div_ceil(N, BN), div_ceil(M, BM));                               \
    sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_kernel<                          \
        WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,         \
        WARP_TILE_N, A_PAD, B_PAD, (stages), false>                            \
        <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),             \
                          reinterpret_cast<float *>(b.data_ptr()),             \
                          reinterpret_cast<float *>(c.data_ptr()), M, N, K);   \
  }

// 128x128 w dynamic smem, 98304=96KB < Ampere, Ada, Hopper ...
#define LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(stages, stride)                \
  {                                                                            \
    const int smem_max_size = ((stages) * BM * (BK + A_PAD) * sizeof(float) +  \
                               (stages) * BK * (BN + B_PAD) * sizeof(float));  \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                      \
        sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem_kernel<                \
            WMMA_M), WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,     \
            WARP_TILE_N, A_PAD, B_PAD, (stages), true>,                        \
        hipFuncAttributeMaxDynamicSharedMemorySize, 98304);                   \
    const int N_SWIZZLE = (N + (stride) - 1) / (stride);                       \
    dim3 block(NUM_THREADS);                                                   \
    dim3 grid((div_ceil(N, BN) + N_SWIZZLE - 1) / N_SWIZZLE, div_ceil(M, BM),  \
              N_SWIZZLE);                                                      \
    sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem_kernel<                    \
        WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,         \
        WARP_TILE_N, A_PAD, B_PAD, (stages), true>                             \
        <<<grid, block, smem_max_size>>>(                                      \
            reinterpret_cast<float *>(a.data_ptr()),                           \
            reinterpret_cast<float *>(b.data_ptr()),                           \
            reinterpret_cast<float *>(c.data_ptr()), M, N, K);                 \
  }

#define LAUNCH_16168_STAGE_NO_SWIZZLE_DSMEM_KERNEL(stages)                     \
  {                                                                            \
    const int smem_max_size = ((stages) * BM * (BK + A_PAD) * sizeof(float) +  \
                               (stages) * BK * (BN + B_PAD) * sizeof(float));  \
    hipFuncSetAttribute(reinterpret_cast<const void*>(                                                      \
        sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem_kernel<                \
            WMMA_M), WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,     \
            WARP_TILE_N, A_PAD, B_PAD, (stages), false>,                       \
        hipFuncAttributeMaxDynamicSharedMemorySize, 98304);                   \
    dim3 block(NUM_THREADS);                                                   \
    dim3 grid(div_ceil(N, BN), div_ceil(M, BM));                               \
    sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem_kernel<                    \
        WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N, WARP_TILE_M,         \
        WARP_TILE_N, A_PAD, B_PAD, (stages), false>                            \
        <<<grid, block, smem_max_size>>>(                                      \
            reinterpret_cast<float *>(a.data_ptr()),                           \
            reinterpret_cast<float *>(b.data_ptr()),                           \
            reinterpret_cast<float *>(c.data_ptr()), M, N, K);                 \
  }

// 128x128 w/o dynamic smem
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages(torch::Tensor a, torch::Tensor b,
                                               torch::Tensor c, int stages,
                                               bool swizzle,
                                               int swizzle_stride) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  const int Na = M * K;
  const int Nb = K * N;
  constexpr int T = 256;

  f32x4_tf32x4_kernel<<<((Na + T * 4 - 1) / (T * 4)), T>>>(
      reinterpret_cast<float *>(a.data_ptr()),
      reinterpret_cast<float *>(a.data_ptr()), Na);

  f32x4_tf32x4_kernel<<<((Nb + T * 4 - 1) / (T * 4)), T>>>(
      reinterpret_cast<float *>(b.data_ptr()),
      reinterpret_cast<float *>(b.data_ptr()), Nb);

  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 8;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2;
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  // s_a 2 ways bank conflicts within warp, after pad 4 -> 2 ways bank
  // conflicts. s_b 8 ways bank conflicts within warp, after pad 4 -> 4 ways
  // bank conflicts. so, the best padding policy for s_a and s_b is A_PAD=0,
  // B_PAD=0/4/8. B_PAD consume 16x~ less smem than A_PAD, 8xB_PAD vs 128xA_PAD.
  constexpr int A_PAD = 0;
  constexpr int B_PAD = 0;
  constexpr int NUM_THREADS =
      (WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M;
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N;
  constexpr int BK = WMMA_K;
  // s2: 2*128*(8)*4=8KB,  2*8*(128+0~4)*4=8.25KB,   12~13KB
  // s3: 3*128*(8)*4=12KB, 3*8*(128+0~4)*4=12.375KB, 24~25KB
  // s4: 4*128*(8)*4=16KB, 4*8*(128+0~4)*4=16.5KB,   32~33KB

  if (swizzle) {
    assert(swizzle_stride % 256 == 0);
    switch (stages) {
    case 2:
      LAUNCH_16168_STAGE_SWIZZLE_KERNEL(2, swizzle_stride);
      break;
    case 3:
      LAUNCH_16168_STAGE_SWIZZLE_KERNEL(3, swizzle_stride);
      break;
    case 4:
      LAUNCH_16168_STAGE_SWIZZLE_KERNEL(4, swizzle_stride);
      break;
    default:
      LAUNCH_16168_STAGE_SWIZZLE_KERNEL(2, swizzle_stride);
      break;
    }
  } else {
    switch (stages) {
    case 2:
      LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(2);
      break;
    case 3:
      LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(3);
      break;
    case 4:
      LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(4);
      break;
    default:
      LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(2);
      break;
    }
  }
}

// 128x128 with dynamic smem
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem(torch::Tensor a,
                                                     torch::Tensor b,
                                                     torch::Tensor c,
                                                     int stages, bool swizzle,
                                                     int swizzle_stride) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  const int Na = M * K;
  const int Nb = K * N;
  constexpr int T = 256;

  f32x4_tf32x4_kernel<<<((Na + T * 4 - 1) / (T * 4)), T>>>(
      reinterpret_cast<float *>(a.data_ptr()),
      reinterpret_cast<float *>(a.data_ptr()), Na);

  f32x4_tf32x4_kernel<<<((Nb + T * 4 - 1) / (T * 4)), T>>>(
      reinterpret_cast<float *>(b.data_ptr()),
      reinterpret_cast<float *>(b.data_ptr()), Nb);

  constexpr int WMMA_M = 16;
  constexpr int WMMA_N = 16;
  constexpr int WMMA_K = 8;
  constexpr int WMMA_TILE_M = 4;
  constexpr int WMMA_TILE_N = 2;
  constexpr int WARP_TILE_M = 2;
  constexpr int WARP_TILE_N = 4;
  // s_a 2 ways bank conflicts within warp, after pad 4 -> 2 ways bank
  // conflicts. s_b 8 ways bank conflicts within warp, after pad 4 -> 4 ways
  // bank conflicts. so, the best padding policy for s_a and s_b is A_PAD=0,
  // B_PAD=0/4/8. B_PAD consume 16x~ less smem than A_PAD, 8xB_PAD vs 128xA_PAD.
  constexpr int A_PAD = 0;
  constexpr int B_PAD = 0;
  constexpr int NUM_THREADS =
      (WMMA_TILE_M * WMMA_TILE_N * WARP_SIZE); // 2 * 4 * 32 = 256
  constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M;
  constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N;
  constexpr int BK = WMMA_K;
  // s2: 2*128*(8)*4=8KB,  2*8*(128+0~4)*4=8.25KB,   12~13KB
  // s3: 3*128*(8)*4=12KB, 3*8*(128+0~4)*4=12.375KB, 24~25KB
  // s4: 4*128*(8)*4=16KB, 4*8*(128+0~4)*4=16.5KB,   32~33KB

  if (swizzle) {
    assert(swizzle_stride % 256 == 0);
    switch (stages) {
    case 2:
      LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(2, swizzle_stride);
      break;
    case 3:
      LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(3, swizzle_stride);
      break;
    case 4:
      LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(4, swizzle_stride);
      break;
    case 5:
      LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(5, swizzle_stride);
      break;
    default:
      LAUNCH_16168_STAGE_SWIZZLE_DSMEM_KERNEL(2, swizzle_stride);
      break;
    }
  } else {
    switch (stages) {
    case 2:
      LAUNCH_16168_STAGE_NO_SWIZZLE_DSMEM_KERNEL(2);
      break;
    case 3:
      LAUNCH_16168_STAGE_NO_SWIZZLE_DSMEM_KERNEL(3);
      break;
    case 4:
      LAUNCH_16168_STAGE_NO_SWIZZLE_DSMEM_KERNEL(4);
      break;
    case 5:
      LAUNCH_16168_STAGE_NO_SWIZZLE_DSMEM_KERNEL(5);
    default:
      LAUNCH_16168_STAGE_NO_SWIZZLE_KERNEL(2);
      break;
    }
  }
}
