#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <torch/extension.h>
#include <torch/types.h>

#include "hipblas.h"

void cublas_sgemm(float *A, float *B, float *C, size_t M, size_t N, size_t K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);
}

void cublas_sgemm_tf32(float *A, float *B, float *C, size_t M, size_t N,
                       size_t K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

void sgemm_cublas(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  cublas_sgemm(reinterpret_cast<float *>(a.data_ptr()),
               reinterpret_cast<float *>(b.data_ptr()),
               reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

// cublas tensor op
void sgemm_cublas_tf32(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)

  cublas_sgemm_tf32(reinterpret_cast<float *>(a.data_ptr()),
                    reinterpret_cast<float *>(b.data_ptr()),
                    reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}
