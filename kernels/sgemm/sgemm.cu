#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])

// modified from: https://zhuanlan.zhihu.com/p/657632577
// FP32
// SGEMM naive: compute one c[i,j]
// element per threads, all row major
__global__ void sgemm_naive_f32_kernel(float *a, float *b, float *c, int M,
                                       int N, int K) {
  int n = blockIdx.x * blockDim.x + threadIdx.x;
  int m = blockIdx.y * blockDim.y + threadIdx.y;

  if (m < M && n < N) {
    float psum = 0.0;
#pragma unroll
    for (int k = 0; k < K; k++) {
      // m row in a matrix, n col in b matrix
      psum += a[m * K + k] * b[k * N + n];
    }
    c[m * N + n] = psum; // c[m,n]
  }
}

// SGEMM: Block Tile + K Tile, with smem
// Block Tile (BM, BN) + K Tile (BK=32)
// grid((N + BN - 1) / BN, (M + BM - 1) / BM), block(BN, BM)
// a: MxK, b: KxN, c: MxN, compute: c = a * b, all row major
template <const int BM = 32, const int BN = 32, const int BK = 32>
__global__ void sgemm_sliced_k_f32_kernel(float *a, float *b, float *c, int M,
                                          int N, int K) {
  // [1] Block Tile: 32x32的block处理c上一块32x32的元素计算
  // [2]     K Tile: 使用共享内存，并将K分块为BK大小的块
  __shared__ float s_a[BM][BK], s_b[BK][BN];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = threadIdx.y * blockDim.x + tx; // tid within the block
  // load values to shared memory, 32x32 threads working together
  // to fetch data along the row direction of a and b both for s_a
  // and s_b 32x32x4x2=8KB, we use 32x32 threads within block to
  // load 32x32 elements from global memory to shared memory, namely,
  // each thread will load 1 element.
  int load_smem_a_m = tid / 32; // 0~31, tid / 32, tid / BM, threadIdx.y
  int load_smem_a_k = tid % 32; // 0~31, tid % 32, tid % BK, threadIdx.x
  int load_smem_b_k = tid / 32; // 0~31, tid / 32, tid / BK, threadIdx.y
  int load_smem_b_n = tid % 32; // 0~31, tid % 32, tid % BN, threadIdx.x
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
  // if (load_gmem_a_m >= M || load_gmem_b_n >= N) return;

  float sum = 0.f;
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    int load_gmem_a_k = bk * BK + load_smem_a_k;
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    s_a[load_smem_a_m][load_smem_a_k] = a[load_gmem_a_addr];
    int load_gmem_b_k = bk * BK + load_smem_b_k;
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
    s_b[load_smem_b_k][load_smem_b_n] = b[load_gmem_b_addr];
    __syncthreads();
#pragma unroll
    for (int k = 0; k < BK; ++k) {
      int comp_smem_a_m = load_smem_a_m;
      int comp_smem_b_n = load_smem_b_n;
      sum += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
    }
    __syncthreads();
  }
  int store_gmem_c_m = load_gmem_a_m;
  int store_gmem_c_n = load_gmem_b_n;
  int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
  c[store_gmem_c_addr] = sum;
}

// SGEMM: Block Tile + Thread Tile + K Tile + Vec4, with smem
// BK:TILE_K=8 BM=BN=128
// TM=TN=8 增加计算密度 BM/TM=16 BN/TN=16
// dim3 blockDim(BN/TN, BM/TM);
// dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM)
template <const int BM = 128, const int BN = 128, const int BK = 8,
          const int TM = 8, const int TN = 8>
__global__ void sgemm_t_8x8_sliced_k_f32x4_kernel(float *a, float *b, float *c,
                                                  int M, int N, int K) {
  // [1]  Block Tile: 一个16x16的block处理C上大小为128X128的一个目标块
  // [2] Thread Tile: 每个thread负责计算TM*TN(8*8)个元素，增加计算密度
  // [3]      K Tile: 将K分块，每块BK大小，迭代(K+BK-1/BK)次，
  //                  每次计算TM*TN个元素各自的部分乘累加
  // [4]   Vectorize: 减少load和store指令，使用float4

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tid = threadIdx.y * blockDim.x + tx;   // tid within the block
  __shared__ float s_a[BM][BK], s_b[BK][BN]; // 2*128*8*4=8KB

  // 0. 先计算shared memory中的索引
  // tid和需要加载的smem s_a[BM][BK] 之间的索引关系 BM=128 BK=8 按行读取 A行主序
  // 对于s_a每行8个数据，每个线程读取4个，需要2个线程；总共128行，需要128x2刚好256线程
  int load_smem_a_m = tid / 2; // tid/2 (128/8)*(128/8)=256 threads per block,
                               // tid/2->[0,128), BM=128 0~127
  int load_smem_a_k =
      (tid % 2 == 0) ? 0 : 4; // (tid%2 == 0) ? 0 : 4, col of s_a 0,4
  // tid和需要加载的smem s_b[BK][BN] 之间的索引关系 BK=8 BN=128 按行读取 B行主序
  // 对于s_b每行128个数据，每个线程读4个数据，需要32个线程；总共8行，需要32x8=256个线程
  int load_smem_b_k = tid / 32;       // tid/32, row of s_b 256/32=8 行 0~7
  int load_smem_b_n = (tid % 32) * 4; // (tid % 32) * 4, col of s_b 0,4,...,124
  // 1. 再计算全局内存中的索引
  // 要加载到s_a中的元素对应到A全局内存中的行数
  // 每个block负责出C中大小为BM*BN的块
  int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
  int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c

  float r_c[TM][TN] = {0.0}; // 8x8
  // 2. 先对K进行分块，每块BK大小
  for (int bk = 0; bk < (K + BK - 1) / BK; ++bk) {
    // 加载数据到共享内存smem s_a BM*BK 128*8 vectorize float4
    int load_gmem_a_k = bk * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    FLOAT4(s_a[load_smem_a_m][load_smem_a_k]) = FLOAT4(a[load_gmem_a_addr]);
    // 加载数据到共享内存smem s_b BK*BN 8*128 vectorize float4
    int load_gmem_b_k = bk * BK + load_smem_b_k; // global row of b
    int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
    FLOAT4(s_b[load_smem_b_k][load_smem_b_n]) = FLOAT4(b[load_gmem_b_addr]);
    __syncthreads();
#pragma unroll
    for (int k = 0; k < BK; k++) {
// 3. 每个线程负责计算BM*BN(12x128)中的TM*TN(8x8)个元素
#pragma unroll
      for (int m = 0; m < TM; m++) {
#pragma unroll
        for (int n = 0; n < TN; n++) {
          // k from 0~7，0 ~ BK, ty and tx range from 0 to 15, 16x8=128
          int comp_smem_a_m = ty * TM + m; // 128*8 128/TM(8)=16 M方向 16线程
          int comp_smem_b_n = tx * TN + n; // 8*128 128/TN(8)=16 N方向 16线程
          r_c[m][n] += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
        }
      }
    }
    __syncthreads();
  }

#pragma unroll
  for (int m = 0; m < TM; ++m) {
    int store_gmem_c_m = by * BM + ty * TM + m;
#pragma unroll
    for (int n = 0; n < TN; n += 4) {
      int store_gmem_c_n = bx * BN + tx * TN + n;
      int store_gmem_c_addr = store_gmem_c_m * N + store_gmem_c_n;
      FLOAT4(c[store_gmem_c_addr]) = FLOAT4(r_c[m][n]);
    }
  }
}

template <const int BM = 128, const int BN = 128, const int BK = 8,
          const int TM = 8, const int TN = 8, const int OFFSET = 0>
__global__ void
sgemm_t_8x8_sliced_k_f32x4_bcf_kernel(float *a, float *b, float *c, const int M,
                                      const int N, const int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int tid = ty * blockDim.x + tx;

  __shared__ float s_a[BK][BM + OFFSET];
  __shared__ float s_b[BK][BN + OFFSET];
  // __shared__ float s_a[BK][BM + 4];
  // __shared__ float s_b[BK][BN + 4];

  float r_load_a[TM / 2]; // 4
  float r_load_b[TN / 2]; // 4
  float r_comp_a[TM];
  float r_comp_b[TN];
  float r_c[TM][TN] = {0.0};

  // mapping tid to s_a[BK][BM], for each orginal m-th row, load 4 + 4 K-dim
  // row major values from A matrix, and store it in COL major s_a[BK][BM].
  int load_a_smem_m = tid / 2; // tid / 2，(0,1,2,...,128)
  // (0b00000000 & 0b00000001) << 2 = 0
  // (0b00000001 & 0b00000001) << 2 = 4
  // (0b00000010 & 0b00000001) << 2 = 0
  // (0b00000011 & 0b00000001) << 2 = 4
  int load_a_smem_k = (tid & 1) << 2; // (0,4)
  // mapping tid to s_b[BK][BN], for each orginal k-th row, load 4 + 4 N-dim
  // row major values from B matrix, and store it in ROW major s_b[BK][BN].
  int load_b_smem_k = tid / 32; // 0~8
  // (0b00000000 & 0b00011111) << 2 = 0
  // (0b00000001 & 0b00011111) << 2 = 4
  // (0b00000010 & 0b00011111) << 2 = 8
  // (0b00000011 & 0b00011111) << 2 = 12
  int load_b_smem_n = (tid & 31) << 2; // (0,4,8,12,...,124)

  int load_a_gmem_m = by * BM + load_a_smem_m;
  int load_b_gmem_n = bx * BN + load_b_smem_n;

  if (load_a_gmem_m >= M || load_b_gmem_n >= N)
    return;

  for (int bk = 0; bk < (K + BK - 1) / BK; bk++) {
    int load_a_gmem_k = bk * BK + load_a_smem_k;
    int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
    int load_b_gmem_k = bk * BK + load_b_smem_k;
    int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;
    FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
    FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

    // 0. bank layout analysis: s_a[8][128]
    // 4 bytes per bank(32 banks, total 128 bytes, 32 float values),
    // 1 float per bank. smem banks layout for s_a[8][128]:
    // 8*(128/32)=32 bank layers, 4 layers per k-th row.
    // [k=0][m=  [0],   [1],   [2],...,    [31]]
    // layer_0   [b0],  [b1],  [b2],...,   [b31]
    // [k=0][m=  [32],  [33],  [34],...,   [63]]
    // layer_1   [b0],  [b1],  [b2],...,   [b31]
    // [k=0][m=  [64],  [65],  [66],...,   [95]]
    // layer_2   [b0],  [b1],  [b2],...,   [b31]
    // [k=0][m=  [96],  [97],  [98],...,   [127]]
    // layer_3   [b0],  [b1],  [b2],...,   [b31]
    // ...       ...               ...
    // [k=7][m=  [0],   [1],   [2],...,    [31]]
    // layer_28  [b0],  [b1],  [b2],...,   [b31]
    // [k=7][m=  [32],  [33],  [34],...,   [63]]
    // layer_29  [b0],  [b1],  [b2],...,   [b31]
    // [k=7][m=  [64],  [65],  [66],...,   [95]]
    // layer_30  [b0],  [b1],  [b2],...,   [b31]
    // [k=7][m=  [96],  [97],  [98],...,   [127]]
    // layer_31  [b0],  [b1],  [b2],...,   [b31]
    // 1. bank conficts analysis: s_a[8][128]
    // tid 0   -> m 0,   k 0 -> all access bank 0  (layer_0/4/8/12)
    // tid 1   -> m 0,   k 4 -> all access bank 0  (layer_16/20/24/28)
    // tid 2   -> m 1,   k 0 -> all access bank 1  (layer_0/4/8/12)
    // tid 3   -> m 1,   k 4 -> all access bank 1  (layer_16/20/24/28)
    // tid 4   -> m 2,   k 0 -> all access bank 2  (layer_0/4/8/12)
    // tid 5   -> m 2,   k 4 -> all access bank 2  (layer_16/20/24/28)
    // tid 6   -> m 3,   k 0 -> all access bank 3  (layer_0/4/8/12)
    // tid 7   -> m 3,   k 4 -> all access bank 3  (layer_16/20/24/28)
    // ...        ...           ...                ...
    // tid 28  -> m 14,  k 0 -> all access bank 14 (layer_0/4/8/12)
    // tid 29  -> m 14,  k 4 -> all access bank 14 (layer_16/20/24/28)
    // tid 30  -> m 15,  k 0 -> all access bank 15 (layer_0/2/4/6)
    // tid 31  -> m 15,  k 4 -> all access bank 15 (layer_16/20/24/28)
    // conclusion: we still have bank conflicts for smem_a write access,
    // each 2 consecutive threads within warp access the same bank!
    // thus, we still need 2 memory issues as least per warp.
    s_a[load_a_smem_k][load_a_smem_m] = r_load_a[0];     // e.g layer_0  b0
    s_a[load_a_smem_k + 1][load_a_smem_m] = r_load_a[1]; // e.g layer_4  b0
    s_a[load_a_smem_k + 2][load_a_smem_m] = r_load_a[2]; // e.g layer_8  b0
    s_a[load_a_smem_k + 3][load_a_smem_m] = r_load_a[3]; // e.g layer_12 b0
    // 2. bank layout analysis: s_b[8][128] same as s_a[8][128]
    // 3. bank conficts analysis: s_b[8][128]
    // tid 0   -> k 0, n 0   -> all access bank 0~3   (layer_0)
    // tid 1   -> k 0, n 4   -> all access bank 4~7   (layer_0)
    // tid 2   -> k 0, n 8   -> all access bank 7~11  (layer_0)
    // tid 7   -> k 0, n 28  -> all access bank 28~31 (layer_0)
    // tid 8   -> k 0, n 32  -> all access bank 0~3   (layer_1)
    // ...        ...         ...                 ...
    // tid 15  -> k 0, n 60  -> all access bank 28~31 (layer_1)
    // tid 16  -> k 0, n 64  -> all access bank 0~3   (layer_2)
    // ...        ...         ...                 ...
    // tid 31  -> k 0, n 124 -> all access bank 28~31 (layer_3)
    // conclusion: we still have bank conflicts within warp,
    // 0/8/16/24 -> bank 0~3, 1/9/17/25 -> bank 4~7, etc.
    // thus, we still need 4 memory issues at least per warp.
    FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

    __syncthreads();

#pragma unroll
    for (int tk = 0; tk < BK; tk++) {
      // bank conflicts analysis, tx/ty 0~15, 0~7 bank 4*8=32 bytes
      // tid 0~15 access bank 0~3,  tid 16~31 access bank 4~7, etc.
      // tid 0,  tk 0 -> ty 0 -> [0][0+0~3],[0][64+0~3] -> bank 0~3(layer_0/2),
      // tid 0,  tk 7 -> ty 0 -> [7][0+0~3],[0][64+0~3] -> bank
      // 0~3(layer_28/30), tid 15, tk 0 -> ty 0 -> [0][0+0~3],[0][64+0~3] ->
      // bank 0~3(layer_0/2), tid 15, tk 7 -> ty 0 -> [7][0+0~3],[0][64+0~3] ->
      // bank 0~3(layer_28/30), tid 16, tk 0 -> ty 1 -> [0][0+4~7],[0][64+4~7]
      // -> bank 4~7(layer_0/2), tid 16, tk 7 -> ty 1 -> [7][0+4~7],[0][64+4~7]
      // -> bank 4~7(layer_28/30), tid 31, tk 0 -> ty 1 ->
      // [0][0+4~7],[0][64+4~7] -> bank 4~7(layer_0/2), tid 31, tk 7 -> ty 1 ->
      // [7][0+4~7],[0][64+4~7] -> bank 4~7(layer_28/30), tid 255,tk 0 -> ty 15
      // -> [0][0+60~63],[0][64+60~63] -> bank 28~31(layer_1/3), tid 255,tk 7 ->
      // ty 15 -> [7][0+60~63],[0][64+60~63] -> bank 28~31(layer_29/31),
      FLOAT4(r_comp_a[0]) = FLOAT4(s_a[tk][ty * TM / 2]);
      FLOAT4(r_comp_a[4]) = FLOAT4(s_a[tk][ty * TM / 2 + BM / 2]);
      // if (tid == < 32 && bx == 0 && by == 0) {
      //   printf("tid: %d, tx: %d, ty: %d, [%d][%d]\n", tid, tx, ty, tk, ty *
      //   TM / 2); printf("tid: %d, tx: %d, ty: %d, [%d][%d]\n", tid, tx, ty,
      //   tk, ty * TM / 2 + BM / 2);
      // }
      // conclusion: still have bank conflicts, need 16 memory issues ?

      // tid 0/8/16/24  access bank 0~3,  tid 1/9/17/25  access bank 4~7,
      // tid 2/10/18/26 access bank 8~11, tid 7/15/23/31 access bank 28~31, etc.
      // tid 0, tk 0 -> tx 0 -> [0][0+0~3],[0][64+0~3] -> bank 0~3(layer_0/2),
      // tid 0, tk 7 -> tx 0 -> [7][0+0~3],[0][64+0~3] -> bank 0~3(layer_28/30),
      // tid 1, tk 0 -> tx 1 -> [0][0+4~7],[0][64+4~7] -> bank 4~7(layer_0/2),
      // tid 1, tk 7 -> tx 1 -> [7][0+4~7],[0][64+4~7] -> bank 4~7(layer_28/30),
      FLOAT4(r_comp_b[0]) = FLOAT4(s_b[tk][tx * TN / 2]);
      FLOAT4(r_comp_b[4]) = FLOAT4(s_b[tk][tx * TN / 2 + BN / 2]);
      // conclusion: still have some bank conflicts, need 4 memory issues.

#pragma unroll
      for (int tm = 0; tm < TM; tm++) {
#pragma unroll
        for (int tn = 0; tn < TN; tn++) {
          // r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
          r_c[tm][tn] = __fmaf_rn(r_comp_a[tm], r_comp_b[tn], r_c[tm][tn]);
        }
      }
    }
    // sync per BK.
    __syncthreads();
  }

#pragma unroll
  for (int i = 0; i < TM / 2; i++) {
    int store_c_gmem_m = by * BM + ty * TM / 2 + i;
    int store_c_gmem_n = bx * BN + tx * TN / 2;
    int store_c_gmem_addr = store_c_gmem_m * N + store_c_gmem_n;
    FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
    FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
  }
#pragma unroll
  for (int i = 0; i < TM / 2; i++) {
    int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
    int store_c_gmem_n = bx * BN + tx * TN / 2;
    int store_c_gmem_addr = store_c_gmem_m * N + store_c_gmem_n;
    FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
    FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
  }
}

template <const int BM = 128, const int BN = 128, const int BK = 8,
          const int TM = 8, const int TN = 8, const int OFFSET = 0>
__global__ void sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf_kernel(
    float *a, float *b, float *c, const int M, const int N, const int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int tid = ty * blockDim.x + tx;

  __shared__ float s_a[2][BK][BM + OFFSET];
  __shared__ float s_b[2][BK][BN + OFFSET];

  float r_load_a[TM / 2];
  float r_load_b[TN / 2];
  float r_comp_a[TM];
  float r_comp_b[TN];
  float r_c[TM][TN] = {0.0};

  // mapping tid to s_a[BK][BM], for each orginal m-th row, load 4 + 4 K-dim
  // row major values from A matrix, and store it in COL major s_a[BK][BM].
  int load_a_smem_m = tid / 2; // tid / 2，(0,1,2,...,128)
  // (0b00000000 & 0b00000001) << 2 = 0
  // (0b00000001 & 0b00000001) << 2 = 4
  // (0b00000010 & 0b00000001) << 2 = 0
  // (0b00000011 & 0b00000001) << 2 = 4
  int load_a_smem_k = (tid & 1) << 2; // (0,4)
  // mapping tid to s_b[BK][BN], for each orginal k-th row, load 4 + 4 N-dim
  // row major values from B matrix, and store it in ROW major s_b[BK][BN].
  int load_b_smem_k = tid / 32; // 0~8
  // (0b00000000 & 0b00011111) << 2 = 0
  // (0b00000001 & 0b00011111) << 2 = 4
  // (0b00000010 & 0b00011111) << 2 = 8
  // (0b00000011 & 0b00011111) << 2 = 12
  int load_b_smem_n = (tid & 31) << 2; // (0,4,8,12,...,124)

  int load_a_gmem_m = by * BM + load_a_smem_m;
  int load_b_gmem_n = bx * BN + load_b_smem_n;

  // 1）主循环从bk = 1
  // 开始，第一次数据加载在主循环之前，最后一次计算在主循环之后，这是pipeline
  // 的特点决定的； 2）由于计算和下一次访存使用的Shared
  // Memory不同，因此主循环中每次循环只需要一次__syncthreads()即可
  // 3）由于GPU不能向CPU那样支持乱序执行，主循环中需要先将下一次循环计算需要的Gloabal
  // Memory中的数据load
  // 到寄存器，然后进行本次计算，之后再将load到寄存器中的数据写到Shared
  // Memory，这样在LDG指令向Global
  // Memory做load时，不会影响后续FFMA及其它运算指令的 launch
  // 执行，也就达到了Double Buffering的目的。

  // bk = 0 is loading here, buffer 0

  {
    int load_a_gmem_k = load_a_smem_k;
    int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
    int load_b_gmem_k = load_b_smem_k;
    int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;
    FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
    FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

    s_a[0][load_a_smem_k + 0][load_a_smem_m] = r_load_a[0];
    s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
    s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
    s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
    FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
  }
  // Without this synchronization, accuracy may occasionally be abnormal.
  __syncthreads();

  // bk start from 1，需要注意的是，虽然 bk 从 1 开始，但实际上 bk=1时，使用的是
  // 第0块BK中的数据（已经加载到共享内存s_a[0]和s_b[0]）；bk=2时，实际计算的是第1块
  // BK中的数据。其余以此类推，这个循环结束后，剩下最后一块BK大小的数据需要计算。
  for (int bk = 1; bk < (K + BK - 1) / BK; bk++) {
    int smem_sel = (bk - 1) & 1;
    int smem_sel_next = bk & 1;

    int load_a_gmem_k = bk * BK + load_a_smem_k;
    int load_a_gmem_addr = load_a_gmem_m * K + load_a_gmem_k;
    int load_b_gmem_k = bk * BK + load_b_smem_k;
    int load_b_gmem_addr = load_b_gmem_k * N + load_b_gmem_n;
    FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
    FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

#pragma unroll
    for (int tk = 0; tk < BK; tk++) {
      FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2]);
      FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
      FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2]);
      FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

#pragma unroll
      for (int tm = 0; tm < TM; tm++) {
#pragma unroll
        for (int tn = 0; tn < TN; tn++) {
          // r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
          r_c[tm][tn] = __fmaf_rn(r_comp_a[tm], r_comp_b[tn], r_c[tm][tn]);
        }
      }
    }

    // 对比非double buffers版本，此处不需要__syncthreads()，总共节省了
    // ((K + BK - 1) / BK) - 1 次block内的同步操作。比如，bk=1时，HFMA计算
    // 使用的是s_a[0]和s_b[0]，因此，和s_a[1]和s_b[1]的加载是没有依赖关系的。
    // 从global内存到s_a[1]和s_b[1]和HFMA计算可以并行。s_a[1]和s_b[1]用于
    // 加载下一块BK需要的数据到共享内存。
    s_a[smem_sel_next][load_a_smem_k + 0][load_a_smem_m] = r_load_a[0];
    s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
    s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
    s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
    FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) =
        FLOAT4(r_load_b[0]);

    __syncthreads();
  }

// 计算剩下最后一块BK
#pragma unroll
  for (int tk = 0; tk < BK; tk++) {
    FLOAT4(r_comp_a[0]) = FLOAT4(s_a[1][tk][ty * TM / 2]);
    FLOAT4(r_comp_a[4]) = FLOAT4(s_a[1][tk][ty * TM / 2 + BM / 2]);
    FLOAT4(r_comp_b[0]) = FLOAT4(s_b[1][tk][tx * TN / 2]);
    FLOAT4(r_comp_b[4]) = FLOAT4(s_b[1][tk][tx * TN / 2 + BN / 2]);

#pragma unroll
    for (int tm = 0; tm < TM; tm++) {
#pragma unroll
      for (int tn = 0; tn < TN; tn++) {
        // r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
        r_c[tm][tn] = __fmaf_rn(r_comp_a[tm], r_comp_b[tn], r_c[tm][tn]);
      }
    }
  }

#pragma unroll
  for (int i = 0; i < TM / 2; i++) {
    int store_c_gmem_m = by * BM + ty * TM / 2 + i;
    int store_c_gmem_n = bx * BN + tx * TN / 2;
    int store_c_gmem_addr = store_c_gmem_m * N + store_c_gmem_n;
    FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
    FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
  }
#pragma unroll
  for (int i = 0; i < TM / 2; i++) {
    int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
    int store_c_gmem_n = bx * BN + tx * TN / 2;
    int store_c_gmem_addr = store_c_gmem_m * N + store_c_gmem_n;
    FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
    FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
  }
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define CHECK_TORCH_TENSOR_SHAPE(T, S0, S1)                                    \
  if (((T).size(0) != (S0)) || ((T).size(1) != (S1))) {                        \
    throw std::runtime_error("Tensor size mismatch!");                         \
  }

// SGEMM naive: compute one c[i,j] element per threads, all row major
void sgemm_naive_f32(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 32;
  constexpr int BN = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_naive_f32_kernel<<<grid, block>>>(
      reinterpret_cast<float *>(a.data_ptr()),
      reinterpret_cast<float *>(b.data_ptr()),
      reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

// SGEMM: Block Tile + K Tile, with smem
// Block Tile (BM, BN) + K Tile (BK=32)
// grid((N + BN - 1) / BN, (M + BM - 1) / BM), block(BN, BM)
// a: MxK, b: KxN, c: MxN, compute: c = a * b, all row major
void sgemm_sliced_k_f32(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 32;
  constexpr int BN = 32;
  constexpr int BK = 32;

  dim3 block(BN, BM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_sliced_k_f32_kernel<BM, BN, BK>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

// SGEMM: Block Tile + Thread Tile + K Tile + Vec4, with smem
// BK:TILE_K=8 BM=BN=128
// TM=TN=8 增加计算密度 BM/TM=16 BN/TN=16
// dim3 blockDim(BN/TN, BM/TM);
// dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM)
void sgemm_t_8x8_sliced_k_f32x4(torch::Tensor a, torch::Tensor b,
                                torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 8;

  dim3 block(BN / TN, BM / TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_kernel<BM, BN, BK, TM, TN>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

void sgemm_t_8x8_sliced_k_f32x4_bcf(torch::Tensor a, torch::Tensor b,
                                    torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 8;

  dim3 block(BN / TN, BM / TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_bcf_kernel<BM, BN, BK, TM, TN>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

void sgemm_t_8x8_sliced_k_f32x4_bcf_offset(torch::Tensor a, torch::Tensor b,
                                           torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 8;
  constexpr int OFFSET = 4;

  dim3 block(BN / TN, BM / TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_bcf_kernel<BM, BN, BK, TM, TN, OFFSET>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

void sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf(torch::Tensor a, torch::Tensor b,
                                         torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 8;

  dim3 block(BN / TN, BM / TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf_kernel<BM, BN, BK, TM, TN>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

void sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf_offset(torch::Tensor a,
                                                torch::Tensor b,
                                                torch::Tensor c) {
  CHECK_TORCH_TENSOR_DTYPE(a, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(b, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(c, torch::kFloat32)
  const int M = a.size(0);
  const int K = a.size(1);
  const int N = b.size(1);
  CHECK_TORCH_TENSOR_SHAPE(a, M, K)
  CHECK_TORCH_TENSOR_SHAPE(b, K, N)
  CHECK_TORCH_TENSOR_SHAPE(c, M, N)
  constexpr int BM = 128;
  constexpr int BN = 128;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 8;
  constexpr int OFFSET = 4;

  dim3 block(BN / TN, BM / TM);
  dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);

  sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf_kernel<BM, BN, BK, TM, TN, OFFSET>
      <<<grid, block>>>(reinterpret_cast<float *>(a.data_ptr()),
                        reinterpret_cast<float *>(b.data_ptr()),
                        reinterpret_cast<float *>(c.data_ptr()), M, N, K);
}

// from sgemm_async.cu
void sgemm_t_8x4_sliced_k16_f32x4_bcf_dbuf(torch::Tensor a, torch::Tensor b,
                                           torch::Tensor c);
void sgemm_t_8x4_sliced_k16_f32x4_bcf_dbuf_async(torch::Tensor a,
                                                 torch::Tensor b,
                                                 torch::Tensor c);
void sgemm_t_8x8_sliced_k16_f32x4_bcf_dbuf(torch::Tensor a, torch::Tensor b,
                                           torch::Tensor c);
void sgemm_t_8x8_sliced_k16_f32x4_bcf_dbuf_async(torch::Tensor a,
                                                 torch::Tensor b,
                                                 torch::Tensor c);
void sgemm_t_8x16_sliced_k16_f32x4_bcf_dbuf(torch::Tensor a, torch::Tensor b,
                                            torch::Tensor c);
void sgemm_t_8x16_sliced_k16_f32x4_bcf_dbuf_async(torch::Tensor a,
                                                  torch::Tensor b,
                                                  torch::Tensor c);
// from sgemm_cublas.cu
void sgemm_cublas(torch::Tensor a, torch::Tensor b, torch::Tensor c);
void sgemm_cublas_tf32(torch::Tensor a, torch::Tensor b, torch::Tensor c);
// from sgemm_wmma_tf32_stage.cu
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stage2(torch::Tensor a, torch::Tensor b,
                                               torch::Tensor c);
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stage2_offset(torch::Tensor a,
                                                      torch::Tensor b,
                                                      torch::Tensor c);
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stage3(torch::Tensor a, torch::Tensor b,
                                               torch::Tensor c);
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stage3_offset(torch::Tensor a,
                                                      torch::Tensor b,
                                                      torch::Tensor c);

void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages(torch::Tensor a, torch::Tensor b,
                                               torch::Tensor c, int stages,
                                               bool swizzle,
                                               int swizzle_stride);
void sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem(torch::Tensor a,
                                                     torch::Tensor b,
                                                     torch::Tensor c,
                                                     int stages, bool swizzle,
                                                     int swizzle_stride);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // CUDA Cores
  TORCH_BINDING_COMMON_EXTENSION(sgemm_naive_f32)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_sliced_k_f32)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4_bcf)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4_bcf_offset)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k_f32x4_bcf_dbuf_offset)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x4_sliced_k16_f32x4_bcf_dbuf)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x4_sliced_k16_f32x4_bcf_dbuf_async)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k16_f32x4_bcf_dbuf)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x8_sliced_k16_f32x4_bcf_dbuf_async)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x16_sliced_k16_f32x4_bcf_dbuf)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_t_8x16_sliced_k16_f32x4_bcf_dbuf_async)
  // cuBLAS Tensor Cores
  TORCH_BINDING_COMMON_EXTENSION(sgemm_cublas)
  TORCH_BINDING_COMMON_EXTENSION(sgemm_cublas_tf32)
  // WMMA API Tensor Cores, stage, thread block swizzle, dsmem
  TORCH_BINDING_COMMON_EXTENSION(sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages)
  TORCH_BINDING_COMMON_EXTENSION(
      sgemm_wmma_m16n16k8_mma4x2_warp2x4_stages_dsmem)
}
