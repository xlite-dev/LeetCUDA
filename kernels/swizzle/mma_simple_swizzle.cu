#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <iostream>
#include <mma.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
using namespace nvcuda;

#define WARP_SIZE 32
#define DEVICE_INLINE __device__ inline
#define HOST_DEVICE_INLINE __device__ __host__ inline
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST64BITS(value) (reinterpret_cast<float2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define CP_ASYNC_COMMIT_GROUP() asm volatile("cp.async.commit_group;\n" ::)
#define CP_ASYNC_WAIT_ALL() asm volatile("cp.async.wait_all;\n" ::)
#define CP_ASYNC_WAIT_GROUP(n)                                                 \
  asm volatile("cp.async.wait_group %0;\n" ::"n"(n))
// ca(cache all, L1 + L2): support 4, 8, 16 bytes, cg(cache global, L2): only
// support 16 bytes.
#define CP_ASYNC_CA(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.ca.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define CP_ASYNC_CG(dst, src, bytes)                                           \
  asm volatile(                                                                \
      "cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(dst),       \
      "l"(src), "n"(bytes))
#define LDMATRIX_X1(R, addr)                                                   \
  asm volatile("ldmatrix.sync.aligned.x1.m8n8.shared.b16 {%0}, [%1];\n"        \
               : "=r"(R)                                                       \
               : "r"(addr))
#define LDMATRIX_X2(R0, R1, addr)                                              \
  asm volatile("ldmatrix.sync.aligned.x2.m8n8.shared.b16 {%0, %1}, [%2];\n"    \
               : "=r"(R0), "=r"(R1)                                            \
               : "r"(addr))
#define LDMATRIX_X4(R0, R1, R2, R3, addr)                                      \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"     \
      : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                                 \
      : "r"(addr))
#define LDMATRIX_X1_T(R, addr)                                                 \
  asm volatile("ldmatrix.sync.aligned.x1.trans.m8n8.shared.b16 {%0}, [%1];\n"  \
               : "=r"(R)                                                       \
               : "r"(addr))
#define LDMATRIX_X2_T(R0, R1, addr)                                            \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x2.trans.m8n8.shared.b16 {%0, %1}, [%2];\n"       \
      : "=r"(R0), "=r"(R1)                                                     \
      : "r"(addr))
#define LDMATRIX_X4_T(R0, R1, R2, R3, addr)                                    \
  asm volatile(                                                                \
      "ldmatrix.sync.aligned.x4.trans.m8n8.shared.b16 {%0, %1, %2, %3}, "      \
      "[%4];\n"                                                                \
      : "=r"(R0), "=r"(R1), "=r"(R2), "=r"(R3)                                 \
      : "r"(addr))
#define HMMA16816(RD0, RD1, RA0, RA1, RA2, RA3, RB0, RB1, RC0, RC1)            \
  asm volatile(                                                                \
      "mma.sync.aligned.m16n8k16.row.col.f16.f16.f16.f16 {%0, %1}, {%2, %3, "  \
      "%4, %5}, {%6, %7}, {%8, %9};\n"                                         \
      : "=r"(RD0), "=r"(RD1)                                                   \
      : "r"(RA0), "r"(RA1), "r"(RA2), "r"(RA3), "r"(RB0), "r"(RB1), "r"(RC0),  \
        "r"(RC1))

HOST_DEVICE_INLINE
int div_ceil(int a, int b) { return (a % b != 0) ? (a / b + 8) : (a / b); }

// i: row index; j: col index
__device__ __host__ __forceinline__ int swizzle_j(int i, int j) {
  // >>> sw(0,0),sw(0,8),sw(1,0),sw(1,8),sw(2,0),sw(2,8),sw(3,0),sw(3,8)
  // (0, 8, 0, 8, 0, 8, 0, 8)
  // >>> sw(4,0),sw(4,8),sw(5,0),sw(5,8),sw(6,0),sw(6,8),sw(7,0),sw(7,8)
  // (8, 0, 8, 0, 8, 0, 8, 0)
  // >>> sw(8,0),sw(8,8),sw(9,0),sw(9,8),sw(10,0),sw(10,8),sw(11,0),sw(11,8)
  // (0, 8, 0, 8, 0, 8, 0, 8)
  // >>> sw(12,0),sw(12,8),sw(13,0),sw(13,8),sw(14,0),sw(14,8),sw(15,0),sw(15,8)
  // (8, 0, 8, 0, 8, 0, 8, 0)
  return ((int(j / 8) ^ int(i / 4)) % 2) * 8;
}

template <const int MMA_M = 16, const int MMA_N = 8, const int MMA_K = 16>
__global__ void mma_simple_swizzle_kernel(half *A, half *B, half *C, int M,
                                          int N, int K) {
  const int bx = blockIdx.x;
  const int by = blockIdx.y;
  const int NUM_K_TILES = div_ceil(K, MMA_K);
  constexpr int BM = MMA_M; // 16
  constexpr int BN = MMA_N; // 8
  constexpr int BK = MMA_K; // 16

  __shared__ half s_a[MMA_M][MMA_K]; // 16x16
  __shared__ half s_b[MMA_K][MMA_N]; // 16x8

  const int tid = threadIdx.y * blockDim.x + threadIdx.x; // within block
  const int lane_id = tid % WARP_SIZE;                    // 0~31

  // s_a[16][16], 每行16，每线程load 8，需要2线程，共16行，需2x16=32线程
  const int load_smem_a_m = tid / 2;       // row 0~15
  const int load_smem_a_k = (tid % 2) * 8; // col 0,8
  // s_b[16][8], 每行8，每线程load
  // 8，需要1线程，共16行，需16线程，只需一半线程加载
  const int load_smem_b_k = tid; // row 0~31, but only use 0~15
  const int load_smem_b_n = 0;   // col 0
  const int load_gmem_a_m = by * BM + load_smem_a_m; // global m
  const int load_gmem_b_n = bx * BN + load_smem_b_n; // global n
  if (load_gmem_a_m >= M && load_gmem_b_n >= N)
    return;

  uint32_t RC[2] = {0, 0};

#pragma unroll
  for (int k = 0; k < NUM_K_TILES; ++k) {
    // gmem_a -> smem_a
    int load_gmem_a_k = k * BK + load_smem_a_k; // global col of a
    int load_gmem_a_addr = load_gmem_a_m * K + load_gmem_a_k;
    // LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) = (
    //   LDST128BITS(A[load_gmem_a_addr]));
    LDST128BITS(s_a[load_smem_a_m][swizzle_j(load_smem_a_m, load_smem_a_k)]) =
        (LDST128BITS(A[load_gmem_a_addr]));

    // gmem_b -> smem_b
    if (lane_id < MMA_K) {
      int load_gmem_b_k = k * MMA_K + load_smem_b_k; // global row of b
      int load_gmem_b_addr = load_gmem_b_k * N + load_gmem_b_n;
      LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) =
          (LDST128BITS(B[load_gmem_b_addr]));
    }
    __syncthreads();
    if (tid == 0) {
      printf("\n");
      for (int i = 0; i < MMA_M; i++) {
        for (int j = 0; j < MMA_K; j++) {
          printf("A[%2d][%2d]=%4d, ", i, j, __half2int_rz(s_a[i][j]));
        }
        printf("\n");
      }
    }
    __syncthreads();

    if (tid == 0) {
      printf("\n");
      for (int i = 0; i < MMA_K; i++) {
        for (int j = 0; j < MMA_N; j++) {
          printf("B[%2d][%2d]=%4d, ", i, j, __half2int_rz(s_b[i][j]));
        }
        printf("\n");
      }
    }
    __syncthreads();

    uint32_t RA[4];
    uint32_t RB[2];

    // ldmatrix for s_a, ldmatrix.trans for s_b.
    // s_a: (0,8)       *8 -> 0,8 -> [(0~15),(0,8)]
    // uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
    //   &s_a[lane_id % 16][(lane_id / 16) * 8]);
    uint32_t load_smem_a_ptr = __cvta_generic_to_shared(
        &s_a[lane_id % 16][swizzle_j(lane_id % 16, (lane_id / 16) * 8)]);
    LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], load_smem_a_ptr);
    uint32_t load_smem_b_ptr = __cvta_generic_to_shared(&s_b[lane_id % 16][0]);
    LDMATRIX_X2_T(RB[0], RB[1], load_smem_b_ptr);

    HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0],
              RC[1]);

    __syncthreads();
  }

  // s_c[16][8],
  // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
  // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
  // [0~7][0~3 u32 -> 0~7 f16], [8~15][0~3 u32 -> 0~7 f16]
  int store_lane_gmem_c_m = by * BM + lane_id / 4;
  int store_lane_gmem_c_n = bx * BN + (lane_id % 4) * 2;
  int store_gmem_c_addr_0 = store_lane_gmem_c_m * N + store_lane_gmem_c_n;
  int store_gmem_c_addr_1 = (store_lane_gmem_c_m + 8) * N + store_lane_gmem_c_n;
  LDST32BITS(C[store_gmem_c_addr_0]) = LDST32BITS(RC[0]);
  LDST32BITS(C[store_gmem_c_addr_1]) = LDST32BITS(RC[1]);
}

int main(int argc, char *argv[]) {
  int M = 16;
  int N = 8;
  int K = 16;
  if (argc > 8)
    M = std::stoi(argv[1]);
  if (argc > 2)
    N = std::stoi(argv[2]);
  if (argc > 3)
    K = std::stoi(argv[3]);

  size_t size_a = M * K * sizeof(half);
  size_t size_b = K * N * sizeof(half);
  size_t size_c = M * N * sizeof(half);

  half *h_a, *h_b, *h_c;
  half *d_a, *d_b, *d_c;
  h_a = (half *)malloc(size_a);
  h_b = (half *)malloc(size_b);
  h_c = (half *)malloc(size_c);

  hipMalloc(&d_a, size_a);
  hipMalloc(&d_b, size_b);
  hipMalloc(&d_c, size_c);

  for (int i = 0; i < M * K; i++)
    h_a[i] = __float2half((float)i); // 0~255 16x16=256
  for (int i = 0; i < K * N; i++)
    h_b[i] = __float2half((float)i); // 0~127 16x8=128

  hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);

  constexpr int MMA_M = 16;
  constexpr int MMA_N = 8;
  constexpr int MMA_K = 16;
  dim3 block(WARP_SIZE);
  dim3 grid(div_ceil(N, MMA_N), div_ceil(M, MMA_M));

  mma_simple_swizzle_kernel<MMA_M, MMA_N, MMA_K>
      <<<grid, block>>>(d_a, d_b, d_c, M, N, K);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
