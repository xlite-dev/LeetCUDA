#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h> // NOLINT
#include <cute/tensor.hpp>
#include <stdlib.h>

// z = ax + by + c
template <int kNumElemPerThread = 8>
__global__ void vector_add_local_tile_multi_elem_per_thread_half(
    half *z, int num, const half *x, const half *y, const half a, const half b,
    const half c) {
  using namespace cute;

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= num / kNumElemPerThread) { // 未处理非对齐问题
    return;
  }

  Tensor tz = make_tensor(make_gmem_ptr(z), make_shape(num));
  Tensor tx = make_tensor(make_gmem_ptr(x), make_shape(num));
  Tensor ty = make_tensor(make_gmem_ptr(y), make_shape(num));

  Tensor tzr =
      local_tile(tz, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor txr =
      local_tile(tx, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor tyr =
      local_tile(ty, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));

  Tensor txR = make_tensor_like(txr);
  Tensor tyR = make_tensor_like(tyr);
  Tensor tzR = make_tensor_like(tzr);

  // LDG.128
  copy(txr, txR);
  copy(tyr, tyR);

  half2 a2 = {a, a};
  half2 b2 = {b, b};
  half2 c2 = {c, c};

  auto tzR2 = recast<half2>(tzR);
  auto txR2 = recast<half2>(txR);
  auto tyR2 = recast<half2>(tyR);

#pragma unroll
  for (int i = 0; i < size(tzR2); ++i) {
    // two hfma2 instruction
    tzR2(i) = txR2(i) * a2 + (tyR2(i) * b2 + c2);
  }

  auto tzRx = recast<half>(tzR2);

  // STG.128
  copy(tzRx, tzr);
};
