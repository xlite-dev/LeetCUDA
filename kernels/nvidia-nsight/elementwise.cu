#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// FP32
// ElementWise Add grid(N/256),
// block(256) a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32_kernel(float *a, float *b, float *c,
                                           int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

// ElementWise Add + Vec4
// grid(N/256), block(256/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f32x4_kernel(float *a, float *b, float *c,
                                             int N) {
  int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    float4 reg_a = FLOAT4(a[idx]);
    float4 reg_b = FLOAT4(b[idx]);
    float4 reg_c;
    reg_c.x = reg_a.x + reg_b.x;
    reg_c.y = reg_a.y + reg_b.y;
    reg_c.z = reg_a.z + reg_b.z;
    reg_c.w = reg_a.w + reg_b.w;
    FLOAT4(c[idx]) = reg_c;
  }
}

// FP16
// ElementWise Add grid(N/256),
// block(256) a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f16_kernel(half *a, half *b, half *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = __hadd(a[idx], b[idx]);
}

// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_f16x2_kernel(half *a, half *b, half *c, int N) {
  int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    half2 reg_a = HALF2(a[idx]);
    half2 reg_b = HALF2(b[idx]);
    half2 reg_c;
    reinterpret_cast<half&>(reg_c.x) = __hadd(reinterpret_cast<half&>(reg_a.x), reinterpret_cast<half&>(reg_b.x));
    reinterpret_cast<half&>(reg_c.y) = __hadd(reinterpret_cast<half&>(reg_a.y), reinterpret_cast<half&>(reg_b.y));
    HALF2(c[idx]) = reg_c;
  }
}

__global__ void elementwise_add_f16x8_kernel(half *a, half *b, half *c, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  // manual unroll and improve L2 cache hit rate.
  // Only   L2 cache: load 32  bytes in 1 memory issue (default)
  // Enable L1 cache: load 128 bytes in 1 memory issue (-Xptxas -dlcm=ca)
  // why try fp16x8 within 1 threads? ref:
  // https://zhuanlan.zhihu.com/p/641639133 0. first, tid_0 load 32 bytes in 1
  // memory issue and cache data into L2 cache.
  // 1. then, tid_1,...,tid_3 hit L2 cache and load data from L2 cache directly.
  half2 reg_a_0 = HALF2(a[idx + 0]);
  half2 reg_a_1 = HALF2(a[idx + 2]);
  half2 reg_a_2 = HALF2(a[idx + 4]);
  half2 reg_a_3 = HALF2(a[idx + 6]);
  half2 reg_b_0 = HALF2(b[idx + 0]);
  half2 reg_b_1 = HALF2(b[idx + 2]);
  half2 reg_b_2 = HALF2(b[idx + 4]);
  half2 reg_b_3 = HALF2(b[idx + 6]);
  half2 reg_c_0, reg_c_1, reg_c_2, reg_c_3;
  reinterpret_cast<half&>(reg_c_0.x) = __hadd(reinterpret_cast<half&>(reg_a_0.x), reinterpret_cast<half&>(reg_b_0.x));
  reinterpret_cast<half&>(reg_c_0.y) = __hadd(reinterpret_cast<half&>(reg_a_0.y), reinterpret_cast<half&>(reg_b_0.y));
  reinterpret_cast<half&>(reg_c_1.x) = __hadd(reinterpret_cast<half&>(reg_a_1.x), reinterpret_cast<half&>(reg_b_1.x));
  reinterpret_cast<half&>(reg_c_1.y) = __hadd(reinterpret_cast<half&>(reg_a_1.y), reinterpret_cast<half&>(reg_b_1.y));
  reinterpret_cast<half&>(reg_c_2.x) = __hadd(reinterpret_cast<half&>(reg_a_2.x), reinterpret_cast<half&>(reg_b_2.x));
  reinterpret_cast<half&>(reg_c_2.y) = __hadd(reinterpret_cast<half&>(reg_a_2.y), reinterpret_cast<half&>(reg_b_2.y));
  reinterpret_cast<half&>(reg_c_3.x) = __hadd(reinterpret_cast<half&>(reg_a_3.x), reinterpret_cast<half&>(reg_b_3.x));
  reinterpret_cast<half&>(reg_c_3.y) = __hadd(reinterpret_cast<half&>(reg_a_3.y), reinterpret_cast<half&>(reg_b_3.y));
  if ((idx + 0) < N) {
    HALF2(c[idx + 0]) = reg_c_0;
  }
  if ((idx + 2) < N) {
    HALF2(c[idx + 2]) = reg_c_1;
  }
  if ((idx + 4) < N) {
    HALF2(c[idx + 4]) = reg_c_2;
  }
  if ((idx + 6) < N) {
    HALF2(c[idx + 6]) = reg_c_3;
  }
}

__global__ void elementwise_add_f16x8_pack_kernel(half *a, half *b, half *c,
                                                  int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  // temporary register(memory), .local space in ptx, addressable
  half pack_a[8], pack_b[8], pack_c[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_a[0]) = LDST128BITS(a[idx]); // load 128 bits
  LDST128BITS(pack_b[0]) = LDST128BITS(b[idx]); // load 128 bits

#pragma unroll
  for (int i = 0; i < 8; i += 2) {
    // __hadd2 for half2 x 4
    HALF2(pack_c[i]) = __hadd2(HALF2(pack_a[i]), HALF2(pack_b[i]));
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) {
    LDST128BITS(c[idx]) = LDST128BITS(pack_c[0]);
  }
}

int main(int argc, char *argv[]) {
  constexpr int S = 4096;
  constexpr int K = 4096;
  constexpr int N = S * K;
  int R = 10; // repeat
  if (argc > 1)
    R = std::stoi(argv[1]);
  printf("S=%d, K=%d, R=%d\n", S, K, R);

  half *a_host = (half *)malloc(N * sizeof(half));
  half *a_device;
  hipMalloc((void **)&a_device, N * sizeof(half));
  for (int i = 0; i < N; i++)
    a_host[i] = 1.0;
  hipMemcpy(a_device, a_host, N * sizeof(half), hipMemcpyHostToDevice);

  half *b_host = (half *)malloc(N * sizeof(half));
  half *b_device;
  hipMalloc((void **)&b_device, N * sizeof(half));
  for (int i = 0; i < N; i++)
    b_host[i] = 1.0;
  hipMemcpy(b_device, b_host, N * sizeof(half), hipMemcpyHostToDevice);

  half *c_host = (half *)malloc(N * sizeof(half));
  half *c_device;
  hipMalloc((void **)&c_device, N * sizeof(half));

  // naive elementwise fp16
  {
    dim3 block(1024);
    dim3 grid((N + 1024 - 1) / 1024);

    // warmup
    for (int i = 0; i < 5; ++i)
      elementwise_add_f16_kernel<<<grid, block>>>(a_device, b_device, c_device,
                                                  N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      elementwise_add_f16_kernel<<<grid, block>>>(a_device, b_device, c_device,
                                                  N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("naive  elementwise: %f ms\n", time / (float)R);

    hipMemcpy(c_host, c_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // vectorize elementwise fp16x2
  {
    dim3 block(1024 / 2);
    dim3 grid((N + 1024 - 1) / 1024);

    // warmup
    for (int i = 0; i < 5; ++i)
      elementwise_add_f16x2_kernel<<<grid, block>>>(a_device, b_device,
                                                    c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      elementwise_add_f16x2_kernel<<<grid, block>>>(a_device, b_device,
                                                    c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("f16x2  elementwise: %f ms\n", time / (float)R);

    hipMemcpy(c_host, c_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // unpack elementwise fp16x8
  {
    dim3 block(K / (8)); // 4096/8=512
    dim3 grid(S);

    // warmup
    for (int i = 0; i < 5; ++i)
      elementwise_add_f16x8_kernel<<<grid, block>>>(a_device, b_device,
                                                    c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      elementwise_add_f16x8_kernel<<<grid, block>>>(a_device, b_device,
                                                    c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("unpack elementwise: %f ms\n", time / (float)R);

    hipMemcpy(c_host, c_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // pack elementwise fp16x8
  {
    dim3 block(K / (8)); // 4096/8=512
    dim3 grid(S);

    // warmup
    for (int i = 0; i < 5; ++i)
      elementwise_add_f16x8_pack_kernel<<<grid, block>>>(a_device, b_device,
                                                         c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      elementwise_add_f16x8_pack_kernel<<<grid, block>>>(a_device, b_device,
                                                         c_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("pack   elementwise: %f ms\n", time / (float)R);

    hipMemcpy(c_host, c_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  free(a_host);
  free(b_host);
  free(c_host);
  hipFree(a_device);
  hipFree(b_device);
  hipFree(c_device);
  return 0;
}
