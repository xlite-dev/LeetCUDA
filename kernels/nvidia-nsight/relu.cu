#include <algorithm>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// FP32
// Relu x: N, y: N y=max(0,x)
// grid(N/256), block(K=256)
__global__ void relu_f32_kernel(float *x, float *y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    y[idx] = fmaxf(0.0f, x[idx]);
}

// Relu x: N, y: N y=max(0,x) Vec4
// grid(N/256/4), block(256/4)
__global__ void relu_f32x4_kernel(float *x, float *y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (idx < N) {
    float4 reg_x = FLOAT4(x[idx]);
    float4 reg_y;
    reg_y.x = fmaxf(0.0f, reg_x.x);
    reg_y.y = fmaxf(0.0f, reg_x.y);
    reg_y.z = fmaxf(0.0f, reg_x.z);
    reg_y.w = fmaxf(0.0f, reg_x.w);
    FLOAT4(y[idx]) = reg_y;
  }
}

// FP16
__global__ void relu_f16_kernel(half *x, half *y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    y[idx] = __hmax(__float2half(0.0f), x[idx]);
}

__global__ void relu_f16x2_kernel(half *x, half *y, int N) {
  int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
  if (idx < N) {
    half2 reg_x = HALF2(x[idx]);
    half2 reg_y = HALF2(y[idx]);
    reinterpret_cast<half&>(reg_y.x) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x.x));
    reinterpret_cast<half&>(reg_y.y) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x.y));
    HALF2(y[idx]) = reg_y;
  }
}

__global__ void relu_f16x8_kernel(half *x, half *y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  // manual unroll and improve L2 cache hit rate.
  // Only   L2 cache: load 32  bytes in 1 memory issue (default)
  // Enable L1 cache: load 128 bytes in 1 memory issue (-Xptxas -dlcm=ca)
  // why try fp16x8 within 1 threads? ref:
  // https://zhuanlan.zhihu.com/p/641639133 0. first, tid_0 load 32 bytes in 1
  // memory issue and cache data into L2 cache.
  // 1. then, tid_1,...,tid_3 hit L2 cache and load data from L2 cache directly.
  half2 reg_x_0 = HALF2(x[idx + 0]);
  half2 reg_x_1 = HALF2(x[idx + 2]);
  half2 reg_x_2 = HALF2(x[idx + 4]);
  half2 reg_x_3 = HALF2(x[idx + 6]);
  half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
  reinterpret_cast<half&>(reg_y_0.x) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_0.x));
  reinterpret_cast<half&>(reg_y_0.y) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_0.y));
  reinterpret_cast<half&>(reg_y_1.x) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_1.x));
  reinterpret_cast<half&>(reg_y_1.y) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_1.y));
  reinterpret_cast<half&>(reg_y_2.x) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_2.x));
  reinterpret_cast<half&>(reg_y_2.y) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_2.y));
  reinterpret_cast<half&>(reg_y_3.x) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_3.x));
  reinterpret_cast<half&>(reg_y_3.y) = __hmax(__float2half(0.0f), reinterpret_cast<half&>(reg_x_3.y));
  if ((idx + 0) < N) {
    HALF2(y[idx + 0]) = reg_y_0;
  }
  if ((idx + 2) < N) {
    HALF2(y[idx + 2]) = reg_y_1;
  }
  if ((idx + 4) < N) {
    HALF2(y[idx + 4]) = reg_y_2;
  }
  if ((idx + 6) < N) {
    HALF2(y[idx + 6]) = reg_y_3;
  }
}

__global__ void relu_f16x8_pack_kernel(half *x, half *y, int N) {
  int idx = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
  const half2 z2 = {__float2half(0.0f), __float2half(0.0f)};
  // temporary register(memory), .local space in ptx, addressable
  half pack_x[8], pack_y[8]; // 8x16 bits=128 bits.
  // reinterpret as float4 and load 128 bits in 1 memory issue.
  LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]); // load 128 bits

#pragma unroll
  for (int i = 0; i < 8; i += 2) {
    // __hmax2 for half2 x 4
    HALF2(pack_y[i]) = __hmax2(HALF2(pack_x[i]), z2);
  }
  // reinterpret as float4 and store 128 bits in 1 memory issue.
  if ((idx + 7) < N) {
    LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
  }
}

int main(int argc, char *argv[]) {
  constexpr int S = 4096;
  constexpr int K = 4096;
  constexpr int N = S * K;
  int R = 10; // repeat
  if (argc > 1)
    R = std::stoi(argv[1]);
  printf("S=%d, K=%d, R=%d\n", S, K, R);

  half *x_host = (half *)malloc(N * sizeof(half));
  half *x_device;
  hipMalloc((void **)&x_device, N * sizeof(half));
  for (int i = 0; i < N; i++)
    x_host[i] = (i % 2) ? 1.0 : -1.0;
  hipMemcpy(x_device, x_host, N * sizeof(half), hipMemcpyHostToDevice);

  half *y_host = (half *)malloc(N * sizeof(half));
  half *y_device;
  hipMalloc((void **)&y_device, N * sizeof(half));

  // naive relu fp16
  {
    dim3 block(1024);
    dim3 grid((N + 1024 - 1) / 1024);

    // warmup
    for (int i = 0; i < 5; ++i)
      relu_f16_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      relu_f16_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("naive  relu: %f ms\n", time / (float)R);

    hipMemcpy(y_host, y_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // vectorize relu fp16x2
  {
    dim3 block(1024 / 2);
    dim3 grid((N + 1024 - 1) / 1024);

    // warmup
    for (int i = 0; i < 5; ++i)
      relu_f16x2_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      relu_f16x2_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("f16x2  relu: %f ms\n", time / (float)R);

    hipMemcpy(y_host, y_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // unpack relu fp16x8
  {
    dim3 block(K / (8)); // 4096/8=512
    dim3 grid(S);

    // warmup
    for (int i = 0; i < 5; ++i)
      relu_f16x8_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      relu_f16x8_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("unpack relu: %f ms\n", time / (float)R);

    hipMemcpy(y_host, y_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  // pack relu fp16x8
  {
    dim3 block(K / (8)); // 4096/8=512
    dim3 grid(S);

    // warmup
    for (int i = 0; i < 5; ++i)
      relu_f16x8_pack_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < R; ++i)
      relu_f16x8_pack_kernel<<<grid, block>>>(x_device, y_device, N);
    hipDeviceSynchronize(); // synchronzie

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("pack   relu: %f ms\n", time / (float)R);

    hipMemcpy(y_host, y_device, N * sizeof(half), hipMemcpyDeviceToHost);
  }

  free(x_host);
  free(y_host);
  hipFree(x_device);
  hipFree(y_device);
  return 0;
}
