#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define BLOCK_SIZE 256
#define theta 10000.0f

__global__ void rope_f32_kernel(float *x, float *out, int seq_len, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float x1 = x[idx * 2];
  float x2 = x[idx * 2 + 1];
  int token_pos = idx / N;
  int token_idx = idx % N;
  float exp_v = 1.0f / powf(theta, 2 * token_idx / (N * 2.0f));
  float sin_v = sinf(token_pos * exp_v);
  float cos_v = cosf(token_pos * exp_v);
  float out1 = x1 * cos_v - x2 * sin_v;
  float out2 = x1 * sin_v + x2 * cos_v;
  out[idx * 2] = out1;
  out[idx * 2 + 1] = out2;
}

// another index method of rope.
__global__ void rope_f32_v2_kernel(float *x, float *out, int seq_len, int N) {
  int token_pos = blockIdx.x;
  int tid = threadIdx.x;
  float x1 = x[token_pos * N * 2 + tid * 2];
  float x2 = x[token_pos * N * 2 + tid * 2 + 1];
  float exp_v = 1.0f / powf(theta, 2 * tid / (N * 2.0f));
  float sin_v = sinf(token_pos * exp_v);
  float cos_v = cosf(token_pos * exp_v);
  float out1 = x1 * cos_v - x2 * sin_v;
  float out2 = x1 * sin_v + x2 * cos_v;
  out[token_pos * N * 2 + tid * 2] = out1;
  out[token_pos * N * 2 + tid * 2 + 1] = out2;
}

__global__ void rope_f32x4_pack_kernel(float *x, float *out, int seq_len,
                                       int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float4 x_v = FLOAT4(x[idx * 4]);
  int token_pos = idx / N;
  int token_idx = idx % N;
  float exp_f_v = 1.0f / powf(theta, 2 * token_idx * 2 / (N * 4.0f));
  float exp_s_v = 1.0f / powf(theta, 2 * (token_idx * 2 + 1) / (N * 4.0f));
  float sin_f_v = sinf(token_pos * exp_f_v);
  float cos_f_v = cosf(token_pos * exp_f_v);
  float sin_s_v = sinf(token_pos * exp_s_v);
  float cos_s_v = cosf(token_pos * exp_s_v);
  float4 out_v;
  out_v.x = x_v.x * cos_f_v - x_v.y * sin_f_v;
  out_v.y = x_v.x * sin_f_v + x_v.y * cos_f_v;
  out_v.z = x_v.z * cos_s_v - x_v.w * sin_s_v;
  out_v.w = x_v.z * sin_s_v + x_v.w * cos_s_v;
  FLOAT4(out[idx * 4]) = out_v;
}

// --------------------- PyTorch bindings for custom kernel
// -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

void rope_f32(torch::Tensor x, torch::Tensor out) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(out, torch::kFloat32)
  int seq_len = x.size(0);
  int hidden_size = x.size(1);
  int N = (int)(hidden_size / 2);
  dim3 grid((seq_len * N + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 block(BLOCK_SIZE);
  rope_f32_kernel<<<grid, block>>>(x.data_ptr<float>(), out.data_ptr<float>(),
                                   seq_len, N);
}

void rope_f32_v2(torch::Tensor x, torch::Tensor out) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(out, torch::kFloat32)
  int seq_len = x.size(0);
  int hidden_size = x.size(1);
  int N = (int)(hidden_size / 2);
  dim3 grid(seq_len);
  dim3 block(N);
  rope_f32_v2_kernel<<<grid, block>>>(x.data_ptr<float>(),
                                      out.data_ptr<float>(), seq_len, N);
}

void rope_f32x4_pack(torch::Tensor x, torch::Tensor out) {
  CHECK_TORCH_TENSOR_DTYPE(x, torch::kFloat32)
  CHECK_TORCH_TENSOR_DTYPE(out, torch::kFloat32)
  int seq_len = x.size(0);
  int hidden_size = x.size(1);
  int N = (int)(hidden_size / 4);
  dim3 grid((seq_len * N + BLOCK_SIZE - 1) / BLOCK_SIZE);
  dim3 block(BLOCK_SIZE);
  rope_f32x4_pack_kernel<<<grid, block>>>(x.data_ptr<float>(),
                                          out.data_ptr<float>(), seq_len, N);
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(rope_f32)
  TORCH_BINDING_COMMON_EXTENSION(rope_f32_v2)
  TORCH_BINDING_COMMON_EXTENSION(rope_f32x4_pack)
}
