#include "hip/hip_runtime.h"
#include "utils.h"

// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX
// instruction. The input is Q,K,V, 4D tensor with shape [batch_size, num_heads,
// seq_len, head_dim]. The output is O, a 4D tensor with shape [batch_size,
// num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/pdf/2307.08691

// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [B,H,N,d]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]

// Split Q across MMA(Warps) and keep access KV for all MMA(Warps),
// in order to reduce the comm between warps via smem and warp shuffle.

// MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
// |   64x64   |      warp_KV 0       |
// | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x8) |

// MMA = m16n8k16, Br=16x8=128, Bc=8x16=128, layout: 8 warps
// |  128x128  |      warp_KV 0        |
// | warp_QP 0 | MMA 0 ... MMA 0 (x16) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x16) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x16) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x16) |
// | warp_QP 4 | MMA 4 ... MMA 4 (x16) |
// | warp_QP 5 | MMA 5 ... MMA 5 (x16) |
// | warp_QP 6 | MMA 6 ... MMA 6 (x16) |
// | warp_QP 7 | MMA 7 ... MMA 7 (x16) |

// MMA = m16n8k16, Br=16x8=128, Bc=8x8=64, layout: 8 warps
// |  128x64  |      warp_KV 0        |
// | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
// | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
// | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
// | warp_QP 3 | MMA 3 ... MMA 3 (x8) |
// | warp_QP 4 | MMA 4 ... MMA 4 (x8) |
// | warp_QP 5 | MMA 5 ... MMA 5 (x8) |
// | warp_QP 6 | MMA 6 ... MMA 6 (x8) |
// | warp_QP 7 | MMA 7 ... MMA 7 (x8) |

// Fine-grained tiling at the MMA level for all Q@K^T and P@V results in a
// constant SRAM usage of Br * 16 or Bc * 16 for Q, K, V, leading to an overall
// SRAM complexity of O(Br * 16). Consequently, this approach allows us to run
// faster than SDPA w or w/o MMA Acc F32.

template <
    const int kHeadDim,          // Headdim, 32,64,128
    const int kMmaAtomM,         // MMA Atom M, 16
    const int kMmaAtomN,         // MMA Atom N, 8
    const int kMmaAtomK,         // MMA Atom K, 16
    const int kMmaTileSeqLenQ,   // 4, more MMA(warp), M=16*4=64, Q@K^T=[Br(M),
                                 // d(K)]@[d(K),  Bc(N)]
    const int kMmaTileSeqLenK,   // 1, more MMA(warp), N=8*1 =8,  Q@K^T=[Br(M),
                                 // d(K)]@[d(K),  Bc(N)]
    const int kMmaTileSeqLenP,   // 4, more MMA(warp), M=16*4=64, P@V
                                 // =[Br(M),Bc(K)]@[Bc(K), d(N) ]
    const int kMmaTileHeadDimV,  // 1, more MMA(warp), N=8*1 =8,  P@V
                                 // =[Br(M),Bc(K)]@[Bc(K), d(N) ]
    const int kWarpTileSeqLenQ,  // 1, more values, M, Br=64*1=64, matmul M
    const int kWarpTileSeqLenK,  // 8, more values, N, Bc=8*8 =64, matmul N
    const int kWarpTileSeqLenP,  // 1, more values, M, Br=64*1=64, matmul M
    const int kWarpTileHeadDimV, // 8, more values, N,
                                 // d=8*(1|2|3|4|...)=8|...|32|64|96|128|...
    const int kOStorageAccFloat32, // 0/1, MMA Acc always be fp32, but O
                                   // storage can be fp32 or half.
    const int kStage,              // 1,2
    const int kPadQ,               // Pad Q/K/V 0,8
    const int kPadK, const int kPadV>
__global__ void __launch_bounds__(WARP_SIZE *kMmaTileSeqLenQ *kMmaTileSeqLenK)
    flash_attn_mma_stages_split_q_tiling_qkv_acc_f32_kernel(half *Q, half *K,
                                                            half *V, half *O,
                                                            int QKV_seqlen,
                                                            int QKV_head) {
  // Matmul Layout: Q[Br,d]@K^T[d,Bc] NT, P[Br,Bc]@V[Bc,d] NN.
  // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
  static_assert(kMmaAtomM == 16 && kMmaAtomN == 8 &&
                kMmaAtomK == 16);                                 // m16n8k16
  static_assert(kMmaTileSeqLenQ <= 8 && kMmaTileSeqLenK == 1);    // Q@K^T
  static_assert(kMmaTileSeqLenP <= 8 && kMmaTileHeadDimV == 1);   // P@V
  static_assert(kWarpTileSeqLenQ == 1 && kWarpTileSeqLenK <= 16); // Q@K^T
  // kWarpTileHeadDimV: d=8*(1|2|3|4|...) = 8|...|32|64|96|128|..., etc.
  // e.g, kWarpTileHeadDimV = 8 -> d = 8*8 = 64; 16 -> d = 8*16 = 128.
  static_assert(kWarpTileSeqLenP == 1 &&
                kWarpTileHeadDimV ==
                    (kHeadDim / (kMmaAtomN * kMmaTileHeadDimV))); // P@V
  static_assert(kOStorageAccFloat32 == 0 || kOStorageAccFloat32 == 1);
  static_assert(kStage < 3 && kStage > 0);
  static_assert(kPadQ >= 0 && kPadQ % 8 == 0); // 0,8,16
  static_assert(kPadK >= 0 && kPadK % 8 == 0); // 0,8,16
  static_assert(kPadV >= 0 && kPadV % 8 == 0); // 0,8,16
  constexpr int Br =
      kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*4*1=64
  constexpr int Bc =
      kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; //  8*1*8=64
  static_assert(Br >= Bc); // for shared memory reuse.
  constexpr int kNumThreads =
      WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK; // 32*4*1=128, num threads
  // Now, N must be mutliples of Bc(32/64) for KV tiling across seqlen.
  const int Tc = div_ceil(QKV_seqlen, Bc); // Tc K_tile[Bc,d]
  const float scale = 1.0f / sqrt((float)kHeadDim);

  // grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head), (x,y,z)
  const int QKV_batch_id = blockIdx.y / QKV_head; // Batch size
  const int QKV_head_id = blockIdx.y % QKV_head;  // Head num
  const int Q_tile_id = blockIdx.x;               // Q tile_id, range [0, Tr]
  const int O_tile_id = Q_tile_id;                // O tile_id, same as Q.
  const int tid = threadIdx.x;                    // within block
  const int warp_id = tid / WARP_SIZE;            // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE;            // 0~31
  const int warp_QP = warp_id;                    // 0,1,2,3 or 0~7
  const int warp_KV = 0;                          // 0
  // MMA Layout [Br,Bc]=[64,64], MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout:
  // 4 warps |   64x64   |      warp_KV 0       | | warp_QP 0 | MMA 0 ... MMA 0
  // (x8) | | warp_QP 1 | MMA 1 ... MMA 1 (x8) | | warp_QP 2 | MMA 2 ... MMA 2
  // (x8) | | warp_QP 3 | MMA 3 ... MMA 3 (x8) | MMA Layout [Br,Bc]=[128,128],
  // MMA = m16n8k16, Br=16x8=128, Bc=8x16=128, layout: 8 warps |  128x128  |
  // warp_KV 0        | | warp_QP 0 | MMA 0 ... MMA 0 (x16) | | warp_QP 1 | MMA
  // 1 ... MMA 1 (x16) | | warp_QP 2 | MMA 2 ... MMA 2 (x16) | | warp_QP 3 | MMA
  // 3 ... MMA 3 (x16) | | warp_QP 4 | MMA 4 ... MMA 4 (x16) | | warp_QP 5 | MMA
  // 5 ... MMA 5 (x16) | | warp_QP 6 | MMA 6 ... MMA 6 (x16) | | warp_QP 7 | MMA
  // 7 ... MMA 7 (x16) |
  const int Q_gmem_offset =
      ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) +
       (QKV_head_id * QKV_seqlen * kHeadDim)); // Q [seqlen,d]
  const int K_gmem_offset =
      ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) +
       (QKV_head_id * QKV_seqlen * kHeadDim)); // K [seqlen,d]
  const int V_gmem_offset = Q_gmem_offset;     // V [seqlen,d]
  const int O_gmem_offset = Q_gmem_offset;     // O [seqlen,d]

  // Mapping Q gmem -> tid -> smem, Q[Br,kMmaAtomK]=[64/128,16], 128/256
  // threads.
  int load_smem_Q_Br = (tid / (kNumThreads / Br)); // Br 64, tid / 2, row 0~64
  int load_smem_Q_d =
      (tid % (kNumThreads / Br)) *
      (kMmaAtomK / (kNumThreads / Br)); // (tid % 2) * 8, 0,8,...
  // Mapping K gmem -> tid -> smem, K[Bc,kMmaAtomK]=[64/128,16], 128 threads.
  int load_smem_K_Bc = (tid / (kNumThreads / Bc)); // Bc 64, tid / 2, row 0~64
  int load_smem_K_d =
      (tid % (kNumThreads / Bc)) *
      (kMmaAtomK / (kNumThreads / Bc)); // (tid % 2) * 8, 0,8,...
  // Mapping V gmem -> tid -> smem, V[Bc,d_tile=16]=[64,16], 128 threads.
  int load_smem_V_Bc =
      (tid / (kNumThreads / Bc)); // kMmaAtomN*2 16, tid / 8, row 0~15
  int load_smem_V_d =
      (tid % (kNumThreads / Bc)) *
      ((kMmaAtomN * 2) / (kNumThreads / Bc)); // (tid % 2) * 8, 0,8
  // global Q row of current head for tile [Br,d] per block.
  int load_gmem_Q_Br = Q_tile_id * Br + load_smem_Q_Br;
  if (load_gmem_Q_Br >= QKV_seqlen)
    return;

  // Shared memory for Q,K,V, we don not need additional smem for O
  // collective store which perform via registers reuse and warp shuffle.
  extern __shared__ half smem[];
  // Split Q + Shared KV SMEM + Fine grain tiling, only need O(1) SRAM
  // complexity.
  constexpr int Q_tile_size =
      Br * (kMmaAtomK + kPadQ); // Q[Br,16], 64*16*2=2048 bytes, 2M
  constexpr int K_tile_size = Bc * (kMmaAtomK + kPadK);     // K[Bc,16], 2M
  constexpr int V_tile_size = Bc * (kMmaAtomN * 2 + kPadV); // V[Bc,16], 2M
  half *Q_tile_smem = smem;                                 // 8M/16M
  half *K_tile_smem = Q_tile_smem + kStage * Q_tile_size;   // 8M/16M
  half *V_tile_smem = Q_tile_smem; // V may reuse all Q+K smem after Q@K^T.
  uint32_t smem_Q_base_ptr = __cvta_generic_to_shared(Q_tile_smem);
  uint32_t smem_K_base_ptr = __cvta_generic_to_shared(K_tile_smem);
  uint32_t smem_V_base_ptr = __cvta_generic_to_shared(V_tile_smem);

  // Registers/SMEM for thread block
  // block m_old, l_old, store in lane, use float to
  // keep precision.
  float lane_block_row_max_old[kWarpTileSeqLenQ][2]; // [1][2]
  float lane_block_row_sum_old[kWarpTileSeqLenQ][2]; // [1][2]
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_max_old, -INFINITY);
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_sum_old, 0.0f);

  // Registers for S=Q@K^T/O=P@V
  // registers for QKV, S=Q[Br,d]@K[Bc,d]=[Br,Bc]
  // and O=P[Br,Bc]@V[Bc,d]=[Br,d].
  uint32_t R_Q[kWarpTileSeqLenQ][4]; // [1][4]
  uint32_t R_K[kWarpTileSeqLenK][2]; // [8][2]
  uint32_t R_V[2];                   // [2], S=Q@K, only use 2 32bits registers.
  // registers for current tile_K_seqlen within, [64,64] = S_tile[Br,Bc]
  // = Q_tile[Br,d] * K[Bc,d], each thread hold 2x32 bits regs.
  uint32_t R_S[kWarpTileSeqLenQ][kWarpTileSeqLenK][4]; // [1][8][4], acc f32.
  uint32_t R_O[4]; // registers for O=PV[Br,d]=P@V, [4], only use 4 32bits
                   // registers.
  // registers final Output [D]=final rescale(R_O), [2][2/4][2], 8 or 16 regs.
  // 0/1, MMA Acc always be fp32, but O storage(R_D) can be fp32 or half.
  // FP16 can provide precision to approximately 3-4 decimal places. Thus, if
  // the error does not exceed 1e-3, using FP16 storage is sufficient for most
  // applications.
  uint32_t R_D[kWarpTileSeqLenP][kWarpTileHeadDimV]
              [(kOStorageAccFloat32) ? 4 : 2];
  fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV,
               ((kOStorageAccFloat32) ? 4 : 2)>(R_D, 0);

// <loop over K seqlen>: for K^T[d,seqlen] with K^T_tile[d,Bc]
// tile_K_seqlen: compute S_tile[Br,Bc] = Q@K^T = Q_tile[Br,d] * K^T[d,Bc]
#pragma unroll 1
  for (int tile_K_seqlen = 0; tile_K_seqlen < Tc; ++tile_K_seqlen) {
    // TODO: process last tile_K_seqlen ? pad to multiple of 8.

    // Q/K g2s
    if constexpr (kStage > 1) {
#pragma unroll
      for (int stage = 0; stage < (kStage - 1); ++stage) {
        // Q g2s
        int load_gmem_Q_d = (stage * kMmaAtomK) + load_smem_Q_d; // 0,8
        int load_gmem_Q_addr =
            (Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
        uint32_t load_smem_Q_ptr =
            (smem_Q_base_ptr +
             (stage * Q_tile_size + load_smem_Q_Br * (kMmaAtomK + kPadQ) +
              load_smem_Q_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
          CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();

        // K g2s
        int load_gmem_K_Bc = (tile_K_seqlen * Bc) + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d =
            (stage * kMmaAtomK) + load_smem_K_d; // K [Bc,16] from [seqlen,d]
        int load_gmem_K_addr =
            (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr =
            (smem_K_base_ptr +
             (stage * K_tile_size + load_smem_K_Bc * (kMmaAtomK + kPadK) +
              load_smem_K_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      } // end for stage

      CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
      __syncthreads();
    } // end if kStage > 1

    // <loop over K d>: tile_K_d, kMmaAtomK = 16, K_tile_d[kMmaAtomK,Bc]
    // Matmul with NT layout, Q row major, K^T col major.
    // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
    // S_tile[Br,Bc]=Q_tile[Br,d]@K[Bc,d]
    // <HGEMM in shared memory>
    fill_3D_regs<uint32_t, kWarpTileSeqLenQ, kWarpTileSeqLenK, 4>(R_S, 0);
#pragma unroll
    for (int tile_K_d = 0; tile_K_d < (kHeadDim / kMmaAtomK); ++tile_K_d) {
      // s2 tn 0->0, 1->1, 2->0; s3 tn 0->0, 1->1, 2->2, 3->0;
      int smem_sel = (tile_K_d) % kStage;
      // s2 tn 0->1, 1->0, 2->1; s3 tn 0->2, 1->0, 2->1, 3->2;
      int smem_sel_next = (tile_K_d + (kStage - 1)) % kStage;

      // stages for Q, K
      if constexpr (kStage > 1) {
        if ((tile_K_d + 1) < (kHeadDim / kMmaAtomK)) {
          // next Q tile g2s
          int load_gmem_Q_d = ((tile_K_d + 1) * kMmaAtomK) + load_smem_Q_d;
          int load_gmem_Q_addr =
              (Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
          uint32_t load_smem_Q_ptr =
              (smem_Q_base_ptr +
               (smem_sel_next * Q_tile_size +
                load_smem_Q_Br * (kMmaAtomK + kPadQ) + load_smem_Q_d) *
                   sizeof(half));
#pragma unroll
          for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
            CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
          }
          CP_ASYNC_COMMIT_GROUP();

          // next K tile g2s
          int load_gmem_K_Bc = tile_K_seqlen * Bc + load_smem_K_Bc; // < seqlen
          int load_gmem_K_d = ((tile_K_d + 1) * kMmaAtomK) +
                              load_smem_K_d; // K [Bc,16] from [seqlen,d]
          int load_gmem_K_addr =
              (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
          uint32_t load_smem_K_ptr =
              (smem_K_base_ptr +
               (smem_sel_next * K_tile_size +
                load_smem_K_Bc * (kMmaAtomK + kPadK) + load_smem_K_d) *
                   sizeof(half));
#pragma unroll
          for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
            CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
          }
          CP_ASYNC_COMMIT_GROUP();
        }
      } else {
        // sync load curr Q, K g2s
        // curr Q tile g2s
        int load_gmem_Q_d = (tile_K_d * kMmaAtomK) + load_smem_Q_d;
        int load_gmem_Q_addr =
            (Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
        uint32_t load_smem_Q_ptr =
            (smem_Q_base_ptr +
             (smem_sel * Q_tile_size + load_smem_Q_Br * (kMmaAtomK + kPadQ) +
              load_smem_Q_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Br)); i += 8) {
          CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();

        // curr K tile g2s
        int load_gmem_K_Bc = (tile_K_seqlen * Bc) + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d =
            (tile_K_d * kMmaAtomK) + load_smem_K_d; // K [Bc,16] from [seqlen,d]
        int load_gmem_K_addr =
            (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr =
            (smem_K_base_ptr +
             (smem_sel * K_tile_size + load_smem_K_Bc * (kMmaAtomK + kPadK) +
              load_smem_K_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kMmaAtomK / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
        // Wait curr Q, K tile ready.
        CP_ASYNC_WAIT_GROUP(0);
        __syncthreads();
      } // end if kStage > 1

      // Q s2r
      static_assert(kWarpTileSeqLenQ == 1);
      { // kWarpTileSeqLenQ = 1, Q[Br,d]=[M,K]
        int warp_smem_Q_Br =
            warp_QP * (kMmaAtomM * kWarpTileSeqLenQ) + 0 * kMmaAtomM;
        int lane_smem_Q_Br = warp_smem_Q_Br + lane_id % 16; // 0~15
        int lane_smem_Q_d = (lane_id / 16) * 8;             // 0,8
        uint32_t lane_smem_Q_ptr =
            (smem_Q_base_ptr +
             (smem_sel * Q_tile_size + lane_smem_Q_Br * (kMmaAtomK + kPadQ) +
              lane_smem_Q_d) *
                 sizeof(half));
        LDMATRIX_X4(R_Q[0][0], R_Q[0][1], R_Q[0][2], R_Q[0][3],
                    lane_smem_Q_ptr); // now, R_Q[1][4]
      }

// smem -> reg, load k16n8 from smem K, offset d according tile_K_d.
// ldmatrix.x2 for K_tile_smem, [Bc,kMmaAtomK] from [Bc,d]=[K,N]
#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // load k16n8 via ldmatrix.x2 from K_tile_smem[Bc,d].
        // K[Bc,d] with row major means K^T[d,Bc] in col major.
        int warp_smem_K_Bc =
            warp_KV * (kMmaAtomN * kWarpTileSeqLenK) + j * kMmaAtomN;
        int lane_smem_K_Bc = warp_smem_K_Bc + lane_id % 8; // 0~7
        int lane_smem_K_d = ((lane_id / 8) % 2) * 8;       // 0,8
        uint32_t lane_smem_K_ptr =
            (smem_K_base_ptr +
             (smem_sel * K_tile_size + lane_smem_K_Bc * (kMmaAtomK + kPadK) +
              lane_smem_K_d) *
                 sizeof(half));
        LDMATRIX_X2(R_K[j][0], R_K[j][1], lane_smem_K_ptr); // R_K
      } // end for kWarpTileSeqLenK
      if constexpr (kStage < 2) {
        // Wait Q, K s2r ready if kStage < 2 in order to avoid
        // the next Q, K tile g2s overwrite.
        __syncthreads();
      }

      // MMA compute
      static_assert(kWarpTileSeqLenQ == 1);
      { // kWarpTileSeqLenQ = 1
#pragma unroll
        for (int j = 0; j < kWarpTileSeqLenK; ++j) { // 8, 16, 32, ...
          // MMA always accumulate with F32 dtype for high precision.
          HMMA16816F32(R_S[0][j][0], R_S[0][j][1], R_S[0][j][2], R_S[0][j][3],
                       R_Q[0][0], R_Q[0][1], R_Q[0][2], R_Q[0][3], R_K[j][0],
                       R_K[j][1], R_S[0][j][0], R_S[0][j][1], R_S[0][j][2],
                       R_S[0][j][3]);
        }
      }

      if constexpr (kStage > 1) {
        // Wait next Q, K tile g2s ready.
        CP_ASYNC_WAIT_GROUP(kStage - 2);
        __syncthreads();
      }

    } // end loop over d, S=Q@K^T
    __syncthreads();

    // MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
    // |   64x64   |      warp_KV 0       |
    // | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
    // | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
    // | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
    // | warp_QP 3 | MMA 3 ... MMA 3 (x8) |

    // Online safe softmax, warp/block reduce max/sum, row wise
    float lane_row_max_new[kWarpTileSeqLenQ][2]; // [1][2]
    float lane_row_sum_new[kWarpTileSeqLenQ][2]; // [1][2]
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_max_new, -INFINITY);
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_sum_new, 0.0f);

    static_assert(kWarpTileSeqLenQ == 1);
    // Row max for [Br,Bc] tile, Thread -> Warp -> Block.
    { // kWarpTileSeqLenQ = 1
// Thread level reduce max across kWarpTileSeqLenK dim, namely Bc.
#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // reference:
        // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
        // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
        // The layout of the fragments held by different threads for C.
        // (m16n8k16) Row\Col  0    1    2    3    4    5    6    7 0        T0:
        // {c0, c1}  T1: {c0, c1}  T2: {c0, c1}  T3: {c0, c1} 1        T4: {c0,
        // c1}  T5: {c0, c1}  T6: {c0, c1}  T7: {c0, c1} 2        ...
        // ...
        // 7        T28: {c0, c1}  T29: {c0, c1}  T30: {c0, c1}  T31: {c0, c1}
        // 8        T0: {c2, c3}   T1: {c2, c3}   T2: {c2, c3}   T3: {c2, c3}
        // 9        T4: {c2, c3}   T5: {c2, c3}   T6: {c2, c3}   T7: {c2, c3}
        // 10       ...
        // ...
        // 15       T28: {c2, c3}  T29: {c2, c3}  T30: {c2, c3}  T31: {c2, c3}
        // R_S[][][4] 4 32bit registers with each contains 1 F32 element.
        // (x,y) 0~7->{c0, c1}, (z,w)->8~15 {c2, c3}
        float *t_fptr_S_0_1 = reinterpret_cast<float *>(&(R_S[0][j][0]));
        // This should be the row max after S = (Q @ K^T) / sqrt(d)
        float tmp_max_0 = max(t_fptr_S_0_1[0], t_fptr_S_0_1[1]) * scale;
        float tmp_max_1 = max(t_fptr_S_0_1[2], t_fptr_S_0_1[3]) * scale;
        lane_row_max_new[0][0] = max(lane_row_max_new[0][0], tmp_max_0);
        lane_row_max_new[0][1] = max(lane_row_max_new[0][1], tmp_max_1);
      } // end for kWarpTileSeqLenK

      // Warp level reduce max, warp_size = 4
      // Each thread contains the maximum of 2 rows of Br,
      // and only the values of T0, T4, ..., T28 are used.
      lane_row_max_new[0][0] =
          warp_reduce_max<float, 4>(lane_row_max_new[0][0]);
      lane_row_max_new[0][1] =
          warp_reduce_max<float, 4>(lane_row_max_new[0][1]);
    } // end for kWarpTileSeqLenQ

    static_assert(kWarpTileSeqLenQ == 1);
    // Exp sum and mul scale_factor for [Br,Bc] tile, Thread -> Warp -> Block.
    { // kWarpTileSeqLenQ = 1
      // Use latest global row max without update.
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55;
      float block_row_max_new_0 = lane_row_max_new[0][0];
      // Br 1, row_id, 8~15, 24~31, 40~47, 56~63;
      float block_row_max_new_1 = lane_row_max_new[0][1];

      float block_row_max_old_0 = lane_block_row_max_old[0][0];
      float block_row_max_old_1 = lane_block_row_max_old[0][1];
      // Apply m_new = max(m_old, m_new) here.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);

#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // R_S[][][4] 4 32bit registers with each contains 1 F32 element.
        // (x,y) 0~7->{c0, c1}, (z,w)->8~15 {c2, c3}
        float *t_fptr_S_0_1 = reinterpret_cast<float *>(&(R_S[0][j][0]));
        half *t_hptr_S_0_1 = reinterpret_cast<half *>(&(R_S[0][j][0]));
        // P = Exp(S - m_new), fmaf(x, y, z) = x * y + z in registers;
        t_fptr_S_0_1[0] =
            __expf(__fmaf_rn(t_fptr_S_0_1[0], scale, -block_row_max_new_0));
        t_fptr_S_0_1[1] =
            __expf(__fmaf_rn(t_fptr_S_0_1[1], scale, -block_row_max_new_0));
        t_fptr_S_0_1[2] =
            __expf(__fmaf_rn(t_fptr_S_0_1[2], scale, -block_row_max_new_1));
        t_fptr_S_0_1[3] =
            __expf(__fmaf_rn(t_fptr_S_0_1[3], scale, -block_row_max_new_1));
        lane_row_sum_new[0][0] += (t_fptr_S_0_1[0] + t_fptr_S_0_1[1]);
        lane_row_sum_new[0][1] += (t_fptr_S_0_1[2] + t_fptr_S_0_1[3]);
        // Update R_S for P[Br,Bc] = Exp(S-m), point wise.
        // Also convert F32 -> half for P@V MMA, reuse R_S as P.
        t_hptr_S_0_1[0] = __float2half_rn(t_fptr_S_0_1[0]);
        t_hptr_S_0_1[1] = __float2half_rn(t_fptr_S_0_1[1]);
        t_hptr_S_0_1[2] = __float2half_rn(t_fptr_S_0_1[2]);
        t_hptr_S_0_1[3] = __float2half_rn(t_fptr_S_0_1[3]);
      } // end for kWarpTileSeqLenK

      // Warp level reduce sum, warp_size = 4
      lane_row_sum_new[0][0] =
          warp_reduce_sum<float, 4>(lane_row_sum_new[0][0]);
      lane_row_sum_new[0][1] =
          warp_reduce_sum<float, 4>(lane_row_sum_new[0][1]);
    }

    // Prefetch V g2s before row max/sum for P@V if kStage > 1
    static_assert(kWarpTileSeqLenP == 1);
    { // kWarpTileSeqLenP = 1
      if constexpr (kStage > 1) {
#pragma unroll
        for (int stage = 0; stage < (kStage - 1); ++stage) {
          int load_gmem_V_Bc =
              (tile_K_seqlen * Bc) + load_smem_V_Bc; // < seqlen
          int load_gmem_V_d = (stage * kMmaAtomN * 2) +
                              load_smem_V_d; // V [Bc,16] from [seqlen,d]
          int load_gmem_V_addr =
              (V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
          uint32_t load_smem_V_ptr =
              (smem_V_base_ptr +
               (stage * V_tile_size + load_smem_V_Bc * (kMmaAtomN * 2 + kPadV) +
                load_smem_V_d) *
                   sizeof(half));
#pragma unroll
          for (int i = 0; i < (kMmaAtomN * 2 / (kNumThreads / Bc)); i += 8) {
            CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
          }
          CP_ASYNC_COMMIT_GROUP();
        }
      }
    }

    // according to the A matrix layout for MMA m16n8k16 instruction.
    // reference:
    // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
    // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
    // The layout of the fragments held by different threads for A matrix with
    // .f16. R\C  0    1    2    3    4    5    6    7    8    9   10   11   12
    // 13   14   15 0    T0: {a0, a1}  T1: {a0, a1}  T2: {a0, a1}  T3: {a0, a1}
    // T0: {a4, a5}  T1: {a4, a5}  T2: {a4, a5}  T3: {a4, a5} 1    T4: {a0, a1}
    // T5: {a0, a1}  T6: {a0, a1}  T7: {a0, a1}  T4: {a4, a5}  T5: {a4, a5}  T6:
    // {a4, a5}  T7: {a4, a5} 2    (dashed arrow pointing right)
    // ...
    // 7    T28: {a0, a1}  T29: {a0, a1}  T30: {a0, a1}  T31: {a0, a1}  T28:
    // {a4, a5}  T29: {a4, a5}  T30: {a4, a5}  T31: {a4, a5} 8    T0: {a2, a3}
    // T1: {a2, a3}   T2: {a2, a3}   T3: {a2, a3}   T0: {a6, a7}   T1: {a6, a7}
    // T2: {a6, a7}   T3: {a6, a7} 9    T4: {a2, a3}   T5: {a2, a3}   T6: {a2,
    // a3}   T7: {a2, a3}   T4: {a6, a7}   T5: {a6, a7}   T6: {a6, a7}   T7:
    // {a6, a7} 10   (dashed arrow pointing right)
    // ...
    // 15   T28: {a2, a3}  T29: {a2, a3}  T30: {a2, a3}  T31: {a2, a3}  T28:
    // {a6, a7}  T29: {a6, a7}  T30: {a6, a7}  T31: {a6, a7}

    static_assert(kWarpTileSeqLenP == 1);
    {
      // <Prefetch max/sum values>
      // m = max(m_old, m_new), l = exp(m_old - m) * l_old + l_new (FA2 paper)
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55; Br 1, row_id, 8~15, 24~31,
      // 40~47, 56~63
      float block_row_max_new_0 = lane_row_max_new[0][0];
      float block_row_max_new_1 = lane_row_max_new[0][1];
      float block_row_sum_new_0 = lane_row_sum_new[0][0];
      float block_row_sum_new_1 = lane_row_sum_new[0][1];

      float block_row_max_old_0 = lane_block_row_max_old[0][0];
      float block_row_max_old_1 = lane_block_row_max_old[0][1];
      // NOTE: max(-inf, val) = val.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);
      // Avoid inf value while using m_old for rescaling O.
      block_row_max_old_0 =
          (tile_K_seqlen > 0 ? block_row_max_old_0 : block_row_max_new_0);
      block_row_max_old_1 =
          (tile_K_seqlen > 0 ? block_row_max_old_1 : block_row_max_new_1);
      // rescale factor for O and l, exp(m_old - m) for curr tile [Br,d].
      float rescale_o_factor_0 =
          __expf(block_row_max_old_0 - block_row_max_new_0);
      float rescale_o_factor_1 =
          __expf(block_row_max_old_1 - block_row_max_new_1);

      // Wait V g2s stages ready.
      if constexpr (kStage > 1) {
        CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
        __syncthreads();
      }

// <HGEMM in registers>
#pragma unroll
      for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8, 16, 32, ...
        // Compute d tile, P[Br,Bc]@V[Bc,16] = O[Br,16]
        fill_1D_regs<uint32_t, 4>(R_O, 0); // must clear

        int smem_sel_v = (j / 2) % kStage;
        int smem_sel_v_next = ((j / 2) + (kStage - 1)) % kStage;
        // V g2s, V tile smem [Bc,kMmaAtomN*2]=[64,16]
        if (j % 2 == 0) { // 0,2,4,6,...// curr K tile g2s
          if constexpr (kStage > 1) {
            if (((j / 2) + 1) < (kWarpTileHeadDimV / 2)) {
              int load_gmem_V_Bc =
                  (tile_K_seqlen * Bc) + load_smem_V_Bc; // < seqlen
              int load_gmem_V_d = (((j / 2) + 1) * kMmaAtomN * 2) +
                                  load_smem_V_d; // V [Bc,16] from [seqlen,d]
              int load_gmem_V_addr =
                  (V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
              uint32_t load_smem_V_ptr =
                  (smem_V_base_ptr +
                   (smem_sel_v_next * V_tile_size +
                    load_smem_V_Bc * (kMmaAtomN * 2 + kPadV) + load_smem_V_d) *
                       sizeof(half));
#pragma unroll
              for (int i = 0; i < (kMmaAtomN * 2 / (kNumThreads / Bc));
                   i += 8) {
                CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i],
                            16);
              }
              CP_ASYNC_COMMIT_GROUP();
            } // end if < (kWarpTileHeadDimV / 2)
          } else {
            // no stages for V g2s
            int load_gmem_V_Bc =
                (tile_K_seqlen * Bc) + load_smem_V_Bc; // < seqlen
            int load_gmem_V_d = ((j / 2) * kMmaAtomN * 2) +
                                load_smem_V_d; // V [Bc,16] from [seqlen,d]
            int load_gmem_V_addr =
                (V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
            uint32_t load_smem_V_ptr =
                (smem_V_base_ptr +
                 (smem_sel_v * V_tile_size +
                  load_smem_V_Bc * (kMmaAtomN * 2 + kPadV) + load_smem_V_d) *
                     sizeof(half));
#pragma unroll
            for (int i = 0; i < (kMmaAtomN * 2 / (kNumThreads / Bc)); i += 8) {
              CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i],
                          16);
            }
            CP_ASYNC_COMMIT_GROUP();
            // Wait curr V tile g2s ready.
            CP_ASYNC_WAIT_GROUP(0);
            __syncthreads();
          }
        }

#pragma unroll
        for (int tile_V_Bc = 0; tile_V_Bc < (Bc / kMmaAtomK); ++tile_V_Bc) {
          // Load k16n8 V from smem [Bc,8*2] -> regs, R_V, ldmatrix.x2.trans.
          int warp_smem_V_d = warp_KV * (kMmaAtomN * kWarpTileHeadDimV) +
                              (j % 2) * kMmaAtomN; // d, matmaul N
          int lane_smem_V_Bc =
              tile_V_Bc * kMmaAtomK + lane_id % 16; // 0~15; Bc, matmul K
          int lane_smem_V_d = warp_smem_V_d;        // 0
          uint32_t lane_smem_V_ptr =
              (smem_V_base_ptr +
               (smem_sel_v * V_tile_size +
                lane_smem_V_Bc * (kMmaAtomN * 2 + kPadV) + lane_smem_V_d) *
                   sizeof(half));
          LDMATRIX_X2_T(R_V[0], R_V[1], lane_smem_V_ptr); // R_V
          // Compute P[Br,Bc]@V[Bc,d] = O[Br,d]
          // For R_S[1][8][2], mapping the layout below of P matrix.
          // MMA = m16n8k16, Br=16x4=64, Bc=8x8=64, layout: 4 warps
          // |   64x64   |      warp_KV 0       |
          // | warp_QP 0 | MMA 0 ... MMA 0 (x8) |
          // | warp_QP 1 | MMA 1 ... MMA 1 (x8) |
          // | warp_QP 2 | MMA 2 ... MMA 2 (x8) |
          // | warp_QP 3 | MMA 3 ... MMA 3 (x8) |
          // tile_V_Bc = 0, all curr MMAs(0~4) need slice P[:,  0:16], 0, 1;
          // stored in all MMAs. tile_V_Bc = 1, all curr MMAs(0~4) need slice
          // P[:, 16:32], 2, 3; stored in all MMAs. tile_V_Bc = 2, all curr
          // MMAs(0~4) need slice P[:, 32:48], 4, 5; stored in all MMAs.
          // tile_V_Bc = 3, all curr MMAs(0~4) need slice P[:, 48:64], 6, 7;
          // stored in all MMAs.
          int w = tile_V_Bc * 2; // MMA(Warp) selected, 0, 2, 4, 6
          // MMA always accumulate with F32 dtype for high precision.
          HMMA16816F32(R_O[0], R_O[1], R_O[2], R_O[3], R_S[0][w][0],
                       R_S[0][w][1], R_S[0][w + 1][0], R_S[0][w + 1][1], R_V[0],
                       R_V[1], R_O[0], R_O[1], R_O[2], R_O[3]);
        } // end for V Bc.
        if constexpr (kStage < 2) {
          // Wait curr P@V tile ready if kStage < 2 in order to avoid
          // the next V tile g2s overwrite.
          __syncthreads();
        }

        // Now, we get [Br,8] slice of [Br,d], each warp(MMA) contains m16n8.
        // 0. Rescale O: Online rescaling O each tile_K_seqlen step, need m_new,
        // m_old. m = max(m_old, m_new), O_new[Br,d] = exp(m_old - m) * O_old +
        // P@V use exp(m_old - m_new), not 1/(m_old - m_new). O_new[Br,d] =
        // exp(m_old - m_new) * O_old + P@V
        float *t_fptr_O_0_1 = reinterpret_cast<float *>(&(R_O[0]));
        if constexpr (kOStorageAccFloat32) {
          // (x,y) 0~7->{c0, c1}, (z,w)->8~15 {c2, c3}
          float *t_fptr_D_0_1 =
              reinterpret_cast<float *>(&(R_D[0][j][0])); // kWarpTileSeqLenP=1
          t_fptr_D_0_1[0] =
              __fmaf_rn(rescale_o_factor_0, t_fptr_D_0_1[0], t_fptr_O_0_1[0]);
          t_fptr_D_0_1[1] =
              __fmaf_rn(rescale_o_factor_0, t_fptr_D_0_1[1], t_fptr_O_0_1[1]);
          t_fptr_D_0_1[2] =
              __fmaf_rn(rescale_o_factor_1, t_fptr_D_0_1[2], t_fptr_O_0_1[2]);
          t_fptr_D_0_1[3] =
              __fmaf_rn(rescale_o_factor_1, t_fptr_D_0_1[3], t_fptr_O_0_1[3]);
        } else {
          half *t_hptr_D_0_1 =
              reinterpret_cast<half *>(&(R_D[0][j][0])); // kWarpTileSeqLenP=1
          t_hptr_D_0_1[0] = __float2half_rn(
              __fmaf_rn(rescale_o_factor_0, __half2float(t_hptr_D_0_1[0]),
                        t_fptr_O_0_1[0]));
          t_hptr_D_0_1[1] = __float2half_rn(
              __fmaf_rn(rescale_o_factor_0, __half2float(t_hptr_D_0_1[1]),
                        t_fptr_O_0_1[1]));
          t_hptr_D_0_1[2] = __float2half_rn(
              __fmaf_rn(rescale_o_factor_1, __half2float(t_hptr_D_0_1[2]),
                        t_fptr_O_0_1[2]));
          t_hptr_D_0_1[3] = __float2half_rn(
              __fmaf_rn(rescale_o_factor_1, __half2float(t_hptr_D_0_1[3]),
                        t_fptr_O_0_1[3]));
        } // end for tile_V_Bc
        // TODO: Write/Load scaled O -> gmem directly for very large d, e.g
        // 4k,8k,... Thus, reduce the registers usage or SRAM size for R_D.
        // Prefetch O 128bits before P@V tile (>= Ampere) and copy async O
        // 128bits r2g (>= Hopper).
        if constexpr (kStage > 1) {
          // Wait next V tile g2s ready.
          CP_ASYNC_WAIT_GROUP(kStage - 2);
          __syncthreads();
        }
      } // end for kWarpTileHeadDimV.
      // Now, we can update m, l after O has been scaled.
      // 1. First, update block row sum Exp for each lane which
      // need both m_new and m_old.
      float block_row_sum_old_0 = lane_block_row_sum_old[0][0];
      float block_row_sum_old_1 = lane_block_row_sum_old[0][1];
      // Update l = exp(m_old - m_new) * l_old + row_sum(P).
      lane_block_row_sum_old[0][0] = (__fmaf_rn(
          rescale_o_factor_0, block_row_sum_old_0, block_row_sum_new_0));
      lane_block_row_sum_old[0][1] = (__fmaf_rn(
          rescale_o_factor_1, block_row_sum_old_1, block_row_sum_new_1));
      // 2. Then, update block row max for each lane.
      lane_block_row_max_old[0][0] = block_row_max_new_0;
      lane_block_row_max_old[0][1] = block_row_max_new_1;
    } // end P@V
    __syncthreads();

  } // end loop over N
  __syncthreads();

  // Finaly, we still have to rescale O once more.
  // O_output(D) = ( 1/l_final ) * O_final (FA2 paper)
  static_assert(kWarpTileSeqLenP == 1);
  { // kWarpTileSeqLenP = 1
    float rescale_factor_0 = __frcp_rn(lane_block_row_sum_old[0][0]);
    float rescale_factor_1 = __frcp_rn(lane_block_row_sum_old[0][1]);
#pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8, 16, 32, ...
      // Scaling in registers & convert F32 -> half for O collective store.
      if constexpr (kOStorageAccFloat32) {
        float *t_fptr_D_0_1 = reinterpret_cast<float *>(&(R_D[0][j][0]));
        half *t_hptr_D_0_1 = reinterpret_cast<half *>(&(R_D[0][j][0]));
        t_hptr_D_0_1[0] = __float2half_rn(rescale_factor_0 * t_fptr_D_0_1[0]);
        t_hptr_D_0_1[1] = __float2half_rn(rescale_factor_0 * t_fptr_D_0_1[1]);
        t_hptr_D_0_1[2] = __float2half_rn(rescale_factor_1 * t_fptr_D_0_1[2]);
        t_hptr_D_0_1[3] = __float2half_rn(rescale_factor_1 * t_fptr_D_0_1[3]);
      } else {
        half *t_hptr_D_0_1 = reinterpret_cast<half *>(&(R_D[0][j][0]));
        t_hptr_D_0_1[0] =
            __float2half_rn(rescale_factor_0 * __half2float(t_hptr_D_0_1[0]));
        t_hptr_D_0_1[1] =
            __float2half_rn(rescale_factor_0 * __half2float(t_hptr_D_0_1[1]));
        t_hptr_D_0_1[2] =
            __float2half_rn(rescale_factor_1 * __half2float(t_hptr_D_0_1[2]));
        t_hptr_D_0_1[3] =
            __float2half_rn(rescale_factor_1 * __half2float(t_hptr_D_0_1[3]));
      }
    } // end for kWarpTileHeadDimV
  } // end for kWarpTileSeqLenP = 1

  // Store O(D): Write O[Br,d] from regs -> gmem, collective store
  // with reg reuse & warp shuffle.
  static_assert(kWarpTileSeqLenP == 1);
  { // kWarpTileSeqLenP = 1
#pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) { // 8
      // reuse R_Q[1][4], R_K[8][2] for collective store.
      uint32_t *t_uptr_Z_0 = reinterpret_cast<uint32_t *>(&(R_Q[0][0]));
      uint32_t *t_uptr_Z_1 = reinterpret_cast<uint32_t *>(&(R_K[0][0]));
      t_uptr_Z_0[0] = R_D[0][j][0];
      t_uptr_Z_1[0] = R_D[0][j][1];
      t_uptr_Z_0[1] = __shfl_sync((0xffffffff), R_D[0][j][0], lane_id + 1, 4);
      t_uptr_Z_0[2] = __shfl_sync((0xffffffff), R_D[0][j][0], lane_id + 2, 4);
      t_uptr_Z_0[3] = __shfl_sync((0xffffffff), R_D[0][j][0], lane_id + 3, 4);
      t_uptr_Z_1[1] = __shfl_sync((0xffffffff), R_D[0][j][1], lane_id + 1, 4);
      t_uptr_Z_1[2] = __shfl_sync((0xffffffff), R_D[0][j][1], lane_id + 2, 4);
      t_uptr_Z_1[3] = __shfl_sync((0xffffffff), R_D[0][j][1], lane_id + 3, 4);

      // st.global.v4 128 bits. [Br,d]
      if (lane_id % 4 == 0) {
        // (0/1)*32 + (0/1)*16=(0,16,32,48), + 0~7 -> 0~56
        int store_warp_regs_O_Br =
            warp_QP * (kMmaAtomM * kWarpTileSeqLenP) + 0 * kMmaAtomM;
        int store_lane_gmem_O_Br =
            O_tile_id * Br + store_warp_regs_O_Br + lane_id / 4; // 0~7
        // (0~3)*16 + (0/1)*8=(0,8,16,24,...,48,56)
        int store_warp_regs_O_d =
            warp_KV * (kMmaAtomN * kWarpTileHeadDimV) + j * kMmaAtomN;
        int store_lane_gmem_O_d = store_warp_regs_O_d; // (0~3)*16+(0/8)
        int store_gmem_O_addr_0 =
            (O_gmem_offset + (store_lane_gmem_O_Br + 0) * kHeadDim +
             store_lane_gmem_O_d);
        int store_gmem_O_addr_1 =
            (O_gmem_offset + (store_lane_gmem_O_Br + 8) * kHeadDim +
             store_lane_gmem_O_d);
        LDST128BITS(O[store_gmem_O_addr_0]) = LDST128BITS(t_uptr_Z_0[0]);
        LDST128BITS(O[store_gmem_O_addr_1]) = LDST128BITS(t_uptr_Z_1[0]);
      }
    } // end for kWarpTileHeadDimV
  } // kWarpTileSeqLenP = 1
}

template <const int kHeadDim, const int kStage>
void launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32(torch::Tensor Q,
                                                             torch::Tensor K,
                                                             torch::Tensor V,
                                                             torch::Tensor O) {
  // Now: fixed tile BrxBc=128x128 for d>= 128, 64x64 for d<128.
  constexpr int kMmaAtomM = 16;
  constexpr int kMmaAtomN = 8;
  constexpr int kMmaAtomK = 16;
  constexpr int kMmaTileSeqLenQ = (kHeadDim < 128) ? 4 : 8;
  constexpr int kMmaTileSeqLenK = 1;
  constexpr int kMmaTileSeqLenP = (kHeadDim < 128) ? 4 : 8;
  constexpr int kMmaTileHeadDimV = 1;
  constexpr int kWarpTileSeqLenQ = 1;
  constexpr int kWarpTileSeqLenK = (kHeadDim < 128) ? 8 : 16;
  constexpr int kWarpTileSeqLenP = 1;
  constexpr int kWarpTileHeadDimV =
      (kHeadDim / (kMmaAtomN * kMmaTileHeadDimV)); // (d=64)8,(d=128)16,32,....
  constexpr int Br =
      kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*4*1=64
  constexpr int Bc =
      kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; //  8*1*8=64
  constexpr int kNumThreads =
      WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK; // 32*4*1=128, num threads
  constexpr int kPadQ = 8;
  constexpr int kPadK = 8;
  constexpr int kPadV = 8;
  // 0/1, MMA Acc always be fp32, but O storage can be fp32 or half.
  // FP16 can provide precision to approximately 3-4 decimal places.
  // Thus, if the error does not exceed 1e-3, using FP16 storage is
  // sufficient for most applications.
  constexpr int kOStorageAccFloat32 = (kHeadDim < 256) ? 1 : 0;

  // static int kMaxSramPerBlock;
  // hipDeviceGetAttribute(&kMaxSramPerBlock,
  // hipDeviceAttributeMaxSharedMemoryPerBlock, 0); Calculate SRAM size needed per
  // block, Q,K,V smem size, V shared the QK smem.
  constexpr int QK_smem_size = (kStage * (Br * (kMmaAtomK + kPadQ)) +
                                kStage * (Bc * (kMmaAtomK + kPadK)));
  // Now, for V_smem_size, need fixed smem size, e.g 64*16*2/1024=2M;
  // R_D registers, s=2, d=64, 16 regs; d=128, 32 regs;
  // d=256, 64 regs; d=512, 128 regs; d=1024, 256 regs;
  constexpr int V_smem_size = (kStage * (Bc * (kMmaAtomN * 2 + kPadV)));
  // try to let V reuse all Q+K smem after Q@K^T, reduce smem usage.
  const int smem_max_size = max(QK_smem_size, V_smem_size) * sizeof(half);

  const int QKV_batch = Q.size(0);
  const int QKV_head = Q.size(1);
  const int QKV_seqlen = Q.size(2);      // QKV_seqlen
  assert(QKV_seqlen % max(Br, Bc) == 0); // multiple of max(Br, Bc)

  // TODO: How to apply block swizzle to improve L2 Cache hit rate?
  // NOTE: reorder (B,H,Tr) -> (Tr,B*H) seems can improve L2 Cache hit rate.
  // This might be because SM schedules blocks starting from the x-dimension.
  // Placing Tr at the forefront ensures that identical KV pairs are placed
  // in consecutive scheduling queues, thereby improving L2 Cache hit rates.
  // Tr(=N/Br), batch_size x num_heads
  dim3 grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head);
  dim3 block(kNumThreads); // 4/8 warps per block

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      flash_attn_mma_stages_split_q_tiling_qkv_acc_f32_kernel<
          kHeadDim), kMmaAtomM, kMmaAtomN, kMmaAtomK, kMmaTileSeqLenQ,
          kMmaTileSeqLenK, kMmaTileSeqLenP, kMmaTileHeadDimV, kWarpTileSeqLenQ,
          kWarpTileSeqLenK, kWarpTileSeqLenP, kWarpTileHeadDimV,
          kOStorageAccFloat32, kStage, kPadQ, kPadK, kPadV>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      // kMaxSramPerBlock
      98304);

  flash_attn_mma_stages_split_q_tiling_qkv_acc_f32_kernel<
      kHeadDim, kMmaAtomM, kMmaAtomN, kMmaAtomK, kMmaTileSeqLenQ,
      kMmaTileSeqLenK, kMmaTileSeqLenP, kMmaTileHeadDimV, kWarpTileSeqLenQ,
      kWarpTileSeqLenK, kWarpTileSeqLenP, kWarpTileHeadDimV,
      kOStorageAccFloat32, kStage, kPadQ, kPadK, kPadV>
      <<<grid, block, smem_max_size>>>(reinterpret_cast<half *>(Q.data_ptr()),
                                       reinterpret_cast<half *>(K.data_ptr()),
                                       reinterpret_cast<half *>(V.data_ptr()),
                                       reinterpret_cast<half *>(O.data_ptr()),
                                       QKV_seqlen, QKV_head);
}

void flash_attn_mma_stages_split_q_tiling_qkv_acc_f32(torch::Tensor Q,
                                                      torch::Tensor K,
                                                      torch::Tensor V,
                                                      torch::Tensor O,
                                                      int stages) {
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kHalf) // Q [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kHalf) // K [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kHalf) // V [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(O, torch::kHalf) // O [B,H,N,D]
  const int d = Q.size(3);                  // B, H, N, d

  if (stages > 1) {
    switch (d) {
    case 32:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<32, 2>(Q, K, V,
                                                                     O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<64, 2>(Q, K, V,
                                                                     O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<96, 2>(Q, K, V,
                                                                     O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<128, 2>(Q, K, V,
                                                                      O);
      break;
    case 256:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<256, 2>(Q, K, V,
                                                                      O);
      break;
    case 512:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<512, 2>(Q, K, V,
                                                                      O);
      break;
    case 1024:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<1024, 2>(Q, K, V,
                                                                       O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  } else {
    switch (d) {
    case 32:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<32, 1>(Q, K, V,
                                                                     O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<64, 1>(Q, K, V,
                                                                     O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<96, 1>(Q, K, V,
                                                                     O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<128, 1>(Q, K, V,
                                                                      O);
      break;
    case 256:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<256, 1>(Q, K, V,
                                                                      O);
      break;
    case 512:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<512, 1>(Q, K, V,
                                                                      O);
      break;
    case 1024:
      launch_flash_attn_mma_stages_split_q_tiling_qkv_acc_f32<1024, 1>(Q, K, V,
                                                                       O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  }
}
