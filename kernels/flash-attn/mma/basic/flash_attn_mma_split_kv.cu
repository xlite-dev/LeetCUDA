#include "hip/hip_runtime.h"
#include "utils.h"

// Write FlashAttention-2 from scratch using Tensor Cores with MMA PTX
// instruction. The input is Q,K,V, 4D tensor with shape [batch_size, num_heads,
// seq_len, head_dim]. The output is O, a 4D tensor with shape [batch_size,
// num_heads, seq_len, head_dim].

// The FlashAttention-2 algorithm is described in the following paper:
// https://arxiv.org/pdf/2307.08691

// Q,K,V,O: [batch_size, num_heads, seq_len, head_dim], [B,H,N,d]
// each block processes Q_tile with shape [Br,d] and full K,V with shape [N,d]
// Currently, we only support Br = Bc = 64.
template <const int kHeadDim,         // Headdim, 32,64,128
          const int kMmaAtomM,        // MMA Atom M, 16
          const int kMmaAtomN,        // MMA Atom N, 8
          const int kMmaAtomK,        // MMA Atom K, 16
          const int kMmaTileSeqLenQ,  // 2, more MMA(warp), M=16*2=32,
                                      // Q@K^T=[Br(M), d(K)]@[d(K),  Bc(N)]
          const int kMmaTileSeqLenK,  // 4, more MMA(warp), N=8*4= 32,
                                      // Q@K^T=[Br(M), d(K)]@[d(K),  Bc(N)]
          const int kMmaTileSeqLenP,  // 2, more MMA(warp), M=16*2=32, P@V
                                      // =[Br(M),Bc(K)]@[Bc(K), d(N) ]
          const int kMmaTileHeadDimV, // 4, more MMA(warp), N=8*4= 32, P@V
                                      // =[Br(M),Bc(K)]@[Bc(K), d(N) ]
          const int kWarpTileSeqLenQ, // 2, more values, M, Br=32*2=64, matmul M
          const int kWarpTileSeqLenK, // 2, more values, N, Bc=32*2=64, matmul N
          const int kWarpTileSeqLenP, // 2, more values, M, Br=32*2=64, matmul M
          const int kWarpTileHeadDimV, // 2, more values, N,
                                       // d=32*(1|2|3|4|...)=32|64|96|128|...
          const int kStage, const int kPad>
__global__ void __launch_bounds__(WARP_SIZE *kMmaTileSeqLenQ *kMmaTileSeqLenK)
    flash_attn_mma_stages_split_kv_kernel(half *Q, half *K, half *V, half *O,
                                          int QKV_seqlen, int QKV_head) {
  // Matmul Layout: Q[Br,d]@K^T[d,Bc] NT, P[Br,Bc]@V[Bc,d] NN.
  // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
  static_assert(kMmaAtomM == 16 && kMmaAtomN == 8 &&
                kMmaAtomK == 16);                                // m16n8k16
  static_assert(kMmaTileSeqLenQ == 2 && kMmaTileSeqLenK == 4);   // Q@K^T
  static_assert(kMmaTileSeqLenP == 2 && kMmaTileHeadDimV == 4);  // P@V
  static_assert(kWarpTileSeqLenQ == 2 && kWarpTileSeqLenK == 2); // Q@K^T
  // e.g, kWarpTileHeadDimV: 1->d 32, 2->d 64, 3->d 96, 4-> d 128, ..., etc.
  static_assert(kWarpTileSeqLenP == 2 &&
                kWarpTileHeadDimV ==
                    (kHeadDim / (kMmaAtomN * kMmaTileHeadDimV))); // P@V
  static_assert(kStage > 0 && kStage < 3);                        // 1,2
  static_assert(kPad >= 0 && kPad % 8 == 0);                      // 0,8,16
  constexpr int Br =
      kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*2*2=64
  constexpr int Bc = kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; // 8*4*2=64
  constexpr int kNumThreads =
      WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK; // 32*2*4=256, num threads
  // Now, N must be mutliples of Bc(32/64) for KV tiling across seqlen.
  const int Tc = div_ceil(QKV_seqlen, Bc); // Tc K^T_tile[d,Bc]
  const float scale = 1.0f / sqrt((float)kHeadDim);

  // grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head), (x,y,z)
  const int QKV_batch_id = blockIdx.y / QKV_head; // Batch size
  const int QKV_head_id = blockIdx.y % QKV_head;  // Head num
  const int Q_tile_id = blockIdx.x;               // Q tile_id, range [0, Tr]
  const int O_tile_id = Q_tile_id;                // O tile_id, same as Q.
  const int tid = threadIdx.x;                    // within block
  const int warp_id = tid / WARP_SIZE;            // 0~7 warp_id within block
  const int lane_id = tid % WARP_SIZE;            // 0~31
  const int warp_QP = warp_id % 2;                // 0,1
  const int warp_KV = warp_id / 2;                // 0,1,2,3
  // The layout of 8 MMA(2x4) [before] kWarpTileSeqLenQxkWarpTileSeqLenK(2x2) ->
  // 16x2,8x4=32x32: |  [32,32]  | warp_KV 0 | warp_KV 1 | warp_KV 2 | warp_KV 3
  // | | warp_QP 0 |-- MMA 0 --|-- MMA 2 --|-- MMA 4 --|-- MMA 6 --| | warp_QP 1
  // |-- MMA 1 --|-- MMA 3 --|-- MMA 5 --|-- MMA 7 --| The layout of 8 MMA(2x4)
  // [after] kWarpTileSeqLenQxkWarpTileSeqLenK(2x2) -> 32x2,32x2=64x64: |
  // [64,64]  |    warp_KV 0    |    warp_KV 1    |    warp_KV 2    |    warp_KV
  // 3    | | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4
  // --|-- MMA 6,MMA 6 --| | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|--
  // MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA
  // 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| | warp_QP 1 |-- MMA 1,MMA 1
  // --|-- MMA 3,MMA 2 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| gridDim.y =
  // head_num, gridDim.z = N/Br = Tr.
  const int Q_gmem_offset =
      ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) +
       (QKV_head_id * QKV_seqlen * kHeadDim)); // Q [seqlen,d]
  const int K_gmem_offset =
      ((QKV_batch_id * QKV_head * QKV_seqlen * kHeadDim) +
       (QKV_head_id * QKV_seqlen * kHeadDim)); // K [seqlen,d]
  const int V_gmem_offset = Q_gmem_offset;     // V [seqlen,d]
  const int O_gmem_offset = Q_gmem_offset;     // O [seqlen,d]

  // Mapping Q gmem -> tid -> smem, Q[Br,d]=[64,64 or 128], 256 threads.
  int load_smem_Q_Br = (tid / (kNumThreads / Br)); // Br 64, tid / 4, row 0~64
  int load_smem_Q_d =
      (tid % (kNumThreads / Br)) *
      (kHeadDim / (kNumThreads / Br)); // (tid % 4) * 16, 0,16,32,48
  // Mapping K gmem -> tid -> smem, K[Bc,d]=[64 or 128,64], 128 threads.
  int load_smem_K_Bc = (tid / (kNumThreads / Bc)); // Bc 64, tid / 2, row 0~64
  int load_smem_K_d =
      (tid % (kNumThreads / Bc)) *
      (kHeadDim / (kNumThreads / Bc)); // (tid % 4) * 16, 0,16,32,48
  // Mapping V gmem -> tid -> smem, V[Bc,d]=[64,64 or 128], 256 threads.
  int load_smem_V_Bc = (tid / (kNumThreads / Bc)); // Bc 64, tid / 4, row 0~64
  int load_smem_V_d =
      (tid % (kNumThreads / Bc)) *
      (kHeadDim / (kNumThreads / Bc)); // (tid % 4) * 16, 0,16,32,48
  // global Q row of current head for tile [Br,d] per block.
  int load_gmem_Q_Br = Q_tile_id * Br + load_smem_Q_Br;
  if (load_gmem_Q_Br >= QKV_seqlen)
    return;
  // KV tile gmem load index starts from 0 and increments with
  // each iteration as we loop over seqlen.
  int load_gmem_K_Bc_offset = 0;
  int load_gmem_V_Bc_offset = 0;

  // Shared memory for Q,K,V,S, we don not need additional smem for O
  // collective store which perform via registers reuse and warp shuffle.
  extern __shared__ half smem[];
  constexpr int Q_tile_size =
      Br * (kHeadDim + kPad); // 64*64=4096, ~8192 bytes=8M
  constexpr int KV_tile_size =
      Bc * (kHeadDim + kPad);                   // 64*64=4096, ~8192 bytes=8M
  constexpr int S_tile_size = Br * (Bc + kPad); // 64*64=4096, ~8192 bytes=8M
  // K multi-stages: currently, only apply multi stages for K across seq_len.
  half *Q_tile_smem = smem;                      // 8M/16M
  half *K_tile_smem = Q_tile_smem + Q_tile_size; // 8M/16M
  half *V_tile_smem = K_tile_smem + kStage * KV_tile_size;
  half *S_tile_smem = V_tile_smem + KV_tile_size; // for temp S=Q@K^T
  // stage 2, no shared KV smem, Br=Bc=64,  d=64: 8M+(8M)*2+8M   =32M,  shared
  // KV smem: 24M stage 2, no shared KV smem, Br=Bc=64, d=128:
  // 16M+(16M)*2+16M=64M,  shared KV smem: 48M stage 2, no shared KV smem,
  // Br=Bc=64, d=256: 32M+(32M)*2+32M=128M, shared KV smem: 96M stage 1, no
  // shared KV smem, Br=Bc=64, d=256: 32M+(32M)*1+32M=96M,  shared KV smem: 64M

  uint32_t smem_Q_base_ptr = __cvta_generic_to_shared(Q_tile_smem);
  uint32_t smem_K_base_ptr = __cvta_generic_to_shared(K_tile_smem);
  uint32_t smem_V_base_ptr = __cvta_generic_to_shared(V_tile_smem);
  uint32_t smem_S_base_ptr = __cvta_generic_to_shared(S_tile_smem);

  // Registers/SMEM for thread block
  // block m_old, l_old, store in lane, use float to
  // keep precision.
  float lane_block_row_max_old[kWarpTileSeqLenQ][2];
  float lane_block_row_sum_old[kWarpTileSeqLenQ][2];
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_max_old, -INFINITY);
  fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_block_row_sum_old, 0.0f);
  // m[Br], l[Br], for the output of P[Br,Bc]=Q[Br,d]@K^T[d,Bc],
  // 64x(4)x4=1024 bytes, 1M+1M=2M. For now, I have choose to
  // add 1 to reduce bank conflicts, may boost 2~3 TFLOPS.
  __shared__ float block_row_max_new_smem[Br][kMmaTileSeqLenK + 1];
  __shared__ float block_row_sum_new_smem[Br][kMmaTileSeqLenK + 1];

  // Registers for S=Q@K^T/O=P@V
  // registers for QKV, S=Q[Br,d]@K[Bc,d]=[Br,Bc]
  // and O=P[Br,Bc]@V[Bc,d]=[Br,d].
  uint32_t R_Q[kWarpTileSeqLenQ][4];
  uint32_t R_K[kWarpTileSeqLenK][2];
  uint32_t R_V[kWarpTileHeadDimV][2];
  // registers for current tile_K_seqlen within, [64,64] = S_tile[Br,Bc]
  // = Q_tile[Br,d] * K[Bc,d], each thread hold 2x32 bits regs.
  uint32_t R_S[kWarpTileSeqLenQ][kWarpTileSeqLenK][2]; // [2][2][2]
  // registers for tile_K_seqlen O=PV[Br,d]=P@V, [2][2/4][2], 8 or 16 regs.
  // TODO: may reuse R_D as R_O? kWarpTileSeqLenP=kWarpTileSeqLenQ.
  uint32_t R_O[kWarpTileSeqLenP][kWarpTileHeadDimV][2]; // [2][2/4][2]
  // registers final Output [D]=final rescale(R_O), [2][2/4][2], 8 or 16 regs.
  uint32_t R_D[kWarpTileSeqLenP][kWarpTileHeadDimV][2]; // [2][2/4][2]
  fill_3D_regs<uint32_t, kWarpTileSeqLenQ, kWarpTileSeqLenK, 2>(R_S, 0);
  fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_D, 0);
  fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_O, 0);

  // load Q from gmem -> smem, only load once.
  {
    int load_gmem_Q_d = load_smem_Q_d;
    int load_gmem_Q_addr =
        (Q_gmem_offset + load_gmem_Q_Br * kHeadDim + load_gmem_Q_d);
    uint32_t load_smem_Q_ptr =
        (smem_Q_base_ptr +
         (load_smem_Q_Br * (kHeadDim + kPad) + load_smem_Q_d) * sizeof(half));
#pragma unroll
    for (int i = 0; i < (kHeadDim / (kNumThreads / Br)); i += 8) {
      CP_ASYNC_CG(load_smem_Q_ptr + i * 2, &Q[load_gmem_Q_addr + i], 16);
    }
    CP_ASYNC_COMMIT_GROUP();
  }

  // load K from gmem -> smem, (kStage - 1) K^T tiles, [d,Bc]
  if constexpr (kStage > 1) {
#pragma unroll
    for (int stage = 0; stage < (kStage - 1); ++stage) {
      // update the offset of n according to stages
      load_gmem_K_Bc_offset = stage * Bc; // e.g (0~3)*64=(0,64,128,192,...)
      int load_gmem_K_Bc = load_gmem_K_Bc_offset + load_smem_K_Bc; // < seqlen
      int load_gmem_K_d = load_smem_K_d; // K [Bc,d] from [seqlen,d]
      int load_gmem_K_addr =
          (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
      uint32_t load_smem_K_ptr =
          (smem_K_base_ptr +
           (stage * KV_tile_size + load_smem_K_Bc * (kHeadDim + kPad) +
            load_smem_K_d) *
               sizeof(half));
#pragma unroll
      for (int i = 0; i < (kHeadDim / (kNumThreads / Bc)); i += 8) {
        CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
      }
      CP_ASYNC_COMMIT_GROUP();
    }
  }

  // wait Q and at least (kStage - 1) for K ready.
  if constexpr (kStage > 1) {
    CP_ASYNC_WAIT_GROUP(kStage - 2); // s2->0, s3->1, s4->2
    __syncthreads();
  }

// <loop over K seqlen>: for K^T[d,seqlen] with K^T_tile[d,Bc]
// tile_K_seqlen: compute S_tile[Br,Bc] = Q@K^T = Q_tile[Br,d] * K^T[d,Bc]
#pragma unroll 1
  for (int tile_K_seqlen = 0; tile_K_seqlen < Tc; ++tile_K_seqlen) {
    // TODO: process last tile_K_seqlen ? pad to multiple of 8.
    // s2 tn 0->0, 1->1, 2->0; s3 tn 0->0, 1->1, 2->2, 3->0;
    int smem_sel = (tile_K_seqlen) % kStage;
    // s2 tn 0->1, 1->0, 2->1; s3 tn 0->2, 1->0, 2->1, 3->2;
    int smem_sel_next = (tile_K_seqlen + (kStage - 1)) % kStage;

    // multi stages pipeling gmem -> smem
    // NOTE: kStage must be > 1 for pipeling. For s1, smem_sel
    // and smem_sel_next will always equal 0, thus, we can not
    // prefetch KV from gmem to smem before tile_K_seqlen MMA done.

    if constexpr (kStage > 1) {
      // First, prefetch curr V tile_K_seqlen [Bc,d] (no stages)
      {
        load_gmem_V_Bc_offset =
            tile_K_seqlen * Bc; // e.g (0~3)*64=(0,64,128,192,...)
        int load_gmem_V_Bc = load_gmem_V_Bc_offset + load_smem_V_Bc;
        int load_gmem_V_d = load_smem_V_d;
        int load_gmem_V_addr =
            (V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
        uint32_t load_smem_V_ptr =
            (smem_V_base_ptr +
             (load_smem_V_Bc * (kHeadDim + kPad) + load_smem_V_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kHeadDim / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      }

      // Then, prefetch next stage K (tile_K_seqlen + 1) [d,Bc]
      if ((tile_K_seqlen + 1) < Tc) {
        load_gmem_K_Bc_offset =
            (tile_K_seqlen + 1) * Bc; // e.g (0~3)*64=(0,64,128,192,...)
        int load_gmem_K_Bc = load_gmem_K_Bc_offset + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d = load_smem_K_d; // K [Bc,d] from [seqlen,d]
        int load_gmem_K_addr =
            (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr =
            (smem_K_base_ptr +
             (smem_sel_next * KV_tile_size +
              load_smem_K_Bc * (kHeadDim + kPad) + load_smem_K_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kHeadDim / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      }
    } else {
      // If no stages, kStage = 1, we have to load current K tile
      // from gmem to smem and have to wait it ready for Q@K^T MMA.

      // First, prefetch curr K tile_K_seqlen [d,Bc] (no stages)
      {
        load_gmem_K_Bc_offset =
            tile_K_seqlen * Bc; // e.g (0~3)*64=(0,64,128,192,...)
        int load_gmem_K_Bc = load_gmem_K_Bc_offset + load_smem_K_Bc; // < seqlen
        int load_gmem_K_d = load_smem_K_d; // K [Bc,d] from [seqlen,d]
        int load_gmem_K_addr =
            (K_gmem_offset + load_gmem_K_Bc * kHeadDim + load_gmem_K_d);
        uint32_t load_smem_K_ptr =
            (smem_K_base_ptr +
             (smem_sel * KV_tile_size + load_smem_K_Bc * (kHeadDim + kPad) +
              load_smem_K_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kHeadDim / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_K_ptr + i * 2, &K[load_gmem_K_addr + i], 16);
        }

        CP_ASYNC_COMMIT_GROUP();
      }

      // Then, prefetch curr K tile_K_seqlen [d,Bc] (no stages)
      {
        load_gmem_V_Bc_offset =
            tile_K_seqlen * Bc; // e.g (0~3)*64=(0,64,128,192,...)
        int load_gmem_V_Bc = load_gmem_V_Bc_offset + load_smem_V_Bc;
        int load_gmem_V_d = load_smem_V_d;
        int load_gmem_V_addr =
            (V_gmem_offset + load_gmem_V_Bc * kHeadDim + load_gmem_V_d);
        uint32_t load_smem_V_ptr =
            (smem_V_base_ptr +
             (load_smem_V_Bc * (kHeadDim + kPad) + load_smem_V_d) *
                 sizeof(half));
#pragma unroll
        for (int i = 0; i < (kHeadDim / (kNumThreads / Bc)); i += 8) {
          CP_ASYNC_CG(load_smem_V_ptr + i * 2, &V[load_gmem_V_addr + i], 16);
        }
        CP_ASYNC_COMMIT_GROUP();
      }

      // Wait curr Q and K tile ready and let curr V tile copy async.
      CP_ASYNC_WAIT_GROUP(1);
      __syncthreads();
    }

    // <loop over K d>: tile_K_d, kMmaAtomK = 16, K_tile_d[kMmaAtomK,Bc]
    // Matmul with NT layout, Q row major, K^T col major.
    // NOTE: K[Bc,d] with row major means K^T[d,Bc] in col major.
    // S_tile[Br,Bc]=Q_tile[Br,d]@K[Bc,d]
    fill_3D_regs<uint32_t, kWarpTileSeqLenQ, kWarpTileSeqLenK, 2>(R_S, 0);
#pragma unroll
    for (int tile_K_d = 0; tile_K_d < (kHeadDim / kMmaAtomK); ++tile_K_d) {
// smem -> reg, load m16k16 smem Q, offset d according tile_K_d.
// ldmatrix.x4 for Q_tile_smem.
#pragma unroll
      for (int i = 0; i < kWarpTileSeqLenQ; ++i) { // Q[Br,d]=[M,K]
        int warp_smem_Q_Br =
            warp_QP * (kMmaAtomM * kWarpTileSeqLenQ) + i * kMmaAtomM;
        int lane_smem_Q_Br = warp_smem_Q_Br + lane_id % 16;            // 0~15
        int lane_smem_Q_d = tile_K_d * kMmaAtomK + (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_Q_ptr =
            (smem_Q_base_ptr +
             (lane_smem_Q_Br * (kHeadDim + kPad) + lane_smem_Q_d) *
                 sizeof(half));
        LDMATRIX_X4(R_Q[i][0], R_Q[i][1], R_Q[i][2], R_Q[i][3],
                    lane_smem_Q_ptr); // now, R_Q
      }

// smem -> reg, load k16n8 from smem K, offset d according tile_K_d.
// ldmatrix.x2 for K_tile_smem, [Bc,kMmaAtomK] from [Bc,d]=[K,N]
#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // load k16n8 via ldmatrix.x2 from K_tile_smem[Bc,d].
        // K[Bc,d] with row major means K^T[d,Bc] in col major.
        int warp_smem_K_Bc =
            warp_KV * (kMmaAtomN * kWarpTileSeqLenK) + j * kMmaAtomN;
        int lane_smem_K_Bc = warp_smem_K_Bc + lane_id % 8; // 0~7
        int lane_smem_K_d =
            tile_K_d * kMmaAtomK + ((lane_id / 8) % 2) * 8; // 0,8
        uint32_t lane_smem_K_ptr =
            (smem_K_base_ptr +
             (smem_sel * KV_tile_size + lane_smem_K_Bc * (kHeadDim + kPad) +
              lane_smem_K_d) *
                 sizeof(half));
        LDMATRIX_X2(R_K[j][0], R_K[j][1], lane_smem_K_ptr); // R_K
      } // end for kWarpTileSeqLenK

// MMA compute
#pragma unroll
      for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
#pragma unroll
        for (int j = 0; j < kWarpTileSeqLenK; ++j) {
          HMMA16816(R_S[i][j][0], R_S[i][j][1], R_Q[i][0], R_Q[i][1], R_Q[i][2],
                    R_Q[i][3], R_K[j][0], R_K[j][1], R_S[i][j][0],
                    R_S[i][j][1]);
        }
      }
    } // end loop over d, S=Q@K^T
    __syncthreads();

    // The layout of 8 MMA m16n8k16 (2x4)  [after] kWarpTileQPxkWarpTileKV(2x2)
    // -> 32x2,32x2=64x64: |  [64,64]  |    warp_KV 0    |    warp_KV 1    |
    // warp_KV 2    |    warp_KV 3    | | warp_QP 0 |-- MMA 0,MMA 0 --|-- MMA
    // 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| row max | warp_QP 0 |--
    // MMA 0,MMA 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| row
    // max | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 3 --|-- MMA 5,MMA 5 --|--
    // MMA 7,MMA 7 --| row max | warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 3
    // --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| row max

    // Online safe softmax, warp/block reduce max/sum, row wise
    // warp 0/2/4/6, [0][2] row 0~15,  col 0/8/16/32, max, [1][2] row 16~31, col
    // 0/8/16/32, max warp 1/3/5/7, [0][2] row 32~47, col 0/8/16/32, max, [1][2]
    // row 48~61, col 0/8/16/32, max
    float lane_row_max_new[kWarpTileSeqLenQ][2];
    float lane_row_sum_new[kWarpTileSeqLenQ][2];
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_max_new, -INFINITY);
    fill_2D_regs<float, kWarpTileSeqLenQ, 2>(lane_row_sum_new, 0.0f);

// Row max for [Br,Bc] tile, Thread -> Warp -> Block.
#pragma unroll
    for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
// Thread level reduce max across kWarpTileSeqLenK dim, namely Bc.
#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        // reference:
        // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
        // #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
        // The layout of the fragments held by different threads for C.
        // (m16n8k16) Row\Col  0    1    2    3    4    5    6    7 0        T0:
        // {c0, c1}  T1: {c0, c1}  T2: {c0, c1}  T3: {c0, c1} 1        T4: {c0,
        // c1}  T5: {c0, c1}  T6: {c0, c1}  T7: {c0, c1} 2        ...
        // ...
        // 7        T28: {c0, c1}  T29: {c0, c1}  T30: {c0, c1}  T31: {c0, c1}
        // 8        T0: {c2, c3}   T1: {c2, c3}   T2: {c2, c3}   T3: {c2, c3}
        // 9        T4: {c2, c3}   T5: {c2, c3}   T6: {c2, c3}   T7: {c2, c3}
        // 10       ...
        // ...
        // 15       T28: {c2, c3}  T29: {c2, c3}  T30: {c2, c3}  T31: {c2, c3}
        float2 t_reg_S_0 = __half22float2(HALF2(R_S[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_S_1 = __half22float2(HALF2(R_S[i][j][1])); // 8~15 {c2, c3}
        // This should be the row max after S = (Q @ K^T) / sqrt(d)
        float tmp_max_0 = max(t_reg_S_0.x, t_reg_S_0.y) * scale;
        float tmp_max_1 = max(t_reg_S_1.x, t_reg_S_1.y) * scale;
        lane_row_max_new[i][0] = max(lane_row_max_new[i][0], tmp_max_0);
        lane_row_max_new[i][1] = max(lane_row_max_new[i][1], tmp_max_1);
      } // end for kWarpTileSeqLenK

      // Warp level reduce max, warp_size = 4
      // Each thread contains the maximum of 2 rows of Br,
      // and only the values of T0, T4, ..., T28 are used.
      // Br, row_id = warp_QP<0|1> * 32 + i<0|1> * 16 + 0 * 8 + (lane / 4) <0~7>
      lane_row_max_new[i][0] =
          warp_reduce_max<float, 4>(lane_row_max_new[i][0]);
      // Br, row_id = warp_QP<0|1> * 32 + i<0|1> * 16 + 1 * 8 + (lane / 4)
      // <8~15>
      lane_row_max_new[i][1] =
          warp_reduce_max<float, 4>(lane_row_max_new[i][1]);

      if (lane_id % 4 == 0) {   // only need T0,T4,...,T28
        block_row_max_new_smem[ // Br, row_id, 0~7,  16~23, 32~39, 48~55
            warp_QP * 32 + i * 16 + 0 * 8 + (lane_id / 4)][warp_KV] =
            lane_row_max_new[i][0];
        block_row_max_new_smem[ // Br, row_id, 8~15, 24~31, 40~47, 56~63
            warp_QP * 32 + i * 16 + 1 * 8 + (lane_id / 4)][warp_KV] =
            lane_row_max_new[i][1];
      }
    } // end for kWarpTileSeqLenQ
    __syncthreads();

    // Block level reduce max, row wise, 64x4=256. Warp reduce operation
    // is faster than atomaicMaxFloat in my tests.
    float wrp_row_max_new =
        (block_row_max_new_smem[tid / kMmaTileSeqLenK]
                               [tid % kMmaTileSeqLenK]); // [0~63][0~4]
    float blk_row_max_new = warp_reduce_max<float, 4>(wrp_row_max_new);
    block_row_max_new_smem[tid / kMmaTileSeqLenK][tid % kMmaTileSeqLenK] =
        (blk_row_max_new);
    __syncthreads();

// Exp sum and mul scale_factor for [Br,Bc] tile, Thread -> Warp -> Block.
#pragma unroll
    for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
      // Use latest global row max without update.
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55;
      float block_row_max_new_0 =
          block_row_max_new_smem[warp_QP * 32 + i * 16 + 0 * 8 + (lane_id / 4)]
                                [0];
      // Br 1, row_id, 8~15, 24~31, 40~47, 56~63;
      float block_row_max_new_1 =
          block_row_max_new_smem[warp_QP * 32 + i * 16 + 1 * 8 + (lane_id / 4)]
                                [0];

      float block_row_max_old_0 = lane_block_row_max_old[i][0];
      float block_row_max_old_1 = lane_block_row_max_old[i][1];
      // Apply m_new = max(m_old, m_new) here.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);

#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        float2 t_reg_S_0 = __half22float2(HALF2(R_S[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_S_1 = __half22float2(HALF2(R_S[i][j][1])); // 8~15 {c2, c3}
        // P = Exp(S - m_new), fmaf(x, y, z) = x * y + z;
        t_reg_S_0.x =
            __expf(__fmaf_rn(t_reg_S_0.x, scale, -block_row_max_new_0));
        t_reg_S_0.y =
            __expf(__fmaf_rn(t_reg_S_0.y, scale, -block_row_max_new_0));
        t_reg_S_1.x =
            __expf(__fmaf_rn(t_reg_S_1.x, scale, -block_row_max_new_1));
        t_reg_S_1.y =
            __expf(__fmaf_rn(t_reg_S_1.y, scale, -block_row_max_new_1));
        lane_row_sum_new[i][0] += (t_reg_S_0.x + t_reg_S_0.y);
        lane_row_sum_new[i][1] += (t_reg_S_1.x + t_reg_S_1.y);
        // Update R_S for P[Br,Bc] = Exp(S-m), point wise.
        HALF2(R_S[i][j][0]) = __float22half2_rn(t_reg_S_0);
        HALF2(R_S[i][j][1]) = __float22half2_rn(t_reg_S_1);
      } // end for kWarpTileSeqLenK

      // Warp level reduce sum, warp_size = 4
      lane_row_sum_new[i][0] =
          warp_reduce_sum<float, 4>(lane_row_sum_new[i][0]);
      lane_row_sum_new[i][1] =
          warp_reduce_sum<float, 4>(lane_row_sum_new[i][1]);

      if (lane_id % 4 == 0) {   // only need T0,T4,...,T28
        block_row_sum_new_smem[ // Br, row_id, 0~7,  16~23, 32~39, 48~55
            warp_QP * 32 + i * 16 + 0 * 8 + (lane_id / 4)][warp_KV] =
            lane_row_sum_new[i][0];
        block_row_sum_new_smem[ // Br, row_id, 8~15, 24~31, 40~47, 56~63
            warp_QP * 32 + i * 16 + 1 * 8 + (lane_id / 4)][warp_KV] =
            lane_row_sum_new[i][1];
      }
    } // end for kWarpTileSeqLenQ
    __syncthreads();

    // Block level reduce sum, row wise, 64x4=256. Warp reduce operation
    // is faster than atomaicAdd float in my tests.
    float wrp_row_sum_new =
        (block_row_sum_new_smem[tid / kMmaTileSeqLenK]
                               [tid % kMmaTileSeqLenK]); // [0~63][0~4]
    float blk_row_sum_new = warp_reduce_sum<float, 4>(wrp_row_sum_new);
    block_row_sum_new_smem[tid / kMmaTileSeqLenK][tid % kMmaTileSeqLenK] =
        (blk_row_sum_new);
    __syncthreads();

// Retile warp for [Br,d], kWarpTileHeadDimV: 1=32/(4*8); 2=64/(4*8);
// 4=128/(4*8). Compute P[Br,Bc] @ V[Bc,d] = [Br,d] = [64, 64/128], partion
// Attention.

// If headdim=<32>, then, kWarpTileHeadDimV = 1, the layout of 8 MMA m16n8k16
// (2x4) after kWarpTileSeqLenPxkWarpTileHeadDimV(2x1) tiling to
// (32x2,32x1)=(64x32), will look like: |  [64,32]  | warp_KV 0 | warp_KV 1 |
// warp_KV 2 | warp_KV 3 | | warp_QP 0 |-- MMA 0 --|-- MMA 2 --|-- MMA 4 --|--
// MMA 6 --| | warp_QP 0 |-- MMA 0 --|-- MMA 2 --|-- MMA 4 --|-- MMA 6 --| |
// warp_QP 1 |-- MMA 1 --|-- MMA 3 --|-- MMA 5 --|-- MMA 7 --| | warp_QP 1 |--
// MMA 1 --|-- MMA 3 --|-- MMA 5 --|-- MMA 7 --|

// If headdim=<64>, then, kWarpTileHeadDimV = 2, the layout of 8 MMA m16n8k16
// (2x4) after kWarpTileSeqLenPxkWarpTileHeadDimV(2x2) tiling to
// (32x2,32x2)=(64x64), will look like: |  [64,64]  |    warp_KV 0    | warp_KV
// 1    |    warp_KV 2    |    warp_KV 3    | | warp_QP 0 |-- MMA 0,MMA 0 --|--
// MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| | warp_QP 0 |-- MMA 0,MMA
// 0 --|-- MMA 2,MMA 2 --|-- MMA 4,MMA 4 --|-- MMA 6,MMA 6 --| | warp_QP 1 |--
// MMA 1,MMA 1 --|-- MMA 3,MMA 3 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA 7 --| |
// warp_QP 1 |-- MMA 1,MMA 1 --|-- MMA 3,MMA 3 --|-- MMA 5,MMA 5 --|-- MMA 7,MMA
// 7 --|

// If headdim=<128>, then, kWarpTileHeadDimV = 4, the layout of 8 MMA m16n8k16
// (2x4) after kWarpTileSeqLenPxkWarpTileHeadDimV(2x2x2) tiling to
// (32x2,32x2x2)=(64x64x2), will look like: | [64,64x2] |         warp_KV 0 |
// warp_KV 1         |           warp_KV 2         |          warp_KV 3 | |
// warp_QP 0 |-- MMA 0,MMA 0,MMA 0,MMA 0 --|-- MMA 2,MMA 2,MMA 2,MMA 2 --|-- MMA
// 4,MMA 4,MMA 4,MMA 4 --|-- MMA 6,MMA 6,MMA 6,MMA 6 --| | warp_QP 0 |-- MMA
// 0,MMA 0,MMA 0,MMA 0 --|-- MMA 2,MMA 2,MMA 2,MMA 2 --|-- MMA 4,MMA 4,MMA 4,MMA
// 4 --|-- MMA 6,MMA 6,MMA 6,MMA 6 --| | warp_QP 1 |-- MMA 1,MMA 1,MMA 1,MMA 1
// --|-- MMA 3,MMA 3,MMA 3,MMA 3 --|-- MMA 5,MMA 5,MMA 5,MMA 5 --|-- MMA 7,MMA
// 7,MMA 7,MMA 7 --| | warp_QP 1 |-- MMA 1,MMA 1,MMA 1,MMA 1 --|-- MMA 3,MMA
// 3,MMA 3,MMA 3 --|-- MMA 5,MMA 5,MMA 5,MMA 5 --|-- MMA 7,MMA 7,MMA 7,MMA 7 --|

// Write R_P(R_S) to P_smem [Br,Bc]
// store S[Br,Bc] of [seqlen,seqlen] [64,64]
#pragma unroll
    for (int i = 0; i < kWarpTileSeqLenQ; ++i) {
#pragma unroll
      for (int j = 0; j < kWarpTileSeqLenK; ++j) {
        R_Q[0][0] = R_S[i][j][0];
        R_Q[1][0] = R_S[i][j][1]; // warp_size 4
        R_Q[0][1] = __shfl_sync((0xffffffff), R_S[i][j][0], lane_id + 1, 4);
        R_Q[0][2] = __shfl_sync((0xffffffff), R_S[i][j][0], lane_id + 2, 4);
        R_Q[0][3] = __shfl_sync((0xffffffff), R_S[i][j][0], lane_id + 3, 4);
        R_Q[1][1] = __shfl_sync((0xffffffff), R_S[i][j][1], lane_id + 1, 4);
        R_Q[1][2] = __shfl_sync((0xffffffff), R_S[i][j][1], lane_id + 2, 4);
        R_Q[1][3] = __shfl_sync((0xffffffff), R_S[i][j][1], lane_id + 3, 4);

        // st.global.v4 128 bits.
        if (lane_id % 4 == 0) {
          // (0/1)*32 + (0/1)*16=(0,16,32,48), + 0~7 -> 0~56
          int store_warp_regs_S_Br =
              warp_QP * (kMmaAtomM * kWarpTileSeqLenQ) + i * kMmaAtomM;
          int store_lane_smem_S_Br = store_warp_regs_S_Br + lane_id / 4; // 0~7
          // (0~3)*16 + (0/1)*8=(0,8,16,24,...,48,56)
          int store_warp_regs_S_Bc =
              warp_KV * (kMmaAtomN * kWarpTileSeqLenK) + j * kMmaAtomN;
          int store_lane_smem_S_Bc = store_warp_regs_S_Bc; // (0~3)*16+(0/8)
          int store_smem_S_addr_0 =
              ((store_lane_smem_S_Br + 0) * (Bc + kPad) + store_lane_smem_S_Bc);
          int store_smem_S_addr_1 =
              ((store_lane_smem_S_Br + 8) * (Bc + kPad) + store_lane_smem_S_Bc);
          LDST128BITS(S_tile_smem[store_smem_S_addr_0]) =
              LDST128BITS(R_Q[0][0]);
          LDST128BITS(S_tile_smem[store_smem_S_addr_1]) =
              LDST128BITS(R_Q[1][0]);
        }
      } // end for kWarpTileHeadDimV
    } // end for kWarpTileSeqLenQ
    __syncthreads();

    // Compute P[Br,Bc] @ V[Bc,d] = [Br,d] = [64, 64/128], partion Attention.
    // Here, we have to wait V ready before compute O = P @ V
    if constexpr (kStage > 1) {
      // NOTE: For kStage > 1, we have send V mem issues before K
      if ((tile_K_seqlen + 1) < Tc) {
        CP_ASYNC_WAIT_GROUP(1);
      } else {
        CP_ASYNC_WAIT_GROUP(0);
      }
    } else {
      CP_ASYNC_WAIT_GROUP(0);
    }
    __syncthreads();

    // <loop over V Bc>: P[Br,Bc]@V[Bc,d]=[Br,d]=[64,64/128], partion Attention.
    // Matmul with NN layout: P[Br,Bc] row major, V[Bc,d] row major.
    // Make sure to clear the states in R_O before MMA for P@V for each step.
    fill_3D_regs<uint32_t, kWarpTileSeqLenP, kWarpTileHeadDimV, 2>(R_O, 0);
#pragma unroll
    for (int tile_V_Bc = 0; tile_V_Bc < (Bc / kMmaAtomK); ++tile_V_Bc) {
// smem -> reg, load m16k16 smem Q, offset d according tile_K_d.
// ldmatrix.x4 from S_tile_smem.
#pragma unroll
      for (int i = 0; i < kWarpTileSeqLenP; ++i) { // S[Br,Bc]=[M,K]
        int warp_smem_S_Br =
            warp_QP * (kMmaAtomM * kWarpTileSeqLenP) + i * kMmaAtomM;
        int lane_smem_S_Br = warp_smem_S_Br + lane_id % 16;              // 0~15
        int lane_smem_S_Bc = tile_V_Bc * kMmaAtomK + (lane_id / 16) * 8; // 0,8
        uint32_t lane_smem_S_ptr =
            (smem_S_base_ptr +
             (lane_smem_S_Br * (Bc + kPad) + lane_smem_S_Bc) * sizeof(half));
        LDMATRIX_X4(R_Q[i][0], R_Q[i][1], R_Q[i][2], R_Q[i][3],
                    lane_smem_S_ptr); // now, R_P
      }

// Load k16n8 V from smem -> regs, R_KV, ldmatrix.x2.trans.
#pragma unroll
      for (int j = 0; j < kWarpTileHeadDimV; ++j) {
        int warp_smem_V_d = warp_KV * (kMmaAtomN * kWarpTileHeadDimV) +
                            j * kMmaAtomN; // d, matmaul N
        int lane_smem_V_Bc =
            tile_V_Bc * kMmaAtomK + lane_id % 16; // 0~15; Bc, matmul K
        int lane_smem_V_d = warp_smem_V_d;        // 0
        uint32_t lane_smem_V_ptr =
            (smem_V_base_ptr +
             (lane_smem_V_Bc * (kHeadDim + kPad) + lane_smem_V_d) *
                 sizeof(half));
        LDMATRIX_X2_T(R_V[j][0], R_V[j][1], lane_smem_V_ptr); // R_V
      }

// NOTE: Values for P[Br,Bc] already in R_S registers, can we use these
// registers for P(A) matrix directly ? How to do that ?
// according to the A matrix layout for MMA m16n8k16 instruction.
// reference: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html
// #matrix-fragments-for-mma-m16n8k16-with-floating-point-type
// The layout of the fragments held by different threads for A matrix with .f16.
// R\C  0    1    2    3    4    5    6    7    8    9   10   11   12   13   14
// 15 0    T0: {a0, a1}  T1: {a0, a1}  T2: {a0, a1}  T3: {a0, a1}  T0: {a4, a5}
// T1: {a4, a5}  T2: {a4, a5}  T3: {a4, a5} 1    T4: {a0, a1}  T5: {a0, a1}  T6:
// {a0, a1}  T7: {a0, a1}  T4: {a4, a5}  T5: {a4, a5}  T6: {a4, a5}  T7: {a4,
// a5} 2    (dashed arrow pointing right)
// ...
// 7    T28: {a0, a1}  T29: {a0, a1}  T30: {a0, a1}  T31: {a0, a1}  T28: {a4,
// a5}  T29: {a4, a5}  T30: {a4, a5}  T31: {a4, a5} 8    T0: {a2, a3}   T1: {a2,
// a3}   T2: {a2, a3}   T3: {a2, a3}   T0: {a6, a7}   T1: {a6, a7}   T2: {a6,
// a7}   T3: {a6, a7} 9    T4: {a2, a3}   T5: {a2, a3}   T6: {a2, a3}   T7: {a2,
// a3}   T4: {a6, a7}   T5: {a6, a7}   T6: {a6, a7}   T7: {a6, a7} 10   (dashed
// arrow pointing right)
// ...
// 15   T28: {a2, a3}  T29: {a2, a3}  T30: {a2, a3}  T31: {a2, a3}  T28: {a6,
// a7}  T29: {a6, a7}  T30: {a6, a7}  T31: {a6, a7}
#pragma unroll
      for (int i = 0; i < kWarpTileSeqLenP; ++i) { // kWarpTileSeqLenQ=2
#pragma unroll
        for (int j = 0; j < kWarpTileHeadDimV;
             ++j) { // kWarpTileHeadDimV=1,2,3,4,...
          // tile_V_Bc = 0, all curr MMAs(0~7) need P[:,  0:16]; but only stored
          // in MMA 0, MMA 1, S_P tile_V_Bc = 1, all curr MMAs(0~7) need P[:,
          // 16:32]; but only stored in MMA 2, MMA 3, S_P tile_V_Bc = 2, all
          // curr MMAs(0~7) need P[:, 32:48]; but only stored in MMA 4, MMA 5,
          // S_P tile_V_Bc = 3, all curr MMAs(0~7) need P[:, 48:64]; but only
          // stored in MMA 6, MMA 7, S_P We have to comm across warps to get
          // right values for MMA inner loop, namely, we have to use shared
          // memory to collect values from other warps. Thus, S_P can not use as
          // A matrix in MMA in split_kv mode.
          HMMA16816(R_O[i][j][0], R_O[i][j][1], R_Q[i][0], R_Q[i][1], R_Q[i][2],
                    R_Q[i][3], R_V[j][0], R_V[j][1], R_O[i][j][0],
                    R_O[i][j][1]);
        }
      }
    } // end for V Bc.
    __syncthreads();

// Rescale O -> Update row sum Exp -> then, Update row max.
#pragma unroll
    for (int i = 0; i < kWarpTileSeqLenP;
         ++i) { // kWarpTileSeqLenQ=kWarpTileSeqLenP
      // m = max(m_old, m_new), l = exp(m_old - m) * l_old + l_new (FA2 paper)
      // Br 0, row_id, 0~7,  16~23, 32~39, 48~55; Br 1, row_id, 8~15, 24~31,
      // 40~47, 56~63
      float block_row_max_new_0 =
          block_row_max_new_smem[warp_QP * 32 + i * 16 + 0 * 8 + (lane_id / 4)]
                                [0];
      float block_row_max_new_1 =
          block_row_max_new_smem[warp_QP * 32 + i * 16 + 1 * 8 + (lane_id / 4)]
                                [0];
      float block_row_sum_new_0 =
          block_row_sum_new_smem[warp_QP * 32 + i * 16 + 0 * 8 + (lane_id / 4)]
                                [0];
      float block_row_sum_new_1 =
          block_row_sum_new_smem[warp_QP * 32 + i * 16 + 1 * 8 + (lane_id / 4)]
                                [0];

      float block_row_max_old_0 = lane_block_row_max_old[i][0];
      float block_row_max_old_1 = lane_block_row_max_old[i][1];
      // NOTE: max(-inf, val) = val.
      block_row_max_new_0 = max(block_row_max_old_0, block_row_max_new_0);
      block_row_max_new_1 = max(block_row_max_old_1, block_row_max_new_1);
      // Avoid inf value while using m_old for rescaling O.
      block_row_max_old_0 =
          (tile_K_seqlen > 0 ? block_row_max_old_0 : block_row_max_new_0);
      block_row_max_old_1 =
          (tile_K_seqlen > 0 ? block_row_max_old_1 : block_row_max_new_1);

      // rescale factor for O and l, exp(m_old - m)
      float rescale_o_factor_0 =
          __expf(block_row_max_old_0 - block_row_max_new_0);
      float rescale_o_factor_1 =
          __expf(block_row_max_old_1 - block_row_max_new_1);
// 0. Rescale O: Online rescaling O each tile_K_seqlen step, need m_new, m_old.
// m = max(m_old, m_new), O_new[Br,d] = exp(m_old - m) * O_old + P@V
#pragma unroll
      for (int j = 0; j < kWarpTileHeadDimV; ++j) {
        float2 t_reg_O_0 = __half22float2(HALF2(R_O[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_O_1 = __half22float2(HALF2(R_O[i][j][1])); // 8~15 {c2, c3}
        float2 t_reg_D_0 = __half22float2(HALF2(R_D[i][j][0])); // 0~7  {c0, c1}
        float2 t_reg_D_1 = __half22float2(HALF2(R_D[i][j][1])); // 8~15 {c2, c3}
        // Note that the formula in the FA2 paper is incorrect; here,
        // the inverse of the exp function should not be taken, as it
        // would result in an error during rescaling, namely, you have
        // use exp(m_old - m_new), not 1/(m_old - m_new).
        // O_new[Br,d] = exp(m_old - m_new) * O_old + P@V
        t_reg_D_0.x = __fmaf_rn(rescale_o_factor_0, t_reg_D_0.x, t_reg_O_0.x);
        t_reg_D_0.y = __fmaf_rn(rescale_o_factor_0, t_reg_D_0.y, t_reg_O_0.y);
        t_reg_D_1.x = __fmaf_rn(rescale_o_factor_1, t_reg_D_1.x, t_reg_O_1.x);
        t_reg_D_1.y = __fmaf_rn(rescale_o_factor_1, t_reg_D_1.y, t_reg_O_1.y);
        HALF2(R_D[i][j][0]) = __float22half2_rn(t_reg_D_0);
        HALF2(R_D[i][j][1]) = __float22half2_rn(t_reg_D_1);
      } // end for kWarpTileHeadDimV.

      // Now, we can update m, l after O has been scaled.
      // 1. First, update block row sum Exp for each lane which
      // need both m_new and m_old.
      float block_row_sum_old_0 = lane_block_row_sum_old[i][0];
      float block_row_sum_old_1 = lane_block_row_sum_old[i][1];
      // Update l = exp(m_old - m_new) * l_old + row_sum(P).
      lane_block_row_sum_old[i][0] = (__fmaf_rn(
          rescale_o_factor_0, block_row_sum_old_0, block_row_sum_new_0));
      lane_block_row_sum_old[i][1] = (__fmaf_rn(
          rescale_o_factor_1, block_row_sum_old_1, block_row_sum_new_1));
      // 2. Then, update block row max for each lane.
      lane_block_row_max_old[i][0] = block_row_max_new_0;
      lane_block_row_max_old[i][1] = block_row_max_new_1;
    }

    // NOTE: After compute P @ V, we have to wait next K tile ready in smem.
    // do not need to wait any things if kStage == 1.
    if constexpr (kStage > 1) {
      if ((tile_K_seqlen + 1) < Tc) {
        CP_ASYNC_WAIT_GROUP(0);
      }
      __syncthreads();
    }

  } // end loop over N
  __syncthreads();

// Finaly, we still have to rescale O once more.
// O_output(D) = ( 1/l_final ) * O_final (FA2 paper)
#pragma unroll
  for (int i = 0; i < kWarpTileSeqLenP; ++i) {
    float rescale_factor_0 = __frcp_rn(lane_block_row_sum_old[i][0]);
    float rescale_factor_1 = __frcp_rn(lane_block_row_sum_old[i][1]);
#pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) {
      float2 t_reg_D_0 = __half22float2(HALF2(R_D[i][j][0])); // 0~7  {c0, c1}
      float2 t_reg_D_1 = __half22float2(HALF2(R_D[i][j][1])); // 8~15 {c2, c3}
      t_reg_D_0.x = rescale_factor_0 * t_reg_D_0.x;
      t_reg_D_0.y = rescale_factor_0 * t_reg_D_0.y;
      t_reg_D_1.x = rescale_factor_1 * t_reg_D_1.x;
      t_reg_D_1.y = rescale_factor_1 * t_reg_D_1.y;
      HALF2(R_D[i][j][0]) = __float22half2_rn(t_reg_D_0);
      HALF2(R_D[i][j][1]) = __float22half2_rn(t_reg_D_1);
    }
  }

// Store O(D): Write O[Br,d] from regs -> gmem, collective store
// with reg reuse & warp shuffle. need R[2][4], may reuse
// R_Q[kWarpTileSeqLenQ][4]=[2][4].
#pragma unroll
  for (int i = 0; i < kWarpTileSeqLenP; ++i) {
#pragma unroll
    for (int j = 0; j < kWarpTileHeadDimV; ++j) {
      static_assert(kWarpTileSeqLenQ >= 2);
      R_Q[0][0] = R_D[i][j][0];
      R_Q[1][0] = R_D[i][j][1]; // warp_size 4
      R_Q[0][1] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 1, 4);
      R_Q[0][2] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 2, 4);
      R_Q[0][3] = __shfl_sync((0xffffffff), R_D[i][j][0], lane_id + 3, 4);
      R_Q[1][1] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 1, 4);
      R_Q[1][2] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 2, 4);
      R_Q[1][3] = __shfl_sync((0xffffffff), R_D[i][j][1], lane_id + 3, 4);

      // st.global.v4 128 bits. [Br,d]
      if (lane_id % 4 == 0) {
        // (0/1)*32 + (0/1)*16=(0,16,32,48), + 0~7 -> 0~56
        int store_warp_regs_O_Br =
            warp_QP * (kMmaAtomM * kWarpTileSeqLenP) + i * kMmaAtomM;
        int store_lane_gmem_O_Br =
            O_tile_id * Br + store_warp_regs_O_Br + lane_id / 4; // 0~7
        // (0~3)*16 + (0/1)*8=(0,8,16,24,...,48,56)
        int store_warp_regs_O_d =
            warp_KV * (kMmaAtomN * kWarpTileHeadDimV) + j * kMmaAtomN;
        int store_lane_gmem_O_d = store_warp_regs_O_d; // (0~3)*16+(0/8)
        int store_gmem_O_addr_0 =
            (O_gmem_offset + (store_lane_gmem_O_Br + 0) * kHeadDim +
             store_lane_gmem_O_d);
        int store_gmem_O_addr_1 =
            (O_gmem_offset + (store_lane_gmem_O_Br + 8) * kHeadDim +
             store_lane_gmem_O_d);
        LDST128BITS(O[store_gmem_O_addr_0]) = LDST128BITS(R_Q[0][0]);
        LDST128BITS(O[store_gmem_O_addr_1]) = LDST128BITS(R_Q[1][0]);
      }
    } // end for kWarpTileHeadDimV
  } // end for kWarpTileSeqLenQ
}

template <const int kHeadDim, const int kStage>
void launch_flash_attn_mma_stages_split_kv(torch::Tensor Q, torch::Tensor K,
                                           torch::Tensor V, torch::Tensor O) {
  // Now: fixed tile BrxBc=64x64
  // TODO: dynamic tile size for Br, Bc according to kHeadDim and shared memory
  // size.
  constexpr int kMmaAtomM = 16;
  constexpr int kMmaAtomN = 8;
  constexpr int kMmaAtomK = 16;
  constexpr int kMmaTileSeqLenQ = 2;
  constexpr int kMmaTileSeqLenP = 2;
  constexpr int kMmaTileSeqLenK = 4;
  constexpr int kMmaTileHeadDimV = 4;
  constexpr int kWarpTileSeqLenQ = 2;
  constexpr int kWarpTileSeqLenP = 2;
  constexpr int kWarpTileSeqLenK = 2;
  constexpr int kWarpTileHeadDimV = (kHeadDim / (kMmaAtomN * kMmaTileHeadDimV));
  constexpr int Br =
      kMmaAtomM * kMmaTileSeqLenQ * kWarpTileSeqLenQ; // 16*2*2=64
  constexpr int Bc = kMmaAtomN * kMmaTileSeqLenK * kWarpTileSeqLenK; // 8*4*2=64
  constexpr int kNumThreads = WARP_SIZE * kMmaTileSeqLenQ * kMmaTileSeqLenK;
  constexpr int kPad = 8;

  // static int kMaxSramPerBlock;
  // hipDeviceGetAttribute(&kMaxSramPerBlock,
  // hipDeviceAttributeMaxSharedMemoryPerBlock, 0); Calculate SRAM size needed per
  // block, Q,K,V,S smem size
  const int smem_max_size =
      ((Br * (kHeadDim + kPad)) + (kStage * Bc * (kHeadDim + kPad)) +
       (Bc * (kHeadDim + kPad)) + (Br * (Bc + kPad))) *
      sizeof(half);

  const int QKV_batch = Q.size(0);
  const int QKV_head = Q.size(1);
  const int QKV_seqlen = Q.size(2); // QKV_seqlen
  assert(QKV_seqlen % Bc == 0);     // multiple of Bc=64

  // TODO: How to apply block swizzle to improve L2 Cache hit rate?
  // NOTE: reorder (B,H,Tr) -> (Tr,B*H) seems can improve L2 Cache hit rate.
  // This might be because SM schedules blocks starting from the x-dimension.
  // Placing Tr at the forefront ensures that identical KV pairs are placed
  // in consecutive scheduling queues, thereby improving L2 Cache hit rates.
  // Tr(=N/Br), batch_size x num_heads
  dim3 grid(div_ceil(QKV_seqlen, Br), QKV_batch * QKV_head);
  dim3 block(kNumThreads); // 4/8 warps per block

  hipFuncSetAttribute(reinterpret_cast<const void*>(
      flash_attn_mma_stages_split_kv_kernel<
          kHeadDim), kMmaAtomM, kMmaAtomN, kMmaAtomK, kMmaTileSeqLenQ,
          kMmaTileSeqLenK, kMmaTileSeqLenP, kMmaTileHeadDimV, kWarpTileSeqLenQ,
          kWarpTileSeqLenK, kWarpTileSeqLenP, kWarpTileHeadDimV, kStage, kPad>,
      hipFuncAttributeMaxDynamicSharedMemorySize,
      // kMaxSramPerBlock
      98304);

  flash_attn_mma_stages_split_kv_kernel<
      kHeadDim, kMmaAtomM, kMmaAtomN, kMmaAtomK, kMmaTileSeqLenQ,
      kMmaTileSeqLenK, kMmaTileSeqLenP, kMmaTileHeadDimV, kWarpTileSeqLenQ,
      kWarpTileSeqLenK, kWarpTileSeqLenP, kWarpTileHeadDimV, kStage, kPad>
      <<<grid, block, smem_max_size>>>(reinterpret_cast<half *>(Q.data_ptr()),
                                       reinterpret_cast<half *>(K.data_ptr()),
                                       reinterpret_cast<half *>(V.data_ptr()),
                                       reinterpret_cast<half *>(O.data_ptr()),
                                       QKV_seqlen, QKV_head);
}

void flash_attn_mma_stages_split_kv(torch::Tensor Q, torch::Tensor K,
                                    torch::Tensor V, torch::Tensor O,
                                    int stages) {
  CHECK_TORCH_TENSOR_DTYPE(Q, torch::kHalf) // Q [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(K, torch::kHalf) // K [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(V, torch::kHalf) // V [B,H,N,D]
  CHECK_TORCH_TENSOR_DTYPE(O, torch::kHalf) // O [B,H,N,D]
  const int d = Q.size(3);                  // B, H, N, d

  if (stages > 1) {
    switch (d) {
    case 32:
      launch_flash_attn_mma_stages_split_kv<32, 2>(Q, K, V, O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_kv<64, 2>(Q, K, V, O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_kv<96, 2>(Q, K, V, O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_kv<128, 2>(Q, K, V, O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  } else {
    switch (d) {
    case 32:
      launch_flash_attn_mma_stages_split_kv<32, 1>(Q, K, V, O);
      break;
    case 64:
      launch_flash_attn_mma_stages_split_kv<64, 1>(Q, K, V, O);
      break;
    case 96:
      launch_flash_attn_mma_stages_split_kv<96, 1>(Q, K, V, O);
      break;
    case 128:
      launch_flash_attn_mma_stages_split_kv<128, 1>(Q, K, V, O);
      break;
    default:
      throw std::runtime_error("headdim not support!");
      break;
    }
  }
}
