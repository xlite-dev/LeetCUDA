#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])

__global__ void nms_kernel(const float *boxes, const float *scores, int *keep,
                           int num_boxes, float iou_threshold) {
  const int threadsPerBlock = blockDim.x;
  const int threadId = threadIdx.x;
  const int blockId = blockIdx.x;
  const int idx = blockId * threadsPerBlock + threadId;

  if (idx >= num_boxes)
    return;

  float x1 = boxes[idx * 4 + 0];
  float y1 = boxes[idx * 4 + 1];
  float x2 = boxes[idx * 4 + 2];
  float y2 = boxes[idx * 4 + 3];
  int suppressed = 0;

  for (int i = 0; i < idx; ++i) {
    if (keep[i] == 0)
      continue;

    float x1_i = boxes[i * 4 + 0];
    float y1_i = boxes[i * 4 + 1];
    float x2_i = boxes[i * 4 + 2];
    float y2_i = boxes[i * 4 + 3];

    float inter_x1 = max(x1, x1_i);
    float inter_y1 = max(y1, y1_i);
    float inter_x2 = min(x2, x2_i);
    float inter_y2 = min(y2, y2_i);
    float inter_w = max(0.0f, inter_x2 - inter_x1);
    float inter_h = max(0.0f, inter_y2 - inter_y1);
    float inter_area = inter_w * inter_h;

    float area = (x2 - x1) * (y2 - y1);
    float area_i = (x2_i - x1_i) * (y2_i - y1_i);
    float iou = inter_area / (area + area_i - inter_area);

    if (iou > iou_threshold) {
      keep[idx] = 0;
      return;
    }
  }
  keep[idx] = 1;
  return;
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

torch::Tensor nms(torch::Tensor boxes, torch::Tensor scores,
                  float iou_threshold) {
  CHECK_TORCH_TENSOR_DTYPE(boxes, torch::kFloat32);
  CHECK_TORCH_TENSOR_DTYPE(scores, torch::kFloat32);
  const int num_boxes = boxes.size(0);
  auto toption =
      torch::TensorOptions().dtype(torch::kInt32).device(boxes.device());
  auto keep = torch::empty({boxes.size(0)}, toption);
  dim3 block(WARP_SIZE);
  dim3 grid((num_boxes + WARP_SIZE - 1) / WARP_SIZE);
  // sort boxes by scores
  auto order_t = std::get<1>(
      scores.sort(/*stable=*/true, /*dim=*/0, /* descending=*/true));
  auto boxes_sorted = boxes.index_select(0, order_t).contiguous();

  nms_kernel<<<grid, block>>>(
      reinterpret_cast<float *>(boxes_sorted.data_ptr()),
      reinterpret_cast<float *>(scores.data_ptr()),
      reinterpret_cast<int *>(keep.data_ptr()), num_boxes, iou_threshold);
  auto keep_cpu = keep.to(torch::kCPU);

  std::vector<int> keep_indices;
  auto keep_accessor = keep_cpu.accessor<int, 1>();
  for (int i = 0; i < num_boxes; ++i) {
    if (keep_accessor[i] == 1) {
      keep_indices.push_back(i);
    }
  }
  return torch::tensor(keep_indices,
                       torch::TensorOptions().dtype(torch::kInt32));
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) { TORCH_BINDING_COMMON_EXTENSION(nms) }
